#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8) {
if (comp == (var_2 / -1.6323E3f)) {
  for (int i=0; i < var_1; ++i) {
    comp = var_3 / var_4;
if (comp > -1.2094E-17f / var_5 - -1.7201E-36f) {
  float tmp_1 = +1.4236E-35f;
float tmp_2 = +1.6406E35f;
float tmp_3 = var_6 / +0.0f - (var_7 - (+1.4527E-41f * +1.5682E-43f));
comp += tmp_3 + tmp_2 + tmp_1 + (+1.9705E-36f * (var_8 - +1.7016E28f));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9);
  hipDeviceSynchronize();

  return 0;
}
