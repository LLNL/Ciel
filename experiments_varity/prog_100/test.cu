#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = var_3 / var_4 - fabsf(fabsf((var_5 * (var_6 * (-1.6306E34f / +1.2050E-43f)))));
comp = tmp_1 + (var_7 - +1.8558E-43f);
comp += (+1.7264E-36f * expf(+1.8319E-41f + (var_8 / -1.9151E27f)));
comp = (var_9 - +1.0505E-36f + (+1.2561E-15f - var_10 - +1.9349E-44f));
if (comp >= (var_11 / var_12 * var_13)) {
  comp += (var_14 * (-0.0f - var_15 - (var_16 / var_17)));
comp = +1.1422E-36f - var_18 + -1.9082E-36f;
}
for (int i=0; i < var_2; ++i) {
  comp = var_19 * coshf(+1.3859E35f / +1.6905E36f);
}
if (comp <= -1.5568E-42f / cosf((-1.5948E-37f * var_20 * var_21))) {
  comp += var_22 * var_23 / (var_24 - floorf((var_25 * -1.0570E34f)));
float tmp_2 = +1.7322E-36f;
comp += tmp_2 - +1.7309E35f + (+1.8079E34f / var_26);
comp = (-1.3137E35f * expf(+1.8844E36f));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27);
  hipDeviceSynchronize();

  return 0;
}
