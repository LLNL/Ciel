#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11) {
float tmp_1 = var_1 - atanf((var_2 - +0.0f + var_3));
comp += tmp_1 * var_4 * var_5 + var_6 + var_7;
if (comp > +1.0953E-43f * (+1.7008E0f - -1.6158E-35f)) {
  float tmp_2 = (var_8 - +1.3722E10f);
float tmp_3 = (+1.6209E-43f - var_9);
comp = tmp_3 / tmp_2 + (var_10 * (+1.5338E-43f * +0.0f + var_11 - +1.0263E11f));
}
if (comp > +0.0f + +1.7649E11f) {
  comp += (-1.6861E-42f * +1.9745E36f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12);
  hipDeviceSynchronize();

  return 0;
}
