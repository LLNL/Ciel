#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float* var_13,float* var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
float tmp_1 = (var_3 * +0.0f * var_4);
comp += tmp_1 * var_5 * +1.7524E35f * var_6;
comp = +1.1666E-37f + (-1.7689E-36f + var_7 + -0.0f + +1.9863E-37f);
for (int i=0; i < var_1; ++i) {
  comp = asinf((-1.1271E-35f + var_8 - var_9 / -1.2014E-25f));
float tmp_2 = -0.0f;
comp = tmp_2 / (var_10 * var_11 + (-1.8626E-41f / (+0.0f - var_12)));
}
for (int i=0; i < var_2; ++i) {
  var_13[i] = -1.4305E-43f;
var_14[i] = asinf(asinf(-1.9549E-35f / (var_15 * (var_16 * -1.1807E35f * coshf(-1.8153E-26f / -1.4167E15f / atanf(+1.4834E-44f))))));
comp += var_14[i] - var_13[i] - +1.6842E26f / sinhf(sinhf(fmodf(atanf(var_17 * var_18), -1.0377E-42f / atanf(+1.2541E-43f + (var_19 / var_20)))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float* tmp_14 = initPointer( atof(argv[14]) );
  float* tmp_15 = initPointer( atof(argv[15]) );
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
