#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float* var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30) {
for (int i=0; i < var_1; ++i) {
  comp = (+1.8831E12f * ceilf(var_5 - +1.4037E34f + var_6 - var_7));
float tmp_1 = +0.0f;
comp += tmp_1 / (var_8 - (var_9 / +1.6336E22f));
comp = ldexpf(var_10 * +1.9790E36f, 2);
for (int i=0; i < var_2; ++i) {
  comp += var_11 - +0.0f / -1.2610E-41f / (var_12 + (var_13 - +1.3449E19f));
comp += (var_14 - -1.9324E-44f * var_15);
}
for (int i=0; i < var_3; ++i) {
  var_16[i] = +1.8726E-41f - powf((+1.5852E-24f - (var_17 * +1.3628E22f / +1.3281E-41f + -1.5103E-41f * var_18)), (var_19 * (var_20 / (-1.6208E-37f / +1.3022E-37f / var_21))));
comp = var_16[i] / (var_22 - (var_23 * ceilf(var_24 * tanhf((+1.6962E36f + (var_25 / var_26 * var_27 / (var_28 * var_29)))))));
}
for (int i=0; i < var_4; ++i) {
  comp += (+0.0f - (var_30 - (-0.0f + log10f((+1.0330E35f - -1.9413E34f)))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float* tmp_17 = initPointer( atof(argv[17]) );
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31);
  hipDeviceSynchronize();

  return 0;
}
