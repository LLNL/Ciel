#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,int var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
if (comp < var_1 * (+1.2872E-36f + (-1.0993E-43f - (var_2 * sinhf((+1.2431E35f * +1.0740E35f - (-0.0f / (var_3 - var_4 + var_5)))))))) {
  float tmp_1 = +1.0345E-44f;
comp = tmp_1 + tanhf(var_7 * sinf(coshf(+1.5656E-24f - (+1.8390E-35f * -1.2912E16f - -1.9283E-44f))));
comp = (var_8 + ceilf(var_9 - -0.0f));
comp = (+0.0f / (-1.4405E-43f + var_10 * var_11 * +1.5392E-36f));
for (int i=0; i < var_6; ++i) {
  comp += +1.7973E-14f / +1.0895E36f / -1.6782E-3f / -1.2084E-36f;
comp += fabsf(var_12 * var_13 / -1.7342E-43f + fmodf((var_14 * -0.0f / (var_15 / var_16)), +0.0f + var_17));
comp += +1.3136E36f + ldexpf(+0.0f + var_18 * +1.1474E-37f - -1.7400E17f, 2);
}
if (comp < (-1.3745E-26f + (-1.6333E-37f - var_19))) {
  comp = -1.9237E6f * var_20 - -1.9746E-37f;
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  int tmp_7 = atoi(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
