#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float* var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27) {
float tmp_1 = (var_2 / cosf(var_3 + +1.2924E-37f - +0.0f - (var_4 * var_5 + var_6)));
comp = tmp_1 * -1.9477E34f / var_7 / (+1.6999E-2f - sinf((var_8 - +1.5392E-24f)));
comp = (+1.7508E36f * (+1.9787E13f + var_9));
for (int i=0; i < var_1; ++i) {
  float tmp_2 = cosf((var_11 - -1.7486E23f));
var_10[i] = +0.0f;
comp += var_10[i] / tmp_2 - -1.8188E-35f - (-1.6092E-36f + (var_12 - sqrtf((+1.5480E-44f + (-1.8010E26f / (var_13 / (+0.0f + var_14)))))));
}
if (comp > var_15 - (var_16 - (-0.0f / var_17 + var_18))) {
  float tmp_3 = -1.2566E-16f;
comp = tmp_3 * +0.0f + var_19 + (-0.0f + +1.6479E-36f + var_20);
comp += ldexpf(acosf(-1.1331E-12f * (-1.3463E-37f * +1.6486E-37f - +1.2464E-41f)), 2);
}
if (comp >= -1.9416E-5f / (-1.7141E-9f - (var_21 / ceilf(-0.0f)))) {
  comp += (+1.0532E-37f * (var_22 - -0.0f + sinf(var_23 - (var_24 - -1.7583E-26f))));
float tmp_4 = (var_25 + +1.3359E35f + (var_26 - var_27 * -1.7313E-35f));
comp = tmp_4 / +1.7720E36f * (+1.3877E-9f / ldexpf(+1.8074E-36f, 2));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float* tmp_11 = initPointer( atof(argv[11]) );
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28);
  hipDeviceSynchronize();

  return 0;
}
