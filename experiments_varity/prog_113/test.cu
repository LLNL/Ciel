#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = -1.5914E-43f;
float tmp_2 = (-1.3808E-35f / ceilf((-1.0851E-37f * log10f((-1.0437E36f - (var_3 - (-1.9410E-41f * -1.4893E7f + var_4 / +1.7362E-35f)))))));
float tmp_3 = (-0.0f / +1.4785E-8f + var_5);
comp += tmp_3 - tmp_2 * tmp_1 / var_6 + -1.3380E-35f;
for (int i=0; i < var_2; ++i) {
  comp = var_7 * var_8;
}
if (comp == -0.0f / cosf(+1.0663E-37f + -1.8452E-42f - var_9 + logf(+1.5149E-36f / -1.0874E19f * logf(+1.7400E13f)))) {
  float tmp_4 = -1.3328E-36f;
float tmp_5 = floorf((var_10 + (var_11 * (var_12 - ldexpf((+1.2927E-35f - (var_13 - (var_14 * +1.4363E-42f + var_15))), 2)))));
comp += tmp_5 / tmp_4 / var_16 * (-1.9099E36f + var_17);
}
if (comp >= +1.7018E-44f + +1.3527E-37f) {
  comp += +1.7350E-42f + expf(tanhf((var_18 - +1.0340E-41f)));
comp += atan2f((+0.0f / (var_19 + var_20 - +1.1250E34f / floorf(+1.6097E36f / fmodf(+1.2477E-8f, sqrtf(atan2f(+1.1544E-43f / +1.5844E-43f, -0.0f)))))), (+1.5745E-14f * floorf((+1.8944E-43f + -1.6229E-35f - var_21 - var_22))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
