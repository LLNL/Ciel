#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18) {
float tmp_1 = +1.2394E35f;
comp = tmp_1 * +1.5729E-37f - cosf(log10f(+1.1134E-24f));
float tmp_2 = (var_2 - var_3);
comp += tmp_2 / logf((var_4 - (var_5 * sqrtf(+1.5927E-44f))));
if (comp == (-0.0f * (-1.7980E-13f * (var_6 - (var_7 + -1.5265E-35f))))) {
  comp += (+1.4852E-37f - var_8 * (-1.9264E-42f * (-1.8978E-41f - (var_9 + +0.0f))));
float tmp_3 = -1.1928E5f;
float tmp_4 = -1.9871E-21f;
comp = tmp_4 / tmp_3 + (var_10 - ceilf(coshf(var_11 / +0.0f + atan2f((var_12 - (var_13 + expf(sinf((-1.3683E-43f * +1.5639E-36f + (var_14 / -1.0849E20f - +1.7334E-13f + +1.3758E-7f)))))), -1.7151E-37f))));
}
for (int i=0; i < var_1; ++i) {
  comp += (+1.8253E-43f + (var_15 - (var_16 + +0.0f - -1.6105E36f)));
comp += -1.9053E-37f * -1.7022E-36f + (var_17 - var_18);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19);
  hipDeviceSynchronize();

  return 0;
}
