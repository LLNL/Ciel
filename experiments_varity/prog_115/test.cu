#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
float tmp_1 = var_3 + var_4;
float tmp_2 = +1.9720E12f;
float tmp_3 = +1.2828E-21f;
comp = tmp_3 + tmp_2 - tmp_1 / var_5 * var_6 / atan2f((var_7 / (-1.4202E-41f - -1.5509E-43f * var_8)), (var_9 / coshf((var_10 / (+1.4353E21f + (var_11 / +1.4272E22f / -1.2727E34f - +1.7556E-43f))))));
for (int i=0; i < var_1; ++i) {
  comp += (var_12 * expf(ceilf(var_13 + -1.6095E12f / ldexpf(+1.6936E-36f, 2))));
}
for (int i=0; i < var_2; ++i) {
  comp += +1.8186E6f + logf((-1.8026E-42f - var_14 + (-0.0f * (+1.6522E-44f * var_15 + -1.6749E-35f))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
