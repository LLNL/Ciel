#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(double comp, int var_1,double var_2,double var_3,double var_4,double var_5,double var_6,double var_7,double var_8,double var_9,double var_10,double var_11,double var_12,double var_13,double var_14,double var_15,double var_16,double var_17,double var_18,double var_19,double var_20,double var_21,double var_22) {
comp += -1.4106E13f * -0.0f;
comp += (var_2 * +0.0f);
for (int i=0; i < var_1; ++i) {
  float tmp_1 = expf(var_3 * var_4 + +1.0715E-35f + +1.5554E-42f);
comp += tmp_1 / atanf(+1.5513E-36f);
}
if (comp >= var_5 - var_6 / (var_7 * +1.7223E-37f)) {
  comp = var_8 / +1.1415E16f + var_9;
float tmp_2 = log10f(var_10 / (var_11 - var_12 * (+0.0f - -1.5085E35f / var_13)));
comp = tmp_2 / +1.7958E-37f - (var_14 - (var_15 + +1.0455E-36f - -1.9439E34f));
}
if (comp == sqrtf(-1.1082E-15f / var_16 / +1.9922E-44f - var_17 + -1.9692E28f)) {
  comp += fmodf((+1.4765E-42f / acosf((+1.4805E-44f - (var_18 - (var_19 * sqrtf(sinhf(-1.6459E35f))))))), (var_20 - sqrtf((-0.0f / (+1.0954E21f * var_21 + var_22)))));
float tmp_3 = -1.6980E25f;
comp = tmp_3 + floorf(sinhf(+0.0f));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(double v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
