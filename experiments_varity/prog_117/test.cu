#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21) {
if (comp < var_3 + +1.5921E34f) {
  if (comp >= -1.2111E34f - var_4 / var_5 / coshf((var_6 - -1.9447E-30f))) {
    for (int i=0; i < var_1; ++i) {
      float tmp_1 = -1.2044E-44f;
comp = tmp_1 - +1.5647E-37f * -1.2308E-27f;
comp = (var_7 / -1.4092E-41f - tanhf(var_8 + (var_9 - (-1.0533E-6f + var_10 - -1.0586E-23f))));
for (int i=0; i < var_2; ++i) {
  float tmp_2 = -1.3208E-37f;
float tmp_3 = (var_11 * +1.6128E-37f * (+1.9686E34f + -1.1417E-36f - var_12));
comp += tmp_3 + tmp_2 * (+1.9661E-36f - var_13 - var_14);
}
if (comp < -1.8627E-36f - (+1.7715E-37f + +1.4506E-25f)) {
  comp += (+1.6074E0f + floorf(coshf((+0.0f / (-0.0f - atan2f(+1.7322E19f, +1.4562E-37f / var_15 * -1.7117E34f - +1.8625E-37f - var_16 + -1.1597E17f))))));
float tmp_4 = +1.7180E-41f;
comp = tmp_4 + (+1.8385E9f + log10f((var_17 + acosf(atan2f(logf(+1.9291E17f), -1.2215E35f / var_18 * (-1.4906E12f / -1.8605E-44f))))));
comp += var_19 * var_20 * (var_21 / -1.2871E-42f);
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22);
  hipDeviceSynchronize();

  return 0;
}
