#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float* var_4,float* var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24) {
float tmp_1 = -1.7407E-37f;
comp = tmp_1 * (+1.3653E-44f + (+0.0f - var_2 + (-1.3795E-44f * -0.0f / var_3)));
for (int i=0; i < var_1; ++i) {
  var_4[i] = (var_6 / var_7 - (-1.1248E-42f / +1.6267E35f + +1.8051E2f));
var_5[i] = +1.0201E36f;
comp += var_5[i] + var_4[i] - +0.0f + logf(var_8 + (-1.3514E-17f + +1.2480E28f));
comp += var_9 - (-1.0346E9f - var_10 / +1.1852E-37f + (var_11 * -0.0f));
}
if (comp > var_12 * ldexpf(+0.0f, 2)) {
  float tmp_2 = +1.9453E-36f + (var_13 - (-0.0f - +1.8371E-1f / sinf(var_14 + sqrtf(+1.5774E-36f + var_15 / (+1.9933E1f * (var_16 - (+1.0324E-44f * -0.0f)))))));
comp = tmp_2 * (+1.5923E-27f * +1.7855E19f);
}
if (comp == (-1.0437E34f / +1.6834E-37f + (var_17 - var_18))) {
  float tmp_3 = -1.8511E-36f;
comp = tmp_3 + acosf(-0.0f);
comp += (var_19 * var_20 + powf(atan2f(log10f((var_21 + var_22)), +1.7069E27f / ceilf((var_23 - (+1.1874E35f / +0.0f / (-1.0745E-43f / -1.6644E-13f))))), acosf(-0.0f * -1.1551E-37f - var_24)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float* tmp_5 = initPointer( atof(argv[5]) );
  float* tmp_6 = initPointer( atof(argv[6]) );
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25);
  hipDeviceSynchronize();

  return 0;
}
