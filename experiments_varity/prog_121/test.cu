#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    float tmp_1 = expf(+1.7540E11f);
float tmp_2 = (var_3 * (var_4 - var_5));
comp = tmp_2 + tmp_1 - (var_6 / var_7);
comp += logf(+1.2583E35f);
if (comp == var_8 / var_9 / -1.0420E35f) {
  comp += (+1.4413E-43f - (var_10 - var_11 + fabsf(+1.9447E-42f / (var_12 + -0.0f * -0.0f))));
}
if (comp > tanhf(+1.2154E14f / var_13)) {
  float tmp_3 = var_14 / powf(logf(-1.8003E-37f * var_15 + var_16 + var_17), -1.6451E-42f);
comp += tmp_3 - -0.0f * var_18;
comp += fmodf(-1.9864E-36f, -1.9577E26f);
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19);
  hipDeviceSynchronize();

  return 0;
}
