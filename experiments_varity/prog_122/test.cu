#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23) {
if (comp >= var_3 - tanhf((var_4 * var_5 + var_6))) {
  for (int i=0; i < var_1; ++i) {
    comp = var_7 / fabsf((+1.5544E6f + -1.2034E35f * var_8));
if (comp >= acosf(powf((+1.8476E-36f * (+0.0f * (-1.8124E-41f / +1.1954E-36f))), +0.0f * (var_9 - -1.7000E-36f / +1.1636E-41f)))) {
  comp = -1.7824E-43f + var_10;
float tmp_1 = +1.8058E-26f;
comp = tmp_1 * (var_11 * asinf(+1.5915E-35f * +1.7185E-44f - var_12));
comp += -1.6238E11f / powf(cosf((-1.9095E35f - var_13)), -1.5804E-36f / var_14 - (var_15 - var_16 - var_17));
}
for (int i=0; i < var_2; ++i) {
  comp = (var_18 / var_19);
comp = -1.0345E-44f * (var_20 / (var_21 - var_22 / var_23 / -1.6902E-37f));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24);
  hipDeviceSynchronize();

  return 0;
}
