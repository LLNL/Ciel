#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float* var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    comp = floorf((+1.4947E-37f - (var_4 + atanf((+0.0f + -1.8696E34f + (var_5 + var_6))))));
comp = expf(+1.8374E19f - -1.7572E-42f * (var_7 / var_8));
for (int i=0; i < var_3; ++i) {
  var_9[i] = var_10 - var_11;
comp = var_9[i] - (var_12 - var_13 * var_14 / var_15 + var_16);
}
if (comp < (-1.5008E36f * -0.0f * +0.0f)) {
  comp += +1.5387E-41f / (-1.6833E-42f * (+1.8753E34f * -1.3057E-42f * (+1.4579E34f * +1.9198E-6f)));
float tmp_1 = +1.2768E-41f;
float tmp_2 = -1.5513E34f;
comp = tmp_2 + tmp_1 - var_17 + (+1.9675E-42f - (var_18 / (var_19 / +1.8081E-42f)));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float* tmp_10 = initPointer( atof(argv[10]) );
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20);
  hipDeviceSynchronize();

  return 0;
}
