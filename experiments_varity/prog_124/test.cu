#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24) {
if (comp == (-1.6260E-44f - (-1.2943E-41f * var_3 + +1.6102E-42f))) {
  for (int i=0; i < var_1; ++i) {
    comp += -1.6376E-43f - (var_4 / var_5 + -1.4517E-42f - -0.0f);
comp = (var_6 / log10f(-1.4731E-36f - var_7 / +1.5440E-44f));
float tmp_1 = +1.7830E-36f;
comp = tmp_1 + (-0.0f * fabsf(fabsf(-1.8325E36f / (var_8 + +1.9096E-44f))));
for (int i=0; i < var_2; ++i) {
  float tmp_2 = -1.8887E35f;
comp = tmp_2 + fmodf(var_9 - +1.3094E36f, fmodf((-1.1043E35f + var_10 / var_11 - (var_12 + ceilf((+1.3419E-41f + -0.0f)))), (var_13 + -1.4184E-35f * (-1.7226E-35f + (var_14 - var_15 / var_16)))));
}
if (comp < cosf(powf(var_17 - (var_18 / -1.5893E-44f - (var_19 * +1.9981E34f)), (+1.3788E-28f / -1.9733E-43f / var_20 / var_21)))) {
  float tmp_3 = +1.9394E-16f;
comp += tmp_3 * (-1.1687E-37f + var_22);
comp += var_23 * acosf(sinhf(var_24 * +0.0f));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25);
  hipDeviceSynchronize();

  return 0;
}
