#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14) {
for (int i=0; i < var_1; ++i) {
  comp = (var_2 / (var_3 + +0.0f));
if (comp == (+0.0f / var_4 - -1.9248E-41f)) {
  comp = var_5 - -1.6478E13f * tanhf((-1.8894E36f / acosf((var_6 - (-1.4462E-44f + var_7)))));
comp += (-1.7013E-44f / (-0.0f * +1.0480E-44f));
comp = (+1.6452E9f - var_8 / var_9 - var_10 / var_11);
comp = acosf(+1.4135E-42f + log10f((+1.3245E25f - (-0.0f + coshf(var_12 / (+1.1777E-29f / var_13 * (var_14 - -1.6914E-35f)))))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15);
  hipDeviceSynchronize();

  return 0;
}
