#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float* var_16,float* var_17,float* var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24) {
float tmp_1 = +1.6090E1f;
float tmp_2 = -0.0f;
comp = tmp_2 * tmp_1 * (-1.3900E-37f - (var_3 * -0.0f));
for (int i=0; i < var_1; ++i) {
  float tmp_3 = -1.6793E21f;
comp += tmp_3 * (var_4 + var_5 * floorf(cosf(var_6 / sqrtf(var_7 - var_8))));
}
if (comp <= (var_9 + var_10 + (var_11 / (+1.8532E4f + +1.9727E15f)))) {
  comp += var_12 - -1.4051E-35f + var_13 / (-1.4038E-28f - var_14);
float tmp_4 = +1.2228E-43f;
comp = tmp_4 / -0.0f - var_15;
}
for (int i=0; i < var_2; ++i) {
  float tmp_5 = -0.0f;
  var_16[i] = (var_19 - var_20 + -1.3471E-35f);
var_17[i] = (-0.0f - (var_21 + +1.8118E-23f + -1.6733E-16f));
var_18[i] = (var_22 - var_23 + -1.6458E4f);
tmp_5 = var_18[i];
comp = tmp_5 / var_17[i] - var_16[i] / tanhf(var_24 + +1.3433E35f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float* tmp_17 = initPointer( atof(argv[17]) );
  float* tmp_18 = initPointer( atof(argv[18]) );
  float* tmp_19 = initPointer( atof(argv[19]) );
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25);
  hipDeviceSynchronize();

  return 0;
}
