#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float* var_12,float var_13) {
for (int i=0; i < var_1; ++i) {
  comp = +1.5683E-36f + (+0.0f + var_4);
float tmp_1 = +1.9481E-36f;
comp += tmp_1 * var_5 + (var_6 * -0.0f);
comp = var_7 * (var_8 * sinhf(var_9 + (var_10 / var_11)));
for (int i=0; i < var_2; ++i) {
  var_12[i] = +1.1261E35f;
comp += var_12[i] + (+1.7149E-42f + (-0.0f + -1.1505E-35f));
}
for (int i=0; i < var_3; ++i) {
  comp += (+1.8887E36f * var_13);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float* tmp_13 = initPointer( atof(argv[13]) );
  float tmp_14 = atof(argv[14]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14);
  hipDeviceSynchronize();

  return 0;
}
