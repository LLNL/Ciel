#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = +1.8516E35f;
comp += tmp_1 / (var_2 - +1.3000E-42f + var_3);
float tmp_2 = -1.7066E-42f;
comp = tmp_2 * (+1.9909E36f + (+0.0f - var_4 + var_5 / (+1.5485E17f * var_6)));
if (comp == (var_7 * (var_8 * (-1.5471E34f / (+1.1861E-37f - var_9))))) {
  comp += +1.5187E-36f / (+1.2066E35f - +0.0f);
comp = var_10 - atanf((var_11 + var_12 * -1.1022E-43f));
comp = -1.6505E24f / +1.7740E-35f - (+0.0f + -1.5456E34f / -0.0f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13);
  hipDeviceSynchronize();

  return 0;
}
