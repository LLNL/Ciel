#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11) {
comp += (-1.1816E-43f / tanhf(ceilf(-1.8763E34f)));
comp += logf((-1.3790E-36f / -1.8591E-44f));
for (int i=0; i < var_1; ++i) {
  float tmp_1 = +1.2750E8f;
comp += tmp_1 * (+1.1385E-43f / var_2 - var_3);
float tmp_2 = +0.0f;
comp = tmp_2 * var_4 * (var_5 * (+0.0f * var_6));
}
if (comp <= (var_7 * acosf((+1.5584E-35f / (+1.5569E-35f + (+1.2079E-24f + var_8)))))) {
  comp += (-1.7248E35f * powf(+0.0f, (-0.0f + var_9 - +1.7443E-43f / -1.1671E7f)));
float tmp_3 = +1.3717E34f;
comp = tmp_3 / var_10 + sinhf(-1.0742E-37f);
comp = (var_11 * -0.0f / coshf(-1.7947E-43f));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12);
  hipDeviceSynchronize();

  return 0;
}
