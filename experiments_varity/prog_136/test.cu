#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14) {
float tmp_1 = (+1.2464E29f / +1.9244E-44f);
float tmp_2 = var_2 / +1.8943E-44f - (var_3 - -1.1311E14f);
comp = tmp_2 - tmp_1 + -1.1825E-36f * var_4 * -1.1172E-37f;
if (comp < (var_5 - ldexpf(atanf(var_6 * -1.3105E-35f * (+1.6858E-35f * var_7 - (+1.3891E35f - var_8))), 2))) {
  comp += ceilf(-0.0f + +0.0f + -1.6484E-27f);
}
for (int i=0; i < var_1; ++i) {
  comp = (+1.7501E34f + (-1.5713E36f * +1.9343E-37f * -1.4231E-42f));
comp = (var_9 / var_10);
}
if (comp <= (+1.2267E23f / -1.3600E-44f + (+1.0749E-4f - var_11 / (-1.5771E36f / var_12)))) {
  comp = (var_13 - (var_14 - sinhf(+1.2679E-37f / -1.6524E-18f)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15);
  hipDeviceSynchronize();

  return 0;
}
