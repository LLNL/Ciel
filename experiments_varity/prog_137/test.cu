#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
float tmp_1 = -1.0724E35f;
comp = tmp_1 + (+1.7351E-25f + -1.3599E34f - -1.7297E-35f / var_2);
comp += (var_3 * var_4);
for (int i=0; i < var_1; ++i) {
  comp = var_5 / tanhf((+1.1204E36f / var_6 + (-1.2426E-35f * var_7 * atanf((-1.6000E36f - sqrtf(-1.2593E-44f - -1.8579E6f - var_8 / (var_9 + var_10 - var_11)))))));
comp += (-1.5310E-35f * (-1.4679E-1f * powf((-1.9050E-27f - +1.6574E-25f - var_12), logf(+1.9970E-43f))));
comp = -1.5322E-44f + +1.0538E36f * (var_13 + var_14 / var_15);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
