#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30,float var_31) {
comp = +1.0262E-21f + tanhf((var_3 - -1.8370E6f + (+1.3846E36f - expf(+1.7265E-37f + -1.3232E36f))));
float tmp_1 = -1.2115E-17f;
comp = tmp_1 / var_4 / (var_5 + (var_6 - var_7));
if (comp == expf(-1.0314E-37f * (var_8 / +1.2603E-35f))) {
  float tmp_2 = -1.2533E36f;
float tmp_3 = (var_9 * asinf((var_10 - var_11 * (var_12 - +0.0f))));
comp = tmp_3 + tmp_2 * (+1.9068E-44f * +0.0f * -1.3626E34f);
comp += sinf(ldexpf(+1.2663E-29f * powf(-0.0f, var_13 - +1.7495E29f + -1.5401E-37f + (var_14 + (var_15 * var_16))), 2));
}
for (int i=0; i < var_1; ++i) {
  comp = acosf(-1.2047E-37f * var_17 - var_18);
comp = cosf(sinhf(+1.8500E25f));
comp = acosf((var_19 + +1.7544E35f));
}
for (int i=0; i < var_2; ++i) {
  comp = var_20 - (-1.8848E35f - sqrtf((+1.4556E35f - (+1.6375E-37f - var_21 - -1.9039E34f + var_22 + var_23))));
comp = fmodf(+0.0f + var_24, (+1.0279E-37f / (var_25 - (var_26 + atan2f((var_27 - logf((var_28 / -0.0f * (var_29 + (var_30 * (+1.5087E34f * var_31)))))), +1.9838E21f)))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32);
  hipDeviceSynchronize();

  return 0;
}
