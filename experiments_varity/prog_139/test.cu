#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float* var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = +1.5858E35f;
comp += tmp_1 / (+0.0f + (var_3 + -0.0f));
var_2[i] = (-1.8322E-37f * (var_4 / (var_5 - var_6 + var_7)));
comp += var_2[i] * (var_8 - log10f(+1.3675E-37f * (-1.7369E-35f + +1.3704E-42f)));
if (comp > (var_9 - atan2f(+1.4169E-44f, (var_10 - (var_11 / +0.0f * tanhf((var_12 - powf((var_13 / var_14), -1.9132E-36f)))))))) {
  comp += (-1.2830E15f / -1.3423E-35f);
comp += +1.5729E-37f - -1.3527E-41f + var_15 - +0.0f * ldexpf(var_16 * -1.4488E36f, 2);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float* tmp_3 = initPointer( atof(argv[3]) );
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
