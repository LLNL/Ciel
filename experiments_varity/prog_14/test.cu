#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,int var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    for (int i=0; i < var_3; ++i) {
      for (int i=0; i < var_4; ++i) {
        comp = floorf((-0.0f + var_6 / var_7 + var_8 + -1.0887E35f));
comp += (var_9 - (-1.7852E-43f + (var_10 - var_11)));
comp = (+1.7140E12f * var_12 / (var_13 / (var_14 + +1.4414E34f)));
if (comp < var_15 / var_16) {
  comp = (+1.3538E-43f - var_17 + var_18 - +1.5976E36f * +1.4835E-18f);
}
for (int i=0; i < var_5; ++i) {
  comp = (-1.5892E-41f - log10f(var_19 - (var_20 / -1.9491E-35f / (-0.0f + -1.2808E35f))));
comp += (var_21 * var_22 + (var_23 * tanhf(+1.2446E-43f - sqrtf((+1.7592E23f / -0.0f)))));
comp = -1.7490E-37f - (-1.2559E-37f / (-0.0f * var_24 + var_25));
comp += (var_26 * ceilf(+0.0f));
}
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27);
  hipDeviceSynchronize();

  return 0;
}
