#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,int var_7,int var_8,int var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float* var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30) {
if (comp >= (-1.6656E23f - (var_2 + +1.8501E-43f * (-1.6050E-35f / (-1.3931E-35f - var_3))))) {
  if (comp > -0.0f - (-0.0f * var_4 + var_5)) {
    if (comp < (+1.7990E15f + var_6)) {
      for (int i=0; i < var_1; ++i) {
        comp = -1.5762E-36f / var_10 + (var_11 - var_12);
comp += (var_13 / var_14 / -1.4057E36f);
for (int i=0; i < var_7; ++i) {
  float tmp_1 = (+1.1568E-37f + ldexpf(-1.9930E-35f * sinf((+1.1203E34f + (-1.6866E-43f + (var_15 / var_16)))), 2));
comp += tmp_1 / ldexpf(-1.5442E-36f / -1.6635E34f - var_17, 2);
}
for (int i=0; i < var_8; ++i) {
  var_18[i] = var_19 / coshf(tanhf(ceilf(var_20 - var_21)));
float tmp_2 = logf(var_22 / logf(-1.6411E27f + var_23 - -1.5443E4f));
comp = tmp_2 * var_18[i] - -0.0f + var_24 / +1.5382E16f;
comp = acosf(var_25 - +1.4894E35f / +1.1725E-35f);
}
for (int i=0; i < var_9; ++i) {
  comp += sqrtf(var_26 - logf((-1.4186E-36f * (var_27 - +1.9571E-16f / (var_28 / (var_29 - var_30))))));
}
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  int tmp_8 = atoi(argv[8]);
  int tmp_9 = atoi(argv[9]);
  int tmp_10 = atoi(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float* tmp_19 = initPointer( atof(argv[19]) );
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31);
  hipDeviceSynchronize();

  return 0;
}
