#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27) {
for (int i=0; i < var_1; ++i) {
  comp = cosf((+1.9055E34f - (var_2 + (+1.0827E-35f / var_3))));
comp += -1.1041E-35f + +1.0652E-41f / ldexpf(+1.7815E-36f, 2);
if (comp < floorf(asinf((var_4 - (+1.7791E-29f * var_5))))) {
  float tmp_1 = -1.1839E-41f;
comp += tmp_1 * ceilf(var_6 + (var_7 / (var_8 / var_9 - var_10 + -1.2745E34f)));
comp = atan2f(var_11 / expf(-1.9505E35f * +0.0f * var_12 + sinhf((var_13 - asinf((+0.0f / +1.5375E-42f * atan2f(+1.0716E19f * +1.8304E34f - var_14 - (var_15 * cosf((var_16 / (-0.0f * (+1.1933E6f / (var_17 * expf(cosf((var_18 + +1.6152E-43f + ceilf(var_19 * var_20 - (+1.8872E-42f / +1.5771E34f / (var_21 + var_22)))))))))))), var_23 - var_24)))))), atan2f(+0.0f + +0.0f + (+1.6800E10f + tanhf(+1.5768E34f)), ceilf(+1.7871E-36f * (var_25 / tanhf((var_26 + (+0.0f / var_27)))))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28);
  hipDeviceSynchronize();

  return 0;
}
