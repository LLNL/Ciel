#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    if (comp == (+1.5313E-36f / ceilf((+1.6284E36f - (+1.0025E-41f * -1.2651E8f + (var_3 - -1.7601E35f)))))) {
      if (comp >= var_4 + var_5) {
        comp += +1.8030E-43f / var_6 - (-1.0470E-42f - (var_7 - var_8 + -1.4209E-35f));
if (comp == -1.5788E35f + var_9 / +1.0406E34f) {
  float tmp_1 = -1.7972E-42f * var_10 / -0.0f - var_11;
comp = tmp_1 - var_12 * acosf((var_13 + log10f(var_14 - atan2f(+1.8996E-36f - (-1.8740E-13f * +1.2035E-43f), (-1.0843E-35f + var_15)))));
}
if (comp <= (-1.4347E-43f + -1.7775E-5f)) {
  comp = (-1.0505E34f / var_16);
comp = asinf((var_17 - coshf(-1.8440E-43f)));
comp += coshf((var_18 * var_19 - var_20 / +1.9798E36f * -1.2569E34f * -1.6270E5f));
}
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
