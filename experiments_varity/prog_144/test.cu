#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30,float var_31,float var_32,float var_33,float var_34,float var_35,float var_36,float var_37,float var_38,float var_39,float var_40,float var_41,float var_42) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = sinhf(var_3 * -1.4551E-36f + -1.2625E35f - -1.4661E-43f);
float tmp_2 = (-0.0f + cosf(-1.0306E-37f));
comp += tmp_2 * tmp_1 + var_4 * var_5 * -1.8288E35f;
for (int i=0; i < var_2; ++i) {
  comp += +0.0f - var_6;
}
if (comp >= (var_7 + var_8 - var_9)) {
  comp += var_10 * fabsf(+1.1972E-36f / (-0.0f / (var_11 + +0.0f - var_12)));
comp = (+1.8549E-12f * -1.8673E-44f / var_13);
comp = (-0.0f * (var_14 * -1.6955E-44f + var_15 * var_16));
comp += (var_17 + -1.2955E34f / atanf(ceilf(var_18 + -1.9031E-43f)));
}
if (comp >= (var_19 + -1.8208E34f / var_20 * var_21 * +1.0832E-43f * var_22)) {
  comp = var_23 - powf(floorf((var_24 / (var_25 * expf((var_26 * (-1.5311E-35f + -1.8166E-44f - logf(var_27 / (var_28 / (var_29 + var_30))))))))), atan2f((+1.9611E-36f - log10f((var_31 / var_32 / -1.8654E34f))), var_33 * (var_34 + (+1.2061E-8f + fmodf(var_35 * -0.0f - var_36 / asinf(var_37 / (+1.0358E-43f / -0.0f / (-1.3233E35f * -1.9166E11f * +0.0f))), (-1.9461E-44f / var_38 / +1.0213E-41f))))));
comp += (-1.5714E-44f * (var_39 - var_40 + var_41 / var_42));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);
  float tmp_33 = atof(argv[33]);
  float tmp_34 = atof(argv[34]);
  float tmp_35 = atof(argv[35]);
  float tmp_36 = atof(argv[36]);
  float tmp_37 = atof(argv[37]);
  float tmp_38 = atof(argv[38]);
  float tmp_39 = atof(argv[39]);
  float tmp_40 = atof(argv[40]);
  float tmp_41 = atof(argv[41]);
  float tmp_42 = atof(argv[42]);
  float tmp_43 = atof(argv[43]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32,tmp_33,tmp_34,tmp_35,tmp_36,tmp_37,tmp_38,tmp_39,tmp_40,tmp_41,tmp_42,tmp_43);
  hipDeviceSynchronize();

  return 0;
}
