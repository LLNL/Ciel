#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    comp = +1.2244E-24f * +0.0f - var_4 + (var_5 * +1.2794E-42f);
if (comp == tanhf((var_6 + (-0.0f / var_7)))) {
  comp += sinf((var_8 - -1.5308E-37f / var_9));
}
for (int i=0; i < var_3; ++i) {
  comp = (var_10 + (var_11 * fmodf(expf(+1.3940E-44f), atan2f(-1.9665E-44f * +1.2026E-41f + (+1.4897E-35f * var_12), +1.7929E12f))));
}
if (comp < (-1.2154E-36f + var_13 * var_14)) {
  comp += (+1.7948E-44f * (var_15 / +1.9756E-7f));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
