#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = +1.3264E-42f;
comp += tmp_1 * (-1.9199E9f / var_3 * var_4 + var_5 + +1.5096E7f);
comp = (+0.0f + sinf(-1.5775E-41f));
for (int i=0; i < var_2; ++i) {
  comp = var_6 - var_7;
}
if (comp >= (var_8 * var_9 - -1.2121E28f * (var_10 - var_11))) {
  float tmp_2 = (var_12 * (var_13 + (var_14 - (+0.0f + log10f(+1.5220E16f + +1.0982E-15f - var_15)))));
float tmp_3 = floorf(var_16 * var_17);
float tmp_4 = (var_18 - (-0.0f + +1.3257E-41f / (var_19 / var_20)));
comp = tmp_4 * tmp_3 - tmp_2 - (+1.3525E-41f * (+1.2621E34f * (-1.5093E-4f / sinf((var_21 + (+1.1503E-41f + sinhf(-1.3495E-6f)))))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22);
  hipDeviceSynchronize();

  return 0;
}
