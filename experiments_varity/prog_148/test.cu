#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19) {
float tmp_1 = coshf(acosf(var_3 + log10f(cosf((-1.9336E-44f + (-1.9337E-20f * +0.0f * var_4))))));
comp = tmp_1 * var_5 * log10f(-1.5986E36f / expf(acosf(asinf((var_6 + (var_7 * var_8 + +1.4863E9f))))));
for (int i=0; i < var_1; ++i) {
  float tmp_2 = -1.4378E-41f;
comp += tmp_2 - (-1.2722E-41f + (var_9 - (+0.0f + var_10 - (var_11 * var_12))));
comp += (var_13 * var_14);
comp = coshf((-1.5624E-44f * -1.8369E-37f / (var_15 + -1.9608E1f + var_16)));
}
if (comp == (-1.6344E-37f / +1.6938E36f)) {
  comp += +1.9019E-41f + +1.6335E-36f * var_17 / +0.0f + -1.3284E-42f;
float tmp_3 = +1.5643E36f;
comp += tmp_3 / cosf((+1.1861E-44f - +1.8515E2f));
}
for (int i=0; i < var_2; ++i) {
  comp += +0.0f + +1.4281E-37f;
comp = +1.1717E-44f + var_18 * var_19;
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20);
  hipDeviceSynchronize();

  return 0;
}
