#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
if (comp >= (-1.7566E18f / +1.6715E-36f)) {
  if (comp < (var_1 + (var_2 + (+1.5412E-43f / atanf(tanhf((+1.0319E-35f + atan2f((-1.2109E-4f - +1.3837E36f + coshf(var_3 + +1.6536E34f / var_4 * var_5)), +1.1909E-35f)))))))) {
    comp += logf(var_6 * (var_7 + var_8 / var_9 * +1.9221E-41f));
comp = var_10 / +0.0f;
if (comp >= var_11 * (-1.9011E-42f - sinf((+0.0f + -0.0f - -1.7627E-16f)))) {
  float tmp_1 = -1.9866E-37f;
comp = tmp_1 - (var_12 - powf((+1.5189E34f * +1.7460E-35f), +0.0f));
comp += -0.0f * (var_13 - -0.0f);
comp = fmodf(+1.5411E-8f + var_14, -1.2774E36f / (var_15 / (var_16 / var_17)));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
