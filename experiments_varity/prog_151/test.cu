#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float* var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
comp += (+1.5541E-37f + -1.3636E34f);
comp = ldexpf((var_2 / (var_3 * atan2f(-1.0594E-41f - +1.6283E-36f * -1.9377E-21f - (var_4 / (var_5 / +1.8236E-43f)), -1.5945E-36f + var_6 + (-1.3738E16f / +1.1848E36f)))), 2);
for (int i=0; i < var_1; ++i) {
  comp = var_8 * +1.1670E34f - +1.4449E6f;
var_7[i] = fmodf(powf(-1.6888E-41f, +1.7914E-37f), +0.0f);
comp = var_7[i] - (-1.7142E-14f / -0.0f * cosf(fabsf(floorf((+1.6936E-9f + +1.6304E34f - +1.2672E-41f)))));
comp = var_9 * -0.0f * -1.6677E-36f;
}
if (comp > +0.0f + (-1.7399E-44f - floorf(var_10 / var_11))) {
  comp = tanhf((var_12 - +1.7844E34f));
comp = (+0.0f / sqrtf(var_13 - (var_14 / (-0.0f * acosf(+1.6184E-36f)))));
float tmp_1 = -1.8708E-37f;
comp = tmp_1 + (var_15 / var_16);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float* tmp_8 = initPointer( atof(argv[8]) );
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
