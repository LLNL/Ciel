#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6) {
comp = sinf(+1.0054E-12f - (var_1 * (var_2 * -1.1069E-41f / -0.0f)));
comp = log10f(var_3 * sinf((var_4 + var_5 + ceilf((-1.8343E-20f - var_6)))));
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7);
  hipDeviceSynchronize();

  return 0;
}
