#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23) {
comp = var_2 + var_3 - sinf((var_4 + (+1.2894E-41f / var_5)));
comp = (var_6 * +1.8840E7f + (-1.1719E-36f * (var_7 / (-1.6936E-37f + var_8))));
float tmp_1 = -1.1775E-43f;
comp += tmp_1 * var_9 / (var_10 + -1.1235E17f - var_11);
for (int i=0; i < var_1; ++i) {
  comp = (var_12 - var_13);
}
if (comp > sinf((+1.4881E36f / (-1.4537E-43f * var_14 * (var_15 + log10f((-1.3986E-11f / (-1.5525E-42f / -1.7609E14f + log10f(-1.4843E36f - var_16 * (+0.0f / asinf((var_17 - +1.7630E35f)))))))))))) {
  comp = (var_18 - (+1.2324E-44f * +1.7307E10f));
float tmp_2 = (+1.0020E-35f - var_19 * -1.5871E-44f / atan2f(expf(-1.9903E-17f), -1.1727E36f));
comp += tmp_2 / var_20 * tanhf(var_21 - (var_22 - var_23));
comp = -1.3157E34f / +1.7804E-35f / +1.4261E-37f;
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24);
  hipDeviceSynchronize();

  return 0;
}
