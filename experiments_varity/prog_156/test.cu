#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float* var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19) {
for (int i=0; i < var_1; ++i) {
  comp += (var_4 * -1.7769E-35f / (+1.8131E34f / -1.7173E-35f));
comp += (var_5 + (var_6 / (var_7 - (-1.8712E34f / -0.0f))));
for (int i=0; i < var_2; ++i) {
  var_8[i] = +1.2402E36f;
comp += var_8[i] * (-1.1860E35f + (var_9 * fabsf(+1.2819E-8f)));
}
for (int i=0; i < var_3; ++i) {
  comp = (+0.0f * (var_10 / (-1.1915E34f + var_11 / var_12)));
}
if (comp == (-1.5743E-37f / logf((var_13 - (+1.1502E35f + (+1.9065E-43f / (var_14 / var_15 + var_16))))))) {
  comp += var_17 / var_18 / (var_19 - -1.5312E-26f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float* tmp_9 = initPointer( atof(argv[9]) );
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20);
  hipDeviceSynchronize();

  return 0;
}
