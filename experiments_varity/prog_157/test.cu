#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float* var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10) {
for (int i=0; i < var_1; ++i) {
  var_2[i] = -1.7352E-36f;
comp = var_2[i] + +0.0f * var_3 + -0.0f;
comp += -1.2258E10f - (var_4 + log10f(var_5 - +1.8246E-42f / (var_6 - (var_7 * var_8 - var_9))));
comp += var_10 / -0.0f;
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float* tmp_3 = initPointer( atof(argv[3]) );
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
