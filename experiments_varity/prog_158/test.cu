#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28) {
float tmp_1 = var_1 * -0.0f - var_2 / +0.0f;
float tmp_2 = (var_3 * (var_4 * var_5));
comp += tmp_2 - tmp_1 / +1.0103E-44f * var_6 * var_7 - (+0.0f * sinf(+1.5151E-44f - sqrtf((-0.0f + (var_8 / (var_9 + +0.0f))))));
if (comp <= log10f(atan2f(var_10 / -1.9340E-44f, +1.4925E-18f))) {
  float tmp_3 = +0.0f;
comp = tmp_3 * var_11 - (var_12 * -1.8594E-36f);
}
if (comp == -1.3120E34f + (var_13 / fabsf((var_14 / expf((-1.0795E18f / (+0.0f + atan2f((-0.0f - (var_15 - -1.9336E-37f)), +1.9858E-35f - powf(log10f(-1.5112E-36f), (+1.7540E12f - (-1.6749E25f / +1.1231E-43f / (var_16 - var_17 * var_18)))))))))))) {
  float tmp_4 = (var_19 - (+1.6602E35f + var_20));
comp += tmp_4 - var_21 / var_22 / +1.1368E35f;
comp += (var_23 / sqrtf((-1.5675E-37f + +1.5945E17f)));
}
if (comp == atanf((-1.5185E-43f - var_24))) {
  comp += var_25 * (+1.8088E-37f / -1.7803E35f - (var_26 / (-1.5110E22f - -1.7122E7f)));
float tmp_5 = (-1.2107E-13f + (var_27 / +1.2324E-35f));
comp = tmp_5 - (var_28 / expf(asinf(-0.0f)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29);
  hipDeviceSynchronize();

  return 0;
}
