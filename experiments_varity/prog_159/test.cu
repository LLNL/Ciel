#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2) {
comp += (-1.0551E-19f * (-1.9548E35f - +1.2760E34f * sinf(+1.5972E36f)));
for (int i=0; i < var_1; ++i) {
  comp = (+0.0f / (+1.8596E36f - -1.9720E14f));
comp = var_2 + +1.9419E-44f;
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3);
  hipDeviceSynchronize();

  return 0;
}
