#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
comp += (var_2 - var_3 + var_4 / -1.0902E-37f / +0.0f);
comp = ldexpf((var_5 - +1.2764E-44f), 2);
for (int i=0; i < var_1; ++i) {
  float tmp_1 = +0.0f;
comp += tmp_1 + var_6 * +1.5965E-44f;
comp += sinf((var_7 * +1.8683E35f * var_8 - var_9 * var_10));
}
if (comp <= (var_11 * +0.0f)) {
  comp += (-1.9074E35f + ldexpf(sqrtf(+1.6497E-37f), 2));
comp = (+1.0252E34f / (var_12 * (var_13 - -1.4152E36f / sinhf(+1.6885E-30f))));
comp += fmodf((var_14 + +1.9092E34f / var_15 * (var_16 - (+1.3334E-36f - -0.0f))), tanhf(+1.9956E-36f));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
