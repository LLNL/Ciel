#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
for (int i=0; i < var_1; ++i) {
  if (comp >= var_2 * -1.8049E-43f) {
    float tmp_1 = (+1.9527E36f / (var_3 / sqrtf(+1.5693E-36f - +1.5835E-35f - +1.8234E35f)));
comp += tmp_1 + -1.9299E-37f - floorf(+1.1072E-22f * sqrtf(-1.1609E8f));
comp = log10f((var_4 + fmodf((-1.5779E12f + var_5 - var_6 + var_7), sqrtf((var_8 / +1.7904E36f)))));
comp = (-1.1508E35f / (-0.0f * +1.2612E-36f / var_9));
if (comp > tanhf(ceilf((var_10 + (var_11 * var_12))))) {
  comp += -1.5889E-41f * var_13;
float tmp_2 = atan2f((-1.8460E35f + -1.3465E-35f - -1.9502E28f - var_14), var_15 / ldexpf(-1.4185E-43f, 2));
comp = tmp_2 * (var_16 * -1.5689E-36f);
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
