#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float* var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
float tmp_1 = -0.0f;
comp += tmp_1 + var_2 - var_3 / (var_4 - ceilf((+1.3768E34f - var_5 * var_6 * (+1.5851E-37f / (+0.0f / var_7)))));
for (int i=0; i < var_1; ++i) {
  var_8[i] = var_9 - tanhf(var_10 / var_11 * -0.0f);
comp = var_8[i] - var_12 - var_13 + (+1.6909E-42f - var_14);
comp += atanf(var_15 * +1.3157E-42f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float* tmp_9 = initPointer( atof(argv[9]) );
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
