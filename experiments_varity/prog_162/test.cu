#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8) {
if (comp >= sinf(ceilf(var_1 * ceilf(-1.3211E34f)))) {
  if (comp <= ldexpf(-1.1421E-41f - -1.8646E-36f, 2)) {
    float tmp_1 = -1.8883E-30f;
float tmp_2 = var_3 / +0.0f - (var_4 * -1.8210E4f);
comp = tmp_2 * tmp_1 - asinf(-1.2074E15f - var_5 - (var_6 * var_7));
for (int i=0; i < var_2; ++i) {
  comp += (-1.1728E-37f + +1.5173E-44f);
comp += (-1.1081E24f * -1.3726E-18f * var_8 - -1.2007E22f);
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9);
  hipDeviceSynchronize();

  return 0;
}
