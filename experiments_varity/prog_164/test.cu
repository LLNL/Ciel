#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = -1.1926E-37f;
comp = tmp_1 - tanhf(+1.4245E-35f);
comp += -1.5479E-15f / +1.6065E24f * +1.0225E-25f;
comp = sinhf(+1.0151E-41f + (+1.8537E36f - var_2 / -0.0f));
if (comp < cosf(fmodf(coshf(+1.0752E35f), -1.2361E-43f))) {
  float tmp_2 = -1.6446E-4f;
float tmp_3 = log10f(+0.0f + fabsf(sinf((-1.2535E34f * fmodf(+1.7591E34f, atan2f(-1.7437E35f, +1.9583E35f / var_3 + var_4 - +1.7283E-35f - (-0.0f + +1.2275E-44f)))))));
comp = tmp_3 * tmp_2 * (-1.4203E-44f - (-1.7223E34f - +1.1715E36f * logf(+1.5565E0f)));
comp += -0.0f + var_5;
}
if (comp > (var_6 + -1.2499E34f)) {
  comp = -0.0f + +1.5447E-14f;
}
if (comp < -1.3238E-35f + log10f(cosf(-1.1412E-36f * -1.7173E34f / -0.0f))) {
  comp = fabsf(var_7 - var_8 - var_9 + (-1.8882E-20f / var_10));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
