#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float* var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28) {
float tmp_1 = -0.0f;
comp += tmp_1 + var_2 + log10f(+1.6268E-37f + (var_3 / fabsf((+1.2207E-44f / var_4))));
comp += +1.3976E-36f * +1.2060E-22f;
if (comp == var_5 * ceilf((var_6 / var_7))) {
  comp = -1.2546E-44f + (+1.8689E-41f + cosf(ceilf(var_8 + (-1.7557E-41f / atan2f(-1.4001E8f - var_9, -1.7741E34f)))));
comp += var_10 + (+1.2586E-26f - var_11);
comp += (+0.0f * tanhf(var_12 - (var_13 - var_14 - +0.0f * var_15 + var_16)));
comp = acosf((var_17 * +1.3200E3f + var_18));
}
if (comp == +1.5176E-37f / (var_19 - var_20 * (var_21 - tanhf(+1.6729E-35f)))) {
  comp = +1.4572E-44f * +0.0f / -1.7555E-37f * +1.9372E-14f / +1.1579E-35f * -1.3192E-35f;
}
for (int i=0; i < var_1; ++i) {
  var_22[i] = (var_23 + var_24 - +1.9189E-43f * var_25);
comp += var_22[i] - fabsf((var_26 - floorf((var_27 - var_28))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float* tmp_23 = initPointer( atof(argv[23]) );
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29);
  hipDeviceSynchronize();

  return 0;
}
