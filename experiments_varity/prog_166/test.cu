#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13) {
if (comp <= +1.2245E36f * var_3 / var_4 * (-1.2485E35f / acosf(var_5 - var_6))) {
  float tmp_1 = -1.2996E-42f;
float tmp_2 = +1.9680E34f;
comp += tmp_2 + tmp_1 / var_7 / asinf((-0.0f / -1.2220E-42f));
for (int i=0; i < var_1; ++i) {
  comp = var_8 / (var_9 / +1.3326E-35f);
}
for (int i=0; i < var_2; ++i) {
  comp += -0.0f - var_10;
comp += (-1.4558E-41f + var_11 * var_12);
comp = -1.2444E-41f / (+1.4532E34f * var_13 + -1.6886E-44f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14);
  hipDeviceSynchronize();

  return 0;
}
