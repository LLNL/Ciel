#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29) {
float tmp_1 = (+1.5410E-43f * var_1 * +1.0973E35f + var_2 - (+0.0f - +1.9327E35f));
comp = tmp_1 - (var_3 * +1.7163E36f / var_4 + var_5 + -1.8967E-42f + var_6);
float tmp_2 = (-1.7459E35f - (var_7 * +1.1576E-37f));
comp = tmp_2 / expf(+1.5630E-37f / cosf(+1.3297E-36f - (var_8 - -1.7565E-2f + var_9 - ceilf(var_10 * (var_11 / var_12 + atanf(-1.5981E-37f + +1.3857E-1f - var_13 / atanf(-1.3361E-44f / -1.2044E-44f)))))));
if (comp <= (var_14 - (var_15 * +1.7918E-24f / var_16 - (-0.0f * var_17)))) {
  comp = atanf(atan2f((var_18 - (+1.9056E1f + var_19)), +1.0569E-37f - atan2f(+1.3944E-37f, (-1.0124E35f / var_20 + var_21))));
}
if (comp < (var_22 / fmodf((var_23 / var_24), +1.0852E-35f))) {
  comp += +1.0219E-35f * +0.0f;
comp += sqrtf(+1.7431E-36f);
comp = (var_25 - (+1.6513E35f / +1.1353E15f * var_26 + +1.6437E35f));
comp += var_27 * (var_28 / (var_29 * +1.2062E-42f));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30);
  hipDeviceSynchronize();

  return 0;
}
