#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float* var_13,float var_14,float var_15,float var_16,float var_17,float var_18) {
float tmp_1 = (var_3 / var_4 + +1.4571E36f * var_5 / var_6 + +1.8566E-25f);
comp += tmp_1 * (-1.4927E9f * var_7 + var_8);
for (int i=0; i < var_1; ++i) {
  comp = cosf((-0.0f - coshf(var_9 - (var_10 * -1.4177E34f + (var_11 * var_12)))));
}
for (int i=0; i < var_2; ++i) {
  var_13[i] = -1.5062E-43f;
float tmp_2 = -0.0f;
float tmp_3 = var_14 - var_15 - (var_16 + asinf((+1.3644E-11f + var_17)));
comp = tmp_3 - tmp_2 + var_13[i] + fmodf(coshf((var_18 - +1.2620E-42f * +0.0f)), -1.3425E-43f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float* tmp_14 = initPointer( atof(argv[14]) );
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19);
  hipDeviceSynchronize();

  return 0;
}
