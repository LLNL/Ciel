#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float* var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
for (int i=0; i < var_1; ++i) {
  if (comp >= (var_3 - var_4 - (-1.3841E-36f + +1.0208E-41f / (var_5 - var_6)))) {
    comp += +1.2575E35f + fabsf(var_7 + -0.0f - (var_8 + var_9 - -1.1306E34f * -1.6599E-11f));
for (int i=0; i < var_2; ++i) {
  float tmp_1 = fmodf(+1.1291E34f, (+1.7781E35f + (var_11 + ldexpf(var_12 * (var_13 + +1.9068E36f), 2))));
var_10[i] = -1.6234E-20f;
comp += var_10[i] + tmp_1 * +1.2238E1f * var_14 * (+1.9460E35f * +1.8383E-44f);
}
if (comp >= -0.0f - (+0.0f * (+1.2318E-36f * (var_15 * var_16)))) {
  float tmp_2 = +1.6476E-17f;
float tmp_3 = -1.6120E-44f * var_17 * var_18 - +1.4433E36f / +1.2990E36f;
float tmp_4 = (var_19 + +1.4889E-36f);
comp = tmp_4 / tmp_3 + tmp_2 * var_20 + ldexpf((+1.2313E23f + -1.1584E34f), 2);
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float* tmp_11 = initPointer( atof(argv[11]) );
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
