#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14) {
for (int i=0; i < var_1; ++i) {
  if (comp <= var_3 * -1.0111E-42f) {
    for (int i=0; i < var_2; ++i) {
      comp += +1.9573E35f * var_4 + (var_5 - (-0.0f - var_6));
float tmp_1 = +1.9881E-37f;
comp = tmp_1 - (-1.8594E-43f * var_7 + -1.4813E-43f / -1.4732E34f);
if (comp <= +1.7059E-37f + (-1.1710E-23f / -1.6079E-37f * (var_8 + -0.0f - var_9))) {
  float tmp_2 = -1.9125E19f * (-1.7493E-44f / (var_10 / var_11));
float tmp_3 = -0.0f * (+1.3786E-41f * sqrtf((+1.9976E-42f + fabsf((var_12 + +1.0464E-10f * floorf(sinf(-1.8859E-42f)))))));
comp += tmp_3 * tmp_2 * var_13 * sqrtf(-1.5827E-3f + +1.0035E-35f / -0.0f / (var_14 / -1.1672E34f));
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15);
  hipDeviceSynchronize();

  return 0;
}
