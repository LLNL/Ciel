#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
for (int i=0; i < var_1; ++i) {
  comp = (var_3 * +1.0944E-4f - var_4 * +0.0f);
float tmp_1 = +1.9199E-42f;
float tmp_2 = var_5 / var_6 + +1.8498E34f;
comp = tmp_2 + tmp_1 + atanf(var_7 - (var_8 - cosf(var_9 * (-1.9116E-42f / (-1.1346E34f / var_10 * (-0.0f + -1.3239E-36f))))));
for (int i=0; i < var_2; ++i) {
  comp = (-1.7333E36f * (+1.1525E-35f - (var_11 - (var_12 * +1.5344E35f))));
comp = (var_13 * -1.8627E-43f * +1.7854E-27f - (var_14 / var_15 * var_16));
comp += logf((var_17 + (+0.0f * -1.5729E21f)));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
