#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10) {
float tmp_1 = +1.3526E-43f;
comp = tmp_1 - (-1.6277E-35f * +0.0f / var_1);
if (comp >= -1.4369E-43f / var_2 * +0.0f - -1.1795E-27f - (var_3 + var_4)) {
  comp += (-1.3787E7f + var_5);
}
if (comp > (+1.7559E3f / var_6 * var_7)) {
  comp += sinhf((var_8 + -0.0f * -1.1070E-30f * var_9 * var_10));
comp = (+1.8618E34f / (+0.0f - -1.1786E-37f + +1.3612E35f * +1.0141E35f + -1.5017E19f));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
