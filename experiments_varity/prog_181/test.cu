#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30) {
if (comp <= coshf(atanf(+0.0f + var_1))) {
  comp = var_3 / log10f((var_4 / sinhf(-1.6616E-36f - (var_5 + var_6 - -1.6840E-42f / +1.3050E6f))));
for (int i=0; i < var_2; ++i) {
  comp += (var_7 + (+1.5589E-35f + var_8));
comp = (var_9 - var_10);
}
if (comp < coshf((+1.5870E-36f + var_11 / var_12))) {
  comp += atan2f((var_13 / fabsf((+0.0f - sinhf(sinhf(var_14 - var_15 + var_16 - var_17))))), +1.5389E-25f / (var_18 + +1.8536E-35f - var_19 - var_20 / +1.7585E-36f));
comp = fmodf(var_21 * coshf((var_22 / var_23)), (var_24 * (var_25 + expf(var_26 * (+1.9017E-37f + var_27)))));
comp = sqrtf((var_28 * -1.2157E-43f - (var_29 + var_30)));
comp += (+1.0135E23f + atanf(-1.2016E-12f - +1.2992E36f));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31);
  hipDeviceSynchronize();

  return 0;
}
