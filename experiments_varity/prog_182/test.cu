#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float* var_9,float* var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19) {
comp = (-1.0688E34f / -1.3079E-44f);
float tmp_1 = +1.9766E20f;
comp = tmp_1 + var_4 - (-1.8696E-36f / (-1.3097E-42f * +0.0f - (+1.4178E-36f * var_5)));
for (int i=0; i < var_1; ++i) {
  comp = (var_6 * atan2f((+0.0f + var_7 - (-1.8006E-37f / +0.0f + var_8)), -1.9124E-35f));
}
for (int i=0; i < var_2; ++i) {
  var_9[i] = +0.0f * (var_11 + (var_12 / var_13));
var_10[i] = (-0.0f + var_14);
float tmp_2 = +1.3214E21f;
comp += tmp_2 * var_10[i] / var_9[i] + var_15 / var_16 + (var_17 / -1.2460E-35f / var_18 - +1.8015E-37f);
}
for (int i=0; i < var_3; ++i) {
  comp += (var_19 * (-0.0f / -1.6370E-20f - +1.0699E34f + +1.8704E-42f));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float* tmp_10 = initPointer( atof(argv[10]) );
  float* tmp_11 = initPointer( atof(argv[11]) );
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20);
  hipDeviceSynchronize();

  return 0;
}
