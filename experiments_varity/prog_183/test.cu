#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float* var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27) {
comp += var_2 - +1.9203E-11f + var_3;
float tmp_1 = (var_4 + (var_5 * (var_6 * var_7 / (-1.1925E-35f - var_8))));
comp += tmp_1 / (var_9 * sqrtf(ceilf(var_10 / (+1.9585E-4f - atanf((var_11 / var_12))))));
for (int i=0; i < var_1; ++i) {
  comp += (-1.8728E28f * (var_14 / -1.1331E-35f));
var_13[i] = -1.1449E36f;
comp += var_13[i] / (+1.6099E19f * (-1.1344E-22f / (+1.8134E-44f * (+0.0f + powf(sinhf((var_15 * (var_16 * atanf(+1.4159E-42f)))), (-1.5757E-19f - -1.4257E-13f))))));
}
if (comp >= (var_17 * var_18)) {
  float tmp_2 = (var_19 - var_20 - +1.2701E10f / +1.8868E-42f + (-1.5387E36f + var_21));
comp += tmp_2 - (-1.3896E-37f * floorf((var_22 / +1.5989E-44f)));
float tmp_3 = +0.0f;
comp += tmp_3 + (var_23 + acosf(-1.2172E35f / -1.2134E-41f - var_24 * var_25 - var_26 + var_27));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float* tmp_14 = initPointer( atof(argv[14]) );
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28);
  hipDeviceSynchronize();

  return 0;
}
