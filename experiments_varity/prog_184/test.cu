#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
for (int i=0; i < var_1; ++i) {
  if (comp >= sinf(-1.7216E-22f)) {
    comp = -1.0892E36f * +1.3154E34f;
float tmp_1 = var_3 * atan2f(fmodf(var_4 - var_5 * -0.0f - var_6 + var_7, cosf(var_8 + (+1.8449E-37f / (-1.0696E-37f / atanf(+1.3494E-36f + +0.0f))))), tanhf(-0.0f));
comp += tmp_1 * cosf(-1.7559E34f - var_9 / +1.1626E-36f / var_10);
for (int i=0; i < var_2; ++i) {
  comp += var_11 * (var_12 * +1.7503E-3f * (+1.4228E24f / (-1.4925E-37f * +1.0348E-42f)));
comp += (+1.2734E-41f * var_13);
comp += (var_14 + var_15);
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
