#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30,float var_31,float var_32,float var_33,float var_34,float var_35,float var_36,float var_37,float var_38) {
for (int i=0; i < var_1; ++i) {
  comp = (var_2 + fabsf((var_3 / +1.9136E15f / sqrtf(coshf((var_4 - asinf(atan2f((+0.0f + var_5 - var_6), -1.5388E35f * +1.2421E-44f * -1.6542E-37f + var_7 + (var_8 - -1.4934E-37f)))))))));
comp += logf(sinf((var_9 - -1.3067E19f)));
if (comp > log10f(-1.3278E-35f - var_10 * (-0.0f * coshf(var_11 * (-0.0f + var_12 - (var_13 + var_14 / -1.3503E-41f)))))) {
  comp += var_15 + var_16 * -1.4411E17f - +1.1367E24f - var_17;
float tmp_1 = (var_18 / (+1.3119E35f * +1.5619E-41f / var_19 - +0.0f));
comp = tmp_1 / (+1.3306E34f / var_20);
}
if (comp > atanf(+0.0f)) {
  comp += asinf(floorf(var_21 * -0.0f + +0.0f + floorf(-1.8509E34f)));
comp += -0.0f * (-1.0659E35f / var_22 + var_23 / (-1.3743E-28f / -1.2585E-42f));
comp += var_24 * var_25 * var_26 + var_27;
}
if (comp > -1.2813E-37f + +1.3921E-37f * (var_28 + floorf((var_29 / log10f((-0.0f / tanhf((-1.9777E-37f + var_30 + (var_31 + (var_32 - (+1.1346E36f + var_33))))))))))) {
  comp = logf((var_34 / ldexpf(atanf(var_35 * sinhf((+0.0f - sinf(var_36 / (var_37 + (-1.9344E-37f / var_38)))))), 2)));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);
  float tmp_33 = atof(argv[33]);
  float tmp_34 = atof(argv[34]);
  float tmp_35 = atof(argv[35]);
  float tmp_36 = atof(argv[36]);
  float tmp_37 = atof(argv[37]);
  float tmp_38 = atof(argv[38]);
  float tmp_39 = atof(argv[39]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32,tmp_33,tmp_34,tmp_35,tmp_36,tmp_37,tmp_38,tmp_39);
  hipDeviceSynchronize();

  return 0;
}
