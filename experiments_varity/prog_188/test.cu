#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float* var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28) {
if (comp >= +1.8379E-36f - +1.4853E-10f) {
  comp += asinf(-0.0f + var_3);
comp += -0.0f * var_4 / -1.3796E-8f / +1.4757E-20f;
comp = (+1.8330E-43f + -1.5787E4f + var_5);
comp = var_6 / sinf(var_7 / cosf(atanf(-1.7425E-37f * (var_8 + (+1.6915E-41f * (var_9 - +1.5913E-44f))))));
for (int i=0; i < var_1; ++i) {
  comp = powf((var_10 - +1.7858E-37f), var_11 + var_12);
comp += (var_13 - var_14 - sinf((-1.6028E35f + (+1.0608E-35f - atanf((-1.3608E-37f / -1.5117E28f + fabsf(-1.7286E-8f)))))));
}
for (int i=0; i < var_2; ++i) {
  var_15[i] = +1.6830E36f * (-1.4712E-42f - (var_16 - var_17));
comp = var_15[i] - tanhf((var_18 * var_19 - var_20));
comp += (-0.0f / atan2f((var_21 - -0.0f / expf(+1.3296E34f * +1.2648E36f)), +1.4354E21f - (var_22 - (-1.3582E-35f - -0.0f))));
}
if (comp < var_23 + var_24 / var_25) {
  float tmp_1 = +1.6915E-42f;
comp += tmp_1 * (var_26 - (var_27 / var_28 / +0.0f + +1.0396E-44f - +1.7050E36f));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float* tmp_16 = initPointer( atof(argv[16]) );
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29);
  hipDeviceSynchronize();

  return 0;
}
