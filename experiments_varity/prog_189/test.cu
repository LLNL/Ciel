#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float* var_10,float var_11,float var_12,float* var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
comp += +1.1797E35f / -1.8345E-18f;
if (comp >= +1.4698E17f + (var_3 / -1.8469E34f)) {
  float tmp_1 = (var_4 / var_5);
comp += tmp_1 * (+0.0f * (var_6 + -1.8055E-42f - -1.8605E-1f));
comp += var_7 / log10f((+1.9604E5f * -1.9015E-41f / (var_8 * (+1.6973E-42f + var_9))));
}
for (int i=0; i < var_1; ++i) {
  var_10[i] = -1.7731E36f;
comp = var_10[i] - var_11 + -1.3626E-44f / var_12;
comp = expf(+0.0f * (+1.2017E36f + +1.7938E36f));
}
for (int i=0; i < var_2; ++i) {
  var_13[i] = var_14 - var_15 - -1.4831E-37f / +1.6308E-36f;
comp += var_13[i] / -0.0f - (-0.0f + var_16);
comp = var_17 - (-1.4859E-41f + var_18 / tanhf(+1.5032E34f));
comp = +0.0f * (var_19 + var_20);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float* tmp_11 = initPointer( atof(argv[11]) );
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float* tmp_14 = initPointer( atof(argv[14]) );
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
