#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23) {
for (int i=0; i < var_1; ++i) {
  comp = +1.0779E36f + tanhf(+0.0f * -1.7349E34f);
float tmp_1 = (+1.2043E34f / (var_3 - var_4));
comp += tmp_1 * +1.1885E-42f / (+1.1410E3f * asinf((var_5 / expf((+1.8003E-41f * +1.2444E2f)))));
comp = (var_6 / (-1.1314E-35f * var_7));
for (int i=0; i < var_2; ++i) {
  comp = var_8 / var_9;
comp += acosf(-1.6283E-36f);
}
if (comp > +1.9307E-37f * +0.0f * (-1.1173E-35f + var_10 * floorf(var_11 - +1.4816E-41f))) {
  float tmp_2 = +1.9974E35f - (var_12 + var_13 - (+0.0f * +1.3769E-4f));
comp = tmp_2 + +1.0508E-41f * +1.6762E36f;
}
if (comp >= (-0.0f - log10f(+0.0f))) {
  comp = (var_14 - (var_15 - (var_16 / -0.0f)));
comp += atan2f(-0.0f - var_17, cosf(var_18 / sinhf(+1.4442E36f)));
comp += powf((var_19 - (-1.2191E34f / var_20 * (+0.0f - var_21 - -1.7855E-37f))), var_22 / +1.1783E-36f / var_23);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24);
  hipDeviceSynchronize();

  return 0;
}
