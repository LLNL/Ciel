#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
float tmp_1 = -0.0f;
comp += tmp_1 - var_2 / var_3;
if (comp > log10f((-1.9656E-44f + (-0.0f - var_4)))) {
  comp += +1.2566E35f * floorf((+1.3411E-42f / -1.4854E6f * (+0.0f * (-1.5597E-37f / var_5))));
}
if (comp >= asinf(-0.0f * (+0.0f - (var_6 * +1.8632E-36f)))) {
  float tmp_2 = -1.0954E-42f;
comp = tmp_2 - sqrtf(+1.3396E16f - -1.0856E-26f + var_7 + (var_8 * +1.3301E-41f));
float tmp_3 = (-1.0335E-36f * -1.9535E35f - var_9);
comp += tmp_3 - (var_10 - +1.0314E-36f);
}
for (int i=0; i < var_1; ++i) {
  comp = (-1.9095E36f - +1.4898E34f - -1.2372E-42f + (var_11 - -1.0286E-20f));
comp += var_12 * fabsf(+1.4056E-36f);
comp = var_13 + var_14 - var_15 / powf(-1.2869E35f, +1.6324E-3f * powf(-1.7454E34f, -1.3934E34f * (-1.6833E-35f - (var_16 + +1.1709E15f * (var_17 + -1.0814E-43f)))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
