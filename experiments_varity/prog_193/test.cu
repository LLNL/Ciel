#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4) {
float tmp_1 = (var_2 - (+1.8500E-44f - +0.0f * -0.0f));
comp = tmp_1 - (-1.5346E-37f + -1.5036E27f + (var_3 / -1.0741E-42f));
for (int i=0; i < var_1; ++i) {
  float tmp_2 = +0.0f / -1.7592E35f;
comp += tmp_2 - -1.3442E-36f - (+1.5408E36f * (var_4 * -1.9584E36f));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5);
  hipDeviceSynchronize();

  return 0;
}
