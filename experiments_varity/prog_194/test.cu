#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float* var_13,float* var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23) {
comp = +1.8369E13f + +1.5231E19f - -1.1416E-30f;
float tmp_1 = (-1.6147E24f + var_3);
comp = tmp_1 * +1.6122E-25f + +1.0583E-44f;
comp += (var_4 / -1.2922E-30f + var_5 + var_6);
if (comp == +1.3898E-42f / var_7 + (-1.1999E-36f + (var_8 / var_9 + var_10))) {
  float tmp_2 = sinf(+1.0847E-35f / (-1.8160E-43f * var_11));
comp = tmp_2 / fabsf((var_12 * (-1.5792E19f / +0.0f)));
}
for (int i=0; i < var_1; ++i) {
  comp = (var_15 - var_16);
var_13[i] = var_17 / (var_18 - -1.9131E-22f + (var_19 + +1.7892E34f));
var_14[i] = +1.0286E-37f;
comp += var_14[i] * var_13[i] * (var_20 * (var_21 / +1.8649E-23f));
}
for (int i=0; i < var_2; ++i) {
  comp = ldexpf(var_22 + (var_23 * (-0.0f - (+1.3815E-42f - +1.8187E-35f))), 2);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float* tmp_14 = initPointer( atof(argv[14]) );
  float* tmp_15 = initPointer( atof(argv[15]) );
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24);
  hipDeviceSynchronize();

  return 0;
}
