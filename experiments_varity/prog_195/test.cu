#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24) {
float tmp_1 = (-1.6903E35f / (var_3 + -1.2512E-44f * var_4));
comp = tmp_1 * -0.0f - powf(-1.0657E-42f + (var_5 - fabsf(acosf(+1.1820E-4f))), +1.6837E35f);
comp += powf((var_6 * (var_7 * -1.4600E34f + (var_8 / var_9))), -1.5394E-42f);
if (comp <= ceilf(-0.0f)) {
  float tmp_2 = -1.1465E-44f;
float tmp_3 = var_10 / +1.6744E-35f;
comp = tmp_3 - tmp_2 * var_11 / var_12;
comp += (var_13 + (var_14 / (+1.7517E35f * -1.2382E35f)));
}
for (int i=0; i < var_1; ++i) {
  float tmp_4 = var_15 * fmodf((+0.0f / +1.8851E-35f), (var_16 - var_17 - (-1.6458E35f * -0.0f)));
comp = tmp_4 * fabsf((+1.3641E-18f * (+1.8446E-27f - var_18 * atan2f((var_19 / tanhf(floorf(var_20 * var_21))), fmodf(+1.5539E-43f, (var_22 - -1.4886E-36f + var_23))))));
}
for (int i=0; i < var_2; ++i) {
  comp = ldexpf((-1.0370E-37f + -1.9211E26f / var_24), 2);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25);
  hipDeviceSynchronize();

  return 0;
}
