#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
if (comp < var_2 / var_3 - var_4 + fabsf(+1.6968E21f)) {
  for (int i=0; i < var_1; ++i) {
    float tmp_1 = +0.0f - (var_6 / var_7 * -1.4756E-43f - powf(fmodf(-1.6327E-43f, (-1.1981E-42f / (-1.7529E-37f + +0.0f))), asinf(+1.7393E20f)));
comp += tmp_1 / -1.3600E35f + +1.6405E-35f + cosf((var_8 / (+1.7154E-43f - (-1.4625E9f - var_9 / var_10))));
if (comp >= (+1.5578E-43f - logf(log10f(-1.1290E23f)))) {
  comp = var_11 * var_12 - (+1.5927E8f + +0.0f);
}
for (int i=0; i < var_5; ++i) {
  comp = +1.5094E36f + (var_13 + ldexpf(tanhf(+1.4825E2f + var_14 * (var_15 * fmodf(powf(tanhf(fmodf((-1.9049E-35f / var_16), var_17 - var_18 + +1.5543E-42f)), (+1.3079E-36f * (-1.3300E36f / floorf(-1.6612E35f)))), atan2f(ldexpf(+1.1272E-41f, 2), logf(var_19 * -1.9125E-44f - var_20))))), 2));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
