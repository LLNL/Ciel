#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21) {
comp = -0.0f / +0.0f - -0.0f / (var_2 + var_3);
float tmp_1 = log10f((-1.9931E36f / (+1.2540E-37f - -1.7856E28f)));
comp += tmp_1 - logf((var_4 / var_5 / var_6));
for (int i=0; i < var_1; ++i) {
  float tmp_2 = fmodf(var_7 / -1.7238E-36f, (var_8 / -1.7777E-15f));
comp += tmp_2 + -1.8309E-36f - (var_9 + (-1.7789E-44f - var_10));
comp = (-1.5996E-9f / (var_11 + (-1.3240E36f * var_12 - var_13 - var_14)));
comp += +1.3123E-35f - var_15 + var_16;
}
if (comp == powf(-1.4214E-28f / var_17 - (-1.7691E-14f / var_18 * +1.7414E-42f), (var_19 - +1.8289E36f - +1.6617E-36f))) {
  comp += atan2f(-1.0248E35f, -1.0499E-11f - var_20 / var_21 * -1.0678E-37f * +1.5149E-44f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22);
  hipDeviceSynchronize();

  return 0;
}
