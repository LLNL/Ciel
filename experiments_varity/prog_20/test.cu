#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float* var_11,float* var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24) {
for (int i=0; i < var_1; ++i) {
  comp = var_3 * floorf(+1.5144E35f);
float tmp_1 = -0.0f + (var_4 + fmodf((var_5 * (+1.1032E-36f - (var_6 * -0.0f + +0.0f))), -1.8364E35f));
comp += tmp_1 * (-1.7594E-25f + (var_7 - var_8 - var_9));
comp += tanhf(floorf(+1.6852E-43f + var_10 * -1.1582E-36f));
for (int i=0; i < var_2; ++i) {
  var_11[i] = -1.6470E-22f;
var_12[i] = +1.0940E36f;
comp += var_12[i] / var_11[i] + (var_13 / (var_14 + var_15));
}
if (comp >= -1.4099E35f + (-1.8257E34f + -1.0592E27f)) {
  comp = var_16 - var_17 / var_18 - -1.7750E-37f / -1.9396E-19f + var_19;
comp += var_20 - -0.0f;
comp += (+1.9299E-37f + (+1.9056E12f / var_21));
comp += var_22 - -1.5014E-36f * log10f(+1.0657E-43f);
}
if (comp > -1.2880E1f / +1.7427E-44f - +1.5180E7f) {
  comp = var_23 - (var_24 / -1.6423E11f);
comp = (+1.4207E-36f - +1.3393E34f + -1.1701E-43f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float* tmp_12 = initPointer( atof(argv[12]) );
  float* tmp_13 = initPointer( atof(argv[13]) );
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25);
  hipDeviceSynchronize();

  return 0;
}
