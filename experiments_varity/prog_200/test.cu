#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float* var_22,float var_23,float var_24,float var_25,float var_26) {
for (int i=0; i < var_1; ++i) {
  comp += -1.4565E-35f + (-1.8018E-42f * var_4 - (var_5 / (+1.3201E35f * var_6)));
comp = sqrtf(-1.4949E-37f * (var_7 * (+0.0f / (var_8 + var_9))));
if (comp == var_10 / floorf((var_11 - var_12 * -1.2540E-27f + (var_13 * var_14 * var_15)))) {
  comp += (-1.9437E-37f / fmodf(+1.3188E-41f + (var_16 + (+0.0f - (var_17 * var_18))), var_19 + -1.6687E-43f / -1.3414E-15f / (-0.0f + -1.0465E-42f)));
comp += (+0.0f + (var_20 / -1.0655E34f + +0.0f / +1.8548E24f));
comp = var_21 + tanhf(-1.7508E-44f);
}
for (int i=0; i < var_2; ++i) {
  float tmp_1 = +1.9236E-42f;
float tmp_2 = (var_23 + (+1.8576E-43f + (+1.5835E-43f + -1.3130E34f)));
var_22[i] = -1.7754E-41f;
comp += var_22[i] * tmp_2 - tmp_1 + (+0.0f / (-1.9973E-36f * (+0.0f * var_24 + -1.9126E-43f)));
}
for (int i=0; i < var_3; ++i) {
  comp = var_25 * powf(+1.4865E-43f, +0.0f + (var_26 + -0.0f));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float* tmp_23 = initPointer( atof(argv[23]) );
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27);
  hipDeviceSynchronize();

  return 0;
}
