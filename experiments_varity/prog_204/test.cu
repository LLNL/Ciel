#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10) {
for (int i=0; i < var_1; ++i) {
  if (comp == var_2 / (var_3 * log10f(-0.0f + var_4))) {
    comp += +1.4807E-7f - var_6 + (var_7 / asinf((+1.7648E-43f - -1.9257E6f)));
for (int i=0; i < var_5; ++i) {
  comp += sinhf(+1.2551E21f);
float tmp_1 = -1.7369E10f - (var_8 / -1.2251E34f);
comp += tmp_1 - (var_9 * +1.4292E-41f - var_10);
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
