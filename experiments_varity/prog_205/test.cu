#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float* var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28) {
if (comp > (-1.7917E-36f / +1.6161E-35f)) {
  for (int i=0; i < var_1; ++i) {
    for (int i=0; i < var_2; ++i) {
      var_3[i] = +1.3808E-43f * (+1.4054E-35f - var_4 - var_5);
comp += var_3[i] - (var_6 - log10f(atan2f(cosf(var_7 - var_8 - +1.8063E35f / +0.0f + var_9), atan2f((var_10 * var_11 * var_12 * fmodf((-1.1895E-35f * -1.2946E-21f + var_13), -0.0f)), +1.0473E0f / +1.1172E36f))));
comp = asinf(-1.8887E19f);
comp = (var_14 / (-1.4624E-37f * -1.3550E25f * +0.0f));
if (comp < -1.3526E-42f * atan2f(-1.2584E-41f + fmodf(var_15 / -1.5098E-37f + (-1.7195E5f * (var_16 / var_17)), (var_18 + +1.4954E28f)), logf(-0.0f))) {
  comp = -1.0378E-35f * -1.9795E35f / -1.3511E-41f - var_19 * -1.1929E16f;
}
if (comp <= (var_20 * (-1.6826E-35f - var_21))) {
  comp = (var_22 / (+0.0f * (var_23 - -1.5127E-42f * -1.6680E15f * -1.2287E-41f)));
}
if (comp > (var_24 - ceilf(-1.8055E-37f * (var_25 + fabsf((+1.5670E-37f / (var_26 * +0.0f - var_27 + (-1.3188E-16f * var_28)))))))) {
  float tmp_1 = +0.0f * +1.8601E20f - -1.4860E35f / -1.4571E36f;
comp = tmp_1 + cosf(sinf(+1.3136E35f));
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float* tmp_4 = initPointer( atof(argv[4]) );
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29);
  hipDeviceSynchronize();

  return 0;
}
