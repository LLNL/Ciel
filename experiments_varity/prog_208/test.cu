#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
comp += +1.1887E-43f + var_3;
for (int i=0; i < var_1; ++i) {
  comp += (-1.8394E-8f / -0.0f + var_4);
float tmp_1 = -0.0f;
comp = tmp_1 / (var_5 + (+1.9721E36f - (var_6 + (var_7 / -1.8529E-14f))));
comp = fabsf((var_8 * +1.4873E-41f));
}
if (comp == atanf(expf((-1.6237E-17f * (+1.7814E36f / tanhf(var_9 + -1.4810E34f)))))) {
  comp += (var_10 - +1.2613E-37f);
comp += atan2f(sinf((+1.3937E4f - (+0.0f / (+1.8910E-37f + +1.5442E12f * (-1.4273E-43f - +1.2741E34f))))), +1.9455E-36f);
comp = (-1.4764E35f / (var_11 + var_12));
}
for (int i=0; i < var_2; ++i) {
  float tmp_2 = +0.0f;
comp += tmp_2 / (+1.1586E35f / (var_13 - (var_14 + +1.4786E-6f + logf((-1.6967E-44f - +1.0911E10f * var_15 / var_16)))));
comp += (-1.8385E36f + (+1.9118E-44f * var_17));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
