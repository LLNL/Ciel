#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float* var_19,float* var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30,float var_31,float var_32,float var_33,float var_34,float var_35,float var_36,float var_37,float var_38,float var_39,float var_40,float var_41,float var_42,float var_43,float var_44,float var_45,float var_46,float var_47) {
float tmp_1 = (+1.7017E-35f - (var_2 - sinf(var_3 / var_4)));
comp += tmp_1 * var_5 / (var_6 + var_7 - floorf(cosf(var_8 / -1.8038E-20f)));
comp += var_9 / (var_10 * -1.7062E35f + var_11 / var_12 * var_13);
comp = var_14 + (-1.0880E-36f * var_15 - (var_16 / var_17 * var_18));
for (int i=0; i < var_1; ++i) {
  var_19[i] = +1.1136E-44f;
comp += var_19[i] + fmodf((+1.3711E35f - tanhf(-1.7783E-30f * var_21)), -1.8809E35f * +1.5638E-43f / (+1.7277E27f + (-1.7914E-35f / -1.1059E-43f)));
var_20[i] = (+1.5688E23f - var_22 / logf(powf(fmodf(+0.0f, (-1.0794E-35f * logf((var_23 * (var_24 / ldexpf(+1.1459E-3f / var_25 - +0.0f, 2)))))), var_26 / (-1.0921E34f * +1.3476E-37f + ceilf((+1.1791E34f * atan2f(cosf(var_27 + var_28), +1.4517E27f)))))));
comp = var_20[i] * (+1.4247E-22f * atanf((-0.0f * +1.7139E-43f)));
}
if (comp == atanf(-1.6008E1f)) {
  float tmp_2 = (+1.1075E35f - atan2f(cosf((+0.0f / asinf(-1.3512E23f + var_29))), var_30 - log10f((+1.2994E-36f * -1.8890E-12f))));
comp += tmp_2 + var_31 / (var_32 - +1.1577E-41f);
comp = var_33 / +1.8852E35f - (var_34 / var_35 * +1.9336E-43f - var_36);
}
if (comp >= var_37 + (var_38 + (+1.4578E-41f / (-1.0898E-37f * var_39)))) {
  float tmp_3 = +1.4485E3f;
float tmp_4 = ldexpf(expf(var_40 - -1.3678E-44f + var_41 + var_42), 2);
float tmp_5 = (+0.0f * +1.6230E-44f);
comp += tmp_5 / tmp_4 + tmp_3 * -1.0934E35f + var_43 * (+1.6037E34f * expf(-1.5607E-37f * var_44 + var_45 * var_46 + var_47));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float* tmp_20 = initPointer( atof(argv[20]) );
  float* tmp_21 = initPointer( atof(argv[21]) );
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);
  float tmp_33 = atof(argv[33]);
  float tmp_34 = atof(argv[34]);
  float tmp_35 = atof(argv[35]);
  float tmp_36 = atof(argv[36]);
  float tmp_37 = atof(argv[37]);
  float tmp_38 = atof(argv[38]);
  float tmp_39 = atof(argv[39]);
  float tmp_40 = atof(argv[40]);
  float tmp_41 = atof(argv[41]);
  float tmp_42 = atof(argv[42]);
  float tmp_43 = atof(argv[43]);
  float tmp_44 = atof(argv[44]);
  float tmp_45 = atof(argv[45]);
  float tmp_46 = atof(argv[46]);
  float tmp_47 = atof(argv[47]);
  float tmp_48 = atof(argv[48]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32,tmp_33,tmp_34,tmp_35,tmp_36,tmp_37,tmp_38,tmp_39,tmp_40,tmp_41,tmp_42,tmp_43,tmp_44,tmp_45,tmp_46,tmp_47,tmp_48);
  hipDeviceSynchronize();

  return 0;
}
