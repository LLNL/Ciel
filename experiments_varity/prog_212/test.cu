#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,int var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float* var_26,float* var_27,float var_28,float var_29,float var_30,float var_31,float var_32,float var_33) {
if (comp >= log10f((var_1 - var_2))) {
  if (comp < -1.2261E19f + coshf(var_3 * +0.0f * (var_4 * (-1.4165E-25f * var_5)))) {
    comp += var_7 - (var_8 + var_9 - (-0.0f * -1.2062E23f + +1.4310E-41f));
comp += var_10 + var_11 + (-1.0789E-35f + var_12 - +1.3540E35f - +1.2578E-36f);
if (comp > -1.3189E36f + +1.2720E36f - (var_13 * (-1.6585E36f / var_14 - var_15))) {
  comp += var_16 - (var_17 * (var_18 * acosf((+0.0f / var_19 - -1.2988E-22f / var_20))));
}
if (comp <= (var_21 * -1.9488E-29f * -1.2776E36f)) {
  comp = -0.0f * sinf(+1.4719E26f);
float tmp_1 = logf(+1.3238E-44f);
comp = tmp_1 + +1.3141E35f + (var_22 / (var_23 - -0.0f + (var_24 * var_25)));
}
for (int i=0; i < var_6; ++i) {
  var_26[i] = +0.0f;
float tmp_2 = +1.1754E-17f;
var_27[i] = (var_28 - log10f(var_29 - (-1.3263E-36f - (var_30 / +1.1926E-42f))));
comp = var_27[i] / tmp_2 * var_26[i] / (var_31 - (+1.1612E-36f / (var_32 * var_33 + (-1.0845E-36f * -1.6945E-44f))));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  int tmp_7 = atoi(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float* tmp_27 = initPointer( atof(argv[27]) );
  float* tmp_28 = initPointer( atof(argv[28]) );
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);
  float tmp_33 = atof(argv[33]);
  float tmp_34 = atof(argv[34]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32,tmp_33,tmp_34);
  hipDeviceSynchronize();

  return 0;
}
