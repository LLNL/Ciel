#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30,float var_31,float var_32,float var_33) {
if (comp <= fmodf((var_1 * -1.8254E-35f / -1.6689E35f / (var_2 * (var_3 / var_4))), -1.7343E21f)) {
  float tmp_1 = var_6 * -1.6855E-36f;
float tmp_2 = +0.0f;
comp = tmp_2 * tmp_1 * var_7 - logf(var_8 - +1.1338E-23f);
comp = var_9 + (+0.0f * var_10 / var_11 * -1.3318E34f);
if (comp <= var_12 / var_13 - var_14 * acosf(var_15 + +1.3878E-42f * var_16 * sinf(fmodf(var_17 - var_18, (+1.3409E34f - var_19 / (-0.0f - powf(var_20 / (+1.5133E10f / var_21 + (var_22 + var_23)), -1.0736E34f))))))) {
  comp = var_24 + floorf(acosf((var_25 / var_26)));
float tmp_3 = -1.1663E36f;
float tmp_4 = -1.2158E36f;
comp += tmp_4 + tmp_3 - sqrtf(var_27 - +1.5496E-41f);
}
for (int i=0; i < var_5; ++i) {
  float tmp_5 = (var_28 - (-1.7661E29f - var_29));
comp += tmp_5 - var_30 + -1.1375E-20f / +1.5173E-19f;
comp += (+1.7534E35f / var_31 / var_32 * +1.1428E36f - -1.8270E36f);
comp += floorf((var_33 * -1.0072E-37f));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);
  float tmp_33 = atof(argv[33]);
  float tmp_34 = atof(argv[34]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32,tmp_33,tmp_34);
  hipDeviceSynchronize();

  return 0;
}
