#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11) {
for (int i=0; i < var_1; ++i) {
  comp = coshf(var_2 - (-1.7378E23f + (var_3 / (+1.8335E-42f + var_4))));
comp = (+1.5395E-41f / sinf((var_5 - floorf(-1.4379E-35f - (+1.5941E-43f - -1.0407E-24f - -1.5079E-36f - -0.0f)))));
float tmp_1 = +0.0f;
comp = tmp_1 * (-1.4513E-44f / var_6);
if (comp == var_7 - var_8 - +1.9090E-37f / +1.6882E36f) {
  comp = var_9 - var_10;
comp += var_11 * (-1.3228E-35f / (+1.1695E20f * (-1.5240E35f * (+1.6842E-18f / +1.3278E-44f))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12);
  hipDeviceSynchronize();

  return 0;
}
