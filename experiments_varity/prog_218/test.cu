#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25) {
float tmp_1 = -1.7120E-35f;
comp = tmp_1 / atanf((-1.3472E-44f * var_2));
float tmp_2 = -1.4757E36f;
comp += tmp_2 / (+1.4849E35f * (var_3 - (+0.0f / var_4 * -1.0348E-22f)));
if (comp > asinf(-1.6645E6f)) {
  comp += (-0.0f / var_5);
comp = -1.0268E34f / ceilf((var_6 / (+1.2930E-16f * var_7)));
}
if (comp > (-1.8978E35f / var_8 + var_9)) {
  comp += +1.9970E16f + +1.7339E-3f - sinf(-1.7323E-28f - var_10);
float tmp_3 = +1.5270E-37f - (+0.0f + -1.9261E-35f - (var_11 / var_12));
float tmp_4 = (var_13 - var_14 + powf((var_15 * -1.5619E9f / (var_16 * (var_17 * var_18))), acosf((var_19 / (var_20 / (var_21 + +1.7753E-17f))))));
comp += tmp_4 / tmp_3 + fabsf((var_22 - +1.6365E-42f * -1.1466E35f));
}
for (int i=0; i < var_1; ++i) {
  float tmp_5 = +1.7317E-5f;
float tmp_6 = (+1.0131E35f - (-1.5156E-35f * -0.0f));
float tmp_7 = (+1.6026E-36f - var_23);
comp += tmp_7 * tmp_6 + tmp_5 - -1.3325E24f / atan2f((-0.0f / log10f((var_24 + (var_25 * +1.4263E-36f)))), +0.0f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26);
  hipDeviceSynchronize();

  return 0;
}
