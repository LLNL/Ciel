#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float* var_12,float var_13) {
float tmp_1 = +1.1132E-37f;
float tmp_2 = -1.3383E-28f * var_4 / +1.0794E-36f;
comp += tmp_2 - tmp_1 / (var_5 + +1.4975E-44f - var_6);
comp = (-1.1945E-17f * var_7);
for (int i=0; i < var_1; ++i) {
  comp += var_8 * -1.5590E29f + (-1.8040E36f / var_9);
}
for (int i=0; i < var_2; ++i) {
  comp += (var_10 + (var_11 * -0.0f));
}
for (int i=0; i < var_3; ++i) {
  comp += (-1.4003E35f / (var_13 / -0.0f));
var_12[i] = -1.1611E-43f;
comp += var_12[i] / -0.0f - (+0.0f - -1.5122E-43f * -1.2162E34f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float* tmp_13 = initPointer( atof(argv[13]) );
  float tmp_14 = atof(argv[14]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14);
  hipDeviceSynchronize();

  return 0;
}
