#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11) {
comp += (var_3 + -1.4978E-41f);
for (int i=0; i < var_1; ++i) {
  comp += (var_4 / log10f((-0.0f - (var_5 / (var_6 / powf(+0.0f, -1.6087E8f * fabsf(+1.8141E-41f * (-0.0f + +0.0f + var_7))))))));
}
for (int i=0; i < var_2; ++i) {
  float tmp_1 = (-1.5355E-43f * asinf(-0.0f));
comp += tmp_1 + (+0.0f / (+0.0f + (var_8 - var_9 / (+1.7480E-42f * +0.0f))));
comp += coshf(-1.9356E-36f / var_10 + -1.6899E27f);
comp += (var_11 * -1.9931E-9f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12);
  hipDeviceSynchronize();

  return 0;
}
