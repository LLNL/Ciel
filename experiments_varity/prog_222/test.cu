#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9) {
if (comp < (var_1 / floorf(atanf(-1.7987E-18f)))) {
  float tmp_1 = -1.3824E-44f;
comp = tmp_1 - (var_2 / -1.0897E-42f / -1.3540E-43f / (+1.0519E34f * (+1.9190E35f - var_3)));
comp += (var_4 + expf(+1.8888E-43f * -1.4031E-41f / acosf(cosf(-1.4017E25f))));
if (comp >= (var_5 + (-1.9268E-44f / -1.3701E-37f * (var_6 * (var_7 + +1.6556E-36f))))) {
  comp = tanhf(var_8 * -1.2787E-12f + +1.9470E-36f - +1.1146E-42f / -1.5902E-42f * var_9);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10);
  hipDeviceSynchronize();

  return 0;
}
