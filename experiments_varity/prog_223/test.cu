#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17) {
comp = -0.0f - atanf(floorf((var_1 + +1.0575E-41f)));
comp = (-0.0f - -1.6329E20f + (+1.4273E36f * var_2));
comp = var_3 * var_4 / +1.8775E-44f;
comp = fabsf(var_5 / log10f(var_6 + +1.8933E4f / (var_7 - +1.2792E34f)));
if (comp < expf(acosf((-0.0f - var_8 + (var_9 / (-1.2306E-44f - var_10)))))) {
  comp += var_11 - (var_12 + +1.5643E-19f + expf(atan2f(var_13 + var_14 - var_15, +1.7018E6f + asinf(+1.3032E-43f))));
comp += var_16 * var_17;
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
