#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float* var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float* var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
if (comp <= (var_3 - +0.0f)) {
  for (int i=0; i < var_1; ++i) {
    for (int i=0; i < var_2; ++i) {
      var_4[i] = +1.7374E-42f;
comp += var_4[i] / var_6 + -0.0f;
if (comp <= (var_7 * (+1.5845E-35f - floorf(fmodf(-1.1767E35f - +1.5610E-13f / -1.7831E-42f, (var_8 - var_9 / +1.1565E-42f)))))) {
  float tmp_1 = -1.0796E-42f;
comp = tmp_1 - floorf(-1.1935E-43f);
float tmp_2 = -1.5271E34f;
comp += tmp_2 * (-1.3054E36f - var_10 / ceilf((+1.0139E-27f - +1.4960E-36f)));
}
for (int i=0; i < var_5; ++i) {
  comp += expf((-1.6751E-18f + (-1.0289E36f - (var_12 + (-1.4354E36f * (+1.0830E36f * +1.8676E27f))))));
var_11[i] = var_13 - var_14 + (var_15 + var_16);
float tmp_3 = +1.6604E-35f;
comp = tmp_3 * var_11[i] + +0.0f - +1.0079E-24f;
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float* tmp_5 = initPointer( atof(argv[5]) );
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float* tmp_12 = initPointer( atof(argv[12]) );
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
