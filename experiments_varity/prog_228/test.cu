#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float* var_8,float* var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30,float var_31,float var_32) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    comp = var_5 * var_6 / (+1.4955E35f + var_7);
for (int i=0; i < var_3; ++i) {
  comp = (+1.9220E34f - sqrtf(+1.6696E-5f));
var_8[i] = -1.2810E-44f;
var_9[i] = (var_10 / +1.9766E-35f);
comp += var_9[i] - var_8[i] * (+0.0f - (var_11 * +1.5211E-44f));
}
if (comp < (var_12 / (var_13 + var_14 / (var_15 - (+0.0f + var_16))))) {
  comp = (-1.9038E-41f - acosf(var_17 / -0.0f / -0.0f - (+0.0f - (-0.0f * var_18))));
comp += (+1.8471E-42f / +1.0305E35f + -1.8905E35f);
comp = (var_19 / var_20 / -1.2833E-35f / var_21 - +1.5192E-42f);
comp += (+1.9975E-41f / var_22 / var_23 - (var_24 / var_25 + var_26));
}
for (int i=0; i < var_4; ++i) {
  float tmp_1 = (var_27 + +0.0f);
comp += tmp_1 - floorf(fmodf(coshf(+1.8288E-42f), (-0.0f + var_28 / sqrtf((var_29 * -1.5462E-11f / var_30 / var_31 * +1.9347E36f / var_32)))));
comp += +1.2492E-43f * +1.1390E7f;
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float* tmp_9 = initPointer( atof(argv[9]) );
  float* tmp_10 = initPointer( atof(argv[10]) );
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);
  float tmp_33 = atof(argv[33]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32,tmp_33);
  hipDeviceSynchronize();

  return 0;
}
