#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float* var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
for (int i=0; i < var_1; ++i) {
  comp = -1.8594E-36f + var_5;
var_2[i] = sqrtf(+1.7597E35f);
comp += var_2[i] * var_6 + +0.0f - var_7 * var_8;
for (int i=0; i < var_3; ++i) {
  float tmp_1 = tanhf(-1.1087E-35f * +1.7744E5f);
comp += tmp_1 * atan2f((var_9 - fabsf(-1.7188E-44f * var_10 * var_11 / +1.6139E-25f)), +1.6777E-42f * (var_12 + +1.8675E19f));
comp += (+1.8075E-37f / -1.1232E35f - sinhf(+1.9562E-35f + +1.5666E36f - asinf((-1.5036E34f + -1.6966E19f * -0.0f))));
}
for (int i=0; i < var_4; ++i) {
  comp = +1.1746E-35f - var_13 / var_14 / var_15;
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float* tmp_3 = initPointer( atof(argv[3]) );
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
