#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,int var_3,float var_4,float var_5,float var_6,float var_7) {
if (comp > cosf(+0.0f / (-1.2242E34f * (-1.0814E-37f * sqrtf(+1.0582E-36f + (+1.5143E35f / (var_1 + var_2))))))) {
  float tmp_1 = +1.7283E-37f;
comp += tmp_1 / var_4 + (-1.4541E36f - var_5);
for (int i=0; i < var_3; ++i) {
  float tmp_2 = +1.2085E-35f;
float tmp_3 = -1.5980E36f;
comp += tmp_3 * tmp_2 + (+0.0f / var_6 / (-1.2555E-43f - (+1.7779E35f * (var_7 * -1.7309E-35f))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8);
  hipDeviceSynchronize();

  return 0;
}
