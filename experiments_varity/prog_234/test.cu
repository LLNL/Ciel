#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,int var_4,int var_5,int var_6,float var_7,float var_8,float* var_9,float* var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    for (int i=0; i < var_3; ++i) {
      for (int i=0; i < var_4; ++i) {
        float tmp_1 = -0.0f;
comp += tmp_1 + +1.7959E5f - -0.0f;
comp += -1.4499E34f + var_7 - coshf(+1.2568E-11f * (+1.5309E-41f - var_8));
for (int i=0; i < var_5; ++i) {
  var_9[i] = atan2f(-1.3134E-27f - (+1.0253E-36f / -0.0f * var_11 + var_12), var_13 - (+1.5368E35f - var_14 + -1.3070E-27f));
var_10[i] = var_15 + (-1.7340E28f / (-1.8622E35f * (-1.3255E36f - +1.6146E-37f)));
comp += var_10[i] - var_9[i] - floorf((-1.8950E8f * (+1.8632E34f / var_16)));
comp = -1.4070E-42f - (-1.1692E-42f * +1.1446E-35f * (+0.0f + (var_17 - -1.3687E34f)));
}
for (int i=0; i < var_6; ++i) {
  float tmp_2 = +1.2110E-3f;
comp = tmp_2 + -1.9469E-14f / sqrtf(expf(+1.7079E35f));
comp += +1.5617E-41f + var_18 / (var_19 / (+1.3114E-44f + var_20));
comp += (var_21 * -0.0f + (var_22 * (var_23 - +1.9176E-41f / +1.8814E35f)));
}
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  int tmp_6 = atoi(argv[6]);
  int tmp_7 = atoi(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float* tmp_10 = initPointer( atof(argv[10]) );
  float* tmp_11 = initPointer( atof(argv[11]) );
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24);
  hipDeviceSynchronize();

  return 0;
}
