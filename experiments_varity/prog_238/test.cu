#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
float tmp_1 = -1.5622E-12f;
comp = tmp_1 / var_1 - -1.4215E-36f - (var_2 / +1.9259E-35f);
if (comp >= (-1.4818E35f / +1.1805E-37f - var_3 - -0.0f)) {
  comp = -1.0140E34f / var_4;
float tmp_2 = -1.6795E35f;
comp = tmp_2 + asinf((+1.5713E-17f * +1.8919E-41f / fmodf((var_5 + atan2f(-0.0f, atan2f((var_6 * (var_7 * (var_8 * var_9))), var_10 + (var_11 - var_12 + var_13)))), (+1.1049E-44f - (var_14 * +0.0f + var_15)))));
comp += ldexpf(-1.3949E35f, 2);
}
if (comp <= +0.0f - (-1.7611E-1f / var_16)) {
  comp += tanhf(-1.4925E-35f / +1.5361E34f - -0.0f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
