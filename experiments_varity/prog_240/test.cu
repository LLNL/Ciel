#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float* var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
comp = +1.3931E-37f * +1.9547E-1f * (+1.0944E14f - logf(+1.8493E36f));
comp = (var_2 - -1.3692E7f + (var_3 - -1.9508E35f + -1.6681E36f / var_4));
float tmp_1 = (var_5 / +1.9037E34f);
comp = tmp_1 * -1.4246E-18f * var_6 - fabsf(-1.5294E-42f / (var_7 * (var_8 * (var_9 + +1.0429E24f - var_10))));
for (int i=0; i < var_1; ++i) {
  var_11[i] = tanhf(var_12 - +1.0808E36f * var_13 - -0.0f - var_14);
float tmp_2 = sinhf(var_15 - -1.0151E-35f);
comp = tmp_2 + var_11[i] * var_16 - (-0.0f / -1.3075E-44f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float* tmp_12 = initPointer( atof(argv[12]) );
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
