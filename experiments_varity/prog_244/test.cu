#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22) {
comp += var_2 - +1.6757E-35f - var_3;
float tmp_1 = (var_4 / (+1.4399E-42f / asinf(+0.0f)));
comp = tmp_1 + (+1.0538E-37f * (var_5 + cosf((var_6 - (-1.3881E35f + +0.0f * +1.1851E-43f)))));
comp = (var_7 - (+1.9216E9f / (var_8 / (var_9 / sinf(acosf((+1.2100E-43f * var_10 * var_11 * (+1.5812E-37f / -1.1924E-21f))))))));
if (comp == (var_12 + +1.7700E34f)) {
  comp += +0.0f / (+1.4232E22f + -1.8804E34f * (var_13 / (+1.3265E-35f - +1.1059E-35f)));
comp = (var_14 * +1.9852E-4f + -1.6860E3f * var_15);
}
for (int i=0; i < var_1; ++i) {
  comp = var_16 * var_17 / +1.8808E-37f * +1.6003E-10f;
comp = var_18 - (var_19 / -1.1040E-24f * ceilf((+1.2864E11f / var_20)));
comp = var_21 * (var_22 - -1.5429E34f / -0.0f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
