#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25) {
for (int i=0; i < var_1; ++i) {
  comp += var_3 + powf((var_4 * var_5), (-1.9858E-44f - acosf((+1.0257E-26f - powf((var_6 + asinf(fabsf((var_7 * ldexpf(+1.8798E35f, 2))))), +1.5273E34f * asinf(-1.8499E-37f * (-1.2484E-36f + (var_8 + var_9 * (+1.2591E-41f + var_10)))))))));
comp = +0.0f * (-1.6673E-35f - (var_11 + (-1.3009E-35f - (+1.1342E35f - +1.4982E-35f))));
comp += -1.3040E34f + +0.0f / var_12 - (-1.9604E21f + coshf(-1.3701E-42f / (-1.0553E36f / -0.0f + fmodf((+1.1148E34f + +1.2900E-27f * var_13), sinhf(+1.4706E35f * (+1.6873E-36f * (-1.2804E-43f - (+1.5872E-41f + -1.7754E34f))))))));
if (comp >= (var_14 + var_15)) {
  float tmp_1 = -0.0f;
float tmp_2 = -1.5222E-35f;
float tmp_3 = +1.9014E36f;
comp = tmp_3 - tmp_2 + tmp_1 * atanf(log10f((var_16 * -1.5509E-21f * var_17 / var_18)));
}
if (comp > +1.7469E-44f + (-1.6770E-44f + acosf(var_19 + +1.1268E36f))) {
  float tmp_4 = (-1.6707E-42f - (var_20 - atanf(var_21 - -1.9621E13f / var_22 - -0.0f)));
float tmp_5 = +1.1750E34f;
float tmp_6 = +1.5153E-43f;
comp = tmp_6 / tmp_5 + tmp_4 / +1.7235E12f - var_23 + -1.3209E-42f;
}
for (int i=0; i < var_2; ++i) {
  comp += (+1.7389E-41f + var_24 + var_25 * -1.1585E35f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26);
  hipDeviceSynchronize();

  return 0;
}
