#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,int var_4,float var_5,float var_6,float var_7,float var_8) {
if (comp < acosf((-1.6849E-41f + var_1 + (var_2 * (var_3 / +1.0045E27f))))) {
  float tmp_1 = +1.2779E-37f;
comp = tmp_1 / var_5 * var_6 * coshf(sinhf(+1.4815E5f));
for (int i=0; i < var_4; ++i) {
  comp = var_7 / (var_8 - +0.0f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9);
  hipDeviceSynchronize();

  return 0;
}
