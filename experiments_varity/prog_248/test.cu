#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24) {
comp += cosf((-1.7576E-30f + -1.9558E-37f * (-1.0797E-44f - var_2 * -1.5780E5f)));
for (int i=0; i < var_1; ++i) {
  comp = var_3 + (-1.5776E-42f / +0.0f + acosf(asinf(-1.4089E36f)));
comp += asinf(sinhf((-0.0f * +0.0f - (var_4 + -1.2916E-43f - var_5))));
comp = +1.2099E-35f + var_6 / (var_7 - -1.1441E-42f * sinhf(+1.2249E1f));
}
if (comp <= (-0.0f - var_8)) {
  comp = (var_9 / (var_10 + var_11 + var_12));
float tmp_1 = powf(var_13 - -1.3566E15f * var_14, ldexpf(+0.0f, 2));
comp += tmp_1 * fmodf((-1.2780E-29f + -1.3688E-35f / (var_15 + -1.3110E34f + -1.8922E-35f)), (-1.3659E-41f + var_16));
}
if (comp == atan2f((-1.3190E34f + -1.3193E9f + var_17 - (+1.3294E-35f / var_18 + +1.8826E-42f)), (-1.9068E34f + (+1.5156E36f + -1.7177E35f + var_19)))) {
  comp += +1.6347E-43f * var_20 - -1.7625E-37f + powf((-1.9462E-13f + var_21 - +1.8158E-42f / (-1.5474E34f + -1.0966E-42f + var_22)), var_23 / var_24 * +1.4382E-36f + sinhf((+0.0f + +1.4582E-44f)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25);
  hipDeviceSynchronize();

  return 0;
}
