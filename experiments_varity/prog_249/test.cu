#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12) {
for (int i=0; i < var_1; ++i) {
  comp = (var_2 + var_3 * (-1.2786E35f * (var_4 + -1.1209E34f - +1.7785E-36f)));
comp = (var_5 / +1.1265E35f / var_6 * var_7 - +1.9900E-25f - -1.7878E-29f);
comp += cosf((var_8 * -1.3333E36f / (-1.3913E-42f * -1.5517E-15f + var_9)));
if (comp > logf(+1.1179E-43f * (-1.1860E-41f / var_10 - cosf(tanhf((-1.4085E-42f + +1.4000E-44f)))))) {
  comp += +1.0035E0f / (var_11 + (-1.9734E35f + var_12));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13);
  hipDeviceSynchronize();

  return 0;
}
