#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18) {
comp = +1.7618E-35f * (var_2 - sqrtf((var_3 - +1.1633E-15f)));
if (comp < -0.0f - +1.2487E-43f / (var_4 / var_5)) {
  comp = coshf((+1.2113E-37f * -0.0f / +0.0f / (var_6 + (+1.5627E-44f - +1.7877E-35f))));
}
for (int i=0; i < var_1; ++i) {
  comp += (var_7 - var_8 - (var_9 - fmodf(var_10 + +0.0f, var_11 / (-1.4742E-35f / var_12 + -1.5252E-43f + (-1.6087E-4f / var_13)))));
comp = (var_14 - (var_15 * +1.2097E-43f));
}
if (comp <= var_16 * (+1.7184E-43f / var_17 / expf((+1.2396E36f / var_18)))) {
  comp += (+1.5236E1f - (+1.8120E34f * (-1.2165E36f - (-1.1129E36f / +1.6816E-44f))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19);
  hipDeviceSynchronize();

  return 0;
}
