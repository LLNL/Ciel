#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29) {
for (int i=0; i < var_1; ++i) {
  comp = (+0.0f + var_2 / +1.4133E-21f);
if (comp > (var_3 / logf((+0.0f * fmodf(+1.6305E-41f * coshf(var_4 * floorf(acosf(+1.3974E-37f / +1.7690E34f * (var_5 + +1.9761E36f)))), (var_6 / (var_7 * (var_8 - log10f(var_9 / expf((var_10 / (+1.2161E-42f / -1.4818E22f - (var_11 * var_12))))))))))))) {
  comp = (var_13 / var_14 - var_15 + logf((-1.4388E36f - +1.0373E34f - (var_16 + (var_17 + var_18)))));
float tmp_1 = -0.0f;
float tmp_2 = +1.7996E-17f / +1.6241E-43f - var_19 + atan2f(-1.0049E-3f, +1.0579E-42f - var_20);
comp = tmp_2 / tmp_1 * +0.0f * var_21 * -1.9009E-36f;
}
if (comp == (+1.9458E-44f * var_22)) {
  float tmp_3 = +0.0f;
float tmp_4 = +0.0f;
float tmp_5 = var_23 / (var_24 / log10f((var_25 * (-1.0542E-28f / acosf((+1.6977E-43f - var_26 * +1.8864E34f))))));
comp = tmp_5 - tmp_4 - tmp_3 + -1.1635E-43f - (var_27 * (+0.0f / var_28 * var_29));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30);
  hipDeviceSynchronize();

  return 0;
}
