#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float* var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
if (comp <= (var_2 * (+1.1712E-19f + +1.9126E-21f / var_3))) {
  comp += var_4 * var_5 / asinf(var_6 + (+1.5998E-36f - +1.1812E-36f));
float tmp_1 = +1.2498E-37f;
comp = tmp_1 * (var_7 + (+1.9926E10f + var_8 * sinf(atan2f(var_9 + (var_10 * var_11 * var_12), -1.6014E-42f / (+1.9146E-36f - (-0.0f - (-1.2931E-2f + var_13)))))));
for (int i=0; i < var_1; ++i) {
  var_14[i] = -1.7265E-41f;
comp += var_14[i] * (+1.0268E10f + (var_15 + (+1.7805E-43f * (var_16 * var_17))));
comp = var_18 / (var_19 - var_20);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float* tmp_15 = initPointer( atof(argv[15]) );
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
