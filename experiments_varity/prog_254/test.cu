#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11) {
if (comp < (-1.3332E2f * (var_1 + +1.9777E-37f + -1.7981E26f / (var_2 - var_3)))) {
  comp = (var_4 / +0.0f / atan2f(+1.1750E-37f, -1.5602E-26f / (+0.0f / (+1.5070E-42f / -1.2048E-36f / var_5))));
float tmp_1 = atanf((var_6 + var_7));
comp += tmp_1 + -1.6271E35f * (-1.0488E-37f / (-1.5361E-41f * var_8 / var_9));
comp += powf(var_10 + -0.0f, (+0.0f / (var_11 * (-1.1546E-26f / +1.1412E-42f))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12);
  hipDeviceSynchronize();

  return 0;
}
