#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    if (comp == (+0.0f * (var_3 - (-1.1451E-42f + log10f((-1.9301E19f * atan2f(+1.0375E34f - +1.3706E35f, +1.0134E-42f))))))) {
      if (comp > (+0.0f + (-1.4757E-42f + (+1.9893E-36f + var_4 / -1.0768E-37f)))) {
        comp += (+1.6657E4f + var_5);
float tmp_1 = (+1.3109E-15f * +0.0f * -1.6854E-37f);
comp = tmp_1 / (var_6 - var_7 + +1.1093E-42f);
if (comp >= +1.0722E-37f - (-1.1822E-36f / (var_8 * (+1.4011E-35f * var_9)))) {
  comp += (var_10 - -1.3518E-44f + (var_11 * -1.9255E-9f * (var_12 + -1.9572E-35f)));
}
if (comp == (var_13 + sinf((-1.7602E34f * var_14)))) {
  comp = (var_15 - sinhf(ceilf(log10f((+1.0229E35f * (-1.5434E-44f / var_16))))));
}
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
