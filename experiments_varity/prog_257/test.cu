#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8) {
float tmp_1 = +1.1567E-44f;
float tmp_2 = -1.0589E36f;
comp = tmp_2 + tmp_1 - (var_1 - +1.3563E19f + atan2f((var_2 / (+1.9121E21f / ceilf((var_3 * log10f((-1.9333E36f + (var_4 * var_5 / var_6))))))), var_7 - -1.8153E-43f));
comp += -0.0f / var_8;
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9);
  hipDeviceSynchronize();

  return 0;
}
