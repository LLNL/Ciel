#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float* var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    float tmp_1 = +1.1019E35f;
float tmp_2 = var_4 + +0.0f + +1.6033E-3f / fmodf(var_5 - +1.7091E-37f, -1.2257E-16f - -1.3731E-37f / var_6 / var_7 * (-0.0f / var_8));
comp += tmp_2 * tmp_1 / (-1.2086E36f * +1.0334E-37f);
comp += (var_9 / (-1.6171E-6f * var_10));
if (comp == var_11 + var_12 * var_13) {
  float tmp_3 = var_14 * var_15;
comp = tmp_3 - -1.5535E-6f - -1.7337E-12f;
comp += (var_16 - var_17 - var_18);
comp = (+0.0f - var_19);
}
for (int i=0; i < var_3; ++i) {
  float tmp_4 = coshf((-1.1252E-35f * var_21 / +0.0f - var_22 * (+0.0f * +1.6641E-44f)));
var_20[i] = -1.9894E-20f / atanf(+0.0f * (+1.2795E34f + +0.0f));
comp = var_20[i] + tmp_4 - tanhf((var_23 / -1.5336E-36f / var_24 / -1.0568E-43f));
comp = -1.1037E-42f / (+0.0f - var_25 + var_26);
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float* tmp_21 = initPointer( atof(argv[21]) );
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27);
  hipDeviceSynchronize();

  return 0;
}
