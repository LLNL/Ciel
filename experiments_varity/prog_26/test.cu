#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9) {
float tmp_1 = +1.0038E-36f - (var_1 - -1.4921E-37f + (var_2 / +1.5454E-35f));
comp += tmp_1 - (-0.0f * var_3 / var_4 + (-1.4969E35f * +1.6195E36f / +1.1516E35f));
if (comp == (var_5 - +1.8212E-37f)) {
  float tmp_2 = -1.6362E-22f;
comp = tmp_2 + var_6 / (var_7 / (+1.5262E34f / (+1.8580E25f + expf((+0.0f / (var_8 / ldexpf(floorf(var_9 * +0.0f * -1.8698E-35f), 2)))))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10);
  hipDeviceSynchronize();

  return 0;
}
