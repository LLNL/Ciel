#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
comp += sinf((var_3 + var_4 * sinhf(-1.3639E-28f + +1.4757E-25f)));
comp = +1.3487E-44f / +1.0463E35f - (var_5 / var_6 * +0.0f);
comp += (var_7 - var_8);
if (comp < (-1.0324E2f / -1.8803E36f * var_9 + var_10)) {
  comp = +1.4447E-21f * (+1.5484E-20f - sqrtf(-1.6733E-41f));
}
for (int i=0; i < var_1; ++i) {
  comp += var_11 - (var_12 * var_13);
float tmp_1 = -1.1326E11f;
comp = tmp_1 + (-1.1925E-43f - (-1.9516E-37f * ldexpf(-1.5474E27f + (+1.2125E36f - acosf(ceilf((+1.9126E-35f - +0.0f)))), 2)));
comp = +1.1504E-35f * (+1.1777E36f - +1.1876E34f + (+1.6106E34f - +1.7015E-35f));
}
for (int i=0; i < var_2; ++i) {
  comp = (var_14 - tanhf(var_15 * sinhf(var_16 + (var_17 - acosf(-0.0f / var_18 + var_19 * sinf((var_20 / -1.8291E-36f * -0.0f * (+1.4862E-37f / -1.4657E34f))))))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
