#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float* var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
comp = +1.6275E-36f * acosf((var_3 * +1.7143E-44f * var_4 * var_5));
for (int i=0; i < var_1; ++i) {
  float tmp_1 = -1.0004E-43f;
var_6[i] = +1.9263E-10f * +1.0626E35f * var_7 / coshf(sinhf(atanf((var_8 + +0.0f * floorf((-0.0f + -1.1158E34f - var_9))))));
comp = var_6[i] + tmp_1 + floorf(sinf((-1.8596E-44f / atanf(fmodf(-0.0f, +1.9840E-37f)))));
comp = (-1.3902E15f - (+1.0030E8f * +1.7903E-18f - -1.9076E-43f));
}
if (comp > -1.9972E-36f / var_10) {
  comp += cosf(+1.2899E-41f * (-1.5746E34f + var_11 / -1.4372E-41f));
comp += (var_12 * cosf(var_13 / +1.7551E34f));
}
for (int i=0; i < var_2; ++i) {
  comp += expf(var_14 * (var_15 * var_16));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float* tmp_7 = initPointer( atof(argv[7]) );
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
