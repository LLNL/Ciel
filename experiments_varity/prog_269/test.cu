#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26) {
float tmp_1 = +1.4765E-41f;
float tmp_2 = (-1.1524E35f + var_2 + tanhf((var_3 + -0.0f + var_4)));
comp += tmp_2 * tmp_1 - (var_5 - var_6 + (var_7 / var_8 / +1.1342E36f));
if (comp == tanhf((var_9 / var_10 * (-1.3232E-41f * +1.5065E36f / var_11)))) {
  comp += floorf((var_12 - (var_13 - -1.6960E22f / (+1.6586E-41f / var_14))));
}
for (int i=0; i < var_1; ++i) {
  comp += expf(var_15 / expf(-1.8008E-44f / var_16 / sinf(-1.2851E-29f)));
}
if (comp < (var_17 - +1.1785E-36f - (-1.9774E36f - -1.5073E-44f / (var_18 + var_19)))) {
  comp += (+1.9533E-44f - var_20);
comp = -1.4391E-35f / -1.3850E0f + logf(+1.6538E-43f / (-1.6805E-41f - var_21));
float tmp_3 = (+1.4592E8f * var_22 + -1.0567E5f * var_23 / var_24);
comp += tmp_3 * var_25 * var_26;
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27);
  hipDeviceSynchronize();

  return 0;
}
