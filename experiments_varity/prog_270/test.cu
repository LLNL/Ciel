#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
if (comp >= (-1.6173E28f / (+0.0f + (-1.9592E-35f * (var_2 - acosf(-1.1004E-37f + (+0.0f - +1.8978E-37f))))))) {
  float tmp_1 = +1.2686E-36f;
float tmp_2 = +0.0f;
comp += tmp_2 / tmp_1 * (var_3 - (var_4 - (+0.0f * -0.0f)));
if (comp == coshf(-1.3932E4f)) {
  float tmp_3 = var_5 - +0.0f - +1.0800E-43f + (+1.2976E35f * +1.6316E-36f);
comp = tmp_3 / (+0.0f - var_6);
comp += +1.9634E34f / (+1.4764E-35f * (var_7 - var_8));
}
for (int i=0; i < var_1; ++i) {
  comp += (-0.0f * ceilf((var_9 / (var_10 - var_11 * +1.1111E-35f))));
}
if (comp >= (var_12 / +1.2972E34f)) {
  comp = (+1.6259E-18f - (var_13 * +1.1784E34f + +0.0f * var_14));
comp += var_15 + asinf(+1.1538E-44f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
