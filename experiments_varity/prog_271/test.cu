#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float* var_21,float var_22,float var_23,float var_24) {
comp += (-1.8025E35f + (var_3 + -1.3294E36f * (var_4 - var_5)));
float tmp_1 = -1.7807E-43f;
comp += tmp_1 / cosf(-1.5429E-21f);
comp += (var_6 * (var_7 * -1.3947E-36f - -1.9180E-8f + asinf((+1.4801E-41f - logf((+1.3532E7f / var_8 * -1.1467E35f))))));
if (comp > +1.5884E-44f * cosf(var_9 - sinhf((var_10 * floorf(+1.9366E-41f))))) {
  comp = var_11 + var_12 + var_13;
comp += var_14 / -1.9598E24f;
}
for (int i=0; i < var_1; ++i) {
  float tmp_2 = (var_15 * (-0.0f + (var_16 / (var_17 - -1.1200E34f + var_18))));
float tmp_3 = +1.2603E35f;
comp = tmp_3 - tmp_2 - (var_19 * (-1.2492E-42f * +1.9760E35f * -1.0139E34f * (+0.0f / var_20)));
}
for (int i=0; i < var_2; ++i) {
  var_21[i] = +1.9935E-37f + +1.7535E-37f + var_22 / +1.9573E-41f;
comp = var_21[i] + (var_23 / sinhf((-1.7727E-7f / (var_24 - (+1.5863E-41f * -1.7726E-36f)))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float* tmp_22 = initPointer( atof(argv[22]) );
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25);
  hipDeviceSynchronize();

  return 0;
}
