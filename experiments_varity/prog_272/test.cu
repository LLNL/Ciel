#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float* var_9,float var_10,float var_11,float var_12) {
for (int i=0; i < var_1; ++i) {
  if (comp >= (var_2 / -1.0817E-36f)) {
    comp = cosf(-1.9800E34f / (var_5 + ldexpf((-1.8078E-42f * +1.4986E-41f), 2)));
float tmp_1 = -1.4794E-35f;
float tmp_2 = -1.9424E-37f;
comp += tmp_2 / tmp_1 - (-1.3063E-18f - -1.0401E-35f * +1.2162E-43f - var_6);
if (comp >= +0.0f / var_7 / ceilf(+1.0071E-44f)) {
  comp += log10f((+1.3650E-35f + var_8));
}
for (int i=0; i < var_3; ++i) {
  comp += -1.2400E-37f + +1.4328E-42f;
var_9[i] = +1.6281E-15f;
float tmp_3 = +1.6286E36f;
comp = tmp_3 / var_9[i] * -1.9352E-41f * -1.4034E-44f + -1.6408E-13f + var_10 * +1.1786E7f;
}
for (int i=0; i < var_4; ++i) {
  comp = (var_11 + var_12);
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float* tmp_10 = initPointer( atof(argv[10]) );
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13);
  hipDeviceSynchronize();

  return 0;
}
