#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float* var_4,float var_5,float var_6) {
comp = -1.8675E-37f / (-1.3941E-18f / +1.3800E-43f - (-1.6037E-9f / var_2));
float tmp_1 = -1.9551E34f;
comp += tmp_1 / (+1.5946E-26f - var_3);
for (int i=0; i < var_1; ++i) {
  var_4[i] = -1.5607E-42f;
float tmp_2 = -0.0f;
comp += tmp_2 * var_4[i] / (var_5 / coshf((var_6 * +1.3524E-41f)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float* tmp_5 = initPointer( atof(argv[5]) );
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7);
  hipDeviceSynchronize();

  return 0;
}
