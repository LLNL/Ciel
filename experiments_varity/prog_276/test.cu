#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float* var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18) {
for (int i=0; i < var_1; ++i) {
  comp += (var_4 * var_5 / -1.6512E-35f / (var_6 / (-1.1800E-36f - var_7)));
var_2[i] = (+1.7674E28f + (-1.3341E-42f / -1.8035E1f));
comp = var_2[i] * var_8 * var_9 + (var_10 + (-0.0f + var_11 - var_12));
comp = (+0.0f * (+1.5812E-41f / +1.3020E-36f * (-1.2470E-35f - +1.6987E-41f)));
for (int i=0; i < var_3; ++i) {
  float tmp_1 = var_13 + -0.0f - var_14 + (var_15 * +1.4818E-43f);
comp = tmp_1 / var_16 / +1.8243E36f - (var_17 + (var_18 * -1.1123E35f));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float* tmp_3 = initPointer( atof(argv[3]) );
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19);
  hipDeviceSynchronize();

  return 0;
}
