#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,int var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
for (int i=0; i < var_1; ++i) {
  if (comp > (var_2 * (var_3 - (+1.8532E-43f / +1.9626E14f * (+1.7886E-37f * var_4))))) {
    if (comp >= (var_5 * (var_6 - var_7))) {
      comp += (-0.0f - sinhf((-0.0f / var_9 + floorf(+1.5832E-42f))));
float tmp_1 = -1.1252E-15f;
float tmp_2 = -1.3006E-24f;
comp += tmp_2 - tmp_1 - sinhf((-0.0f - (var_10 / var_11 + (var_12 / +1.1462E34f + var_13))));
if (comp >= atanf(+1.0472E-42f)) {
  float tmp_3 = -1.4667E36f;
comp += tmp_3 + (var_14 + (var_15 * var_16 * var_17));
}
for (int i=0; i < var_8; ++i) {
  comp = log10f(+1.2622E-35f);
}
if (comp == atan2f(floorf(+1.8171E-8f), (-1.3632E-43f / var_18 - (+0.0f - +1.9495E-41f)))) {
  comp = (-1.3993E25f / (+1.0080E-42f * var_19 + var_20));
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  int tmp_9 = atoi(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
