#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30,float var_31,float var_32,float var_33) {
comp = sinf((var_1 * var_2 / coshf(+1.6899E-36f - var_3 - var_4 + var_5)));
comp = +1.2287E-43f + -1.8636E-35f;
comp += (var_6 - (var_7 - -1.9095E5f * (+1.9059E-37f - logf((-1.5088E-36f / fmodf(sinf(var_8 + sqrtf((var_9 + (+0.0f + var_10)))), (var_11 + ldexpf((-1.6471E25f * sqrtf((var_12 + -1.3755E-37f + var_13 * powf((var_14 - -1.9217E35f / var_15 / sinf(-1.4114E-42f)), +0.0f / (-1.5518E13f / var_16 / (var_17 / atanf(+1.0853E29f * +1.2548E-3f))))))), 2))))))));
comp += (+1.3202E-44f - +1.1656E-36f / var_18 - +1.7676E-36f * var_19);
if (comp == var_20 + fmodf((var_21 + +1.1755E-37f + var_22 * (var_23 - acosf(cosf((+0.0f - (-1.1293E36f + -0.0f / +1.7082E-42f)))))), +1.7465E-15f / var_24 * (+1.1625E36f + var_25))) {
  comp = (-1.7123E15f - +1.0857E-44f / var_26 * -1.0677E-41f);
comp += (+1.1663E-20f * (+1.0656E-43f - ceilf(ldexpf((+1.8071E-25f * var_27 * var_28 - (var_29 + -1.1838E-28f * +1.2957E-36f)), 2))));
float tmp_1 = floorf(atan2f(+1.3948E-44f + (var_30 + var_31 / +1.9812E36f), (-0.0f + (-1.9448E-36f * atanf(+1.9803E-35f)))));
comp += tmp_1 - (-1.1810E-43f / (var_32 + sqrtf(-1.3258E35f / -1.0834E36f + (+0.0f - (-1.0444E-35f / var_33)))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);
  float tmp_33 = atof(argv[33]);
  float tmp_34 = atof(argv[34]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32,tmp_33,tmp_34);
  hipDeviceSynchronize();

  return 0;
}
