#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
comp += ldexpf(+1.8980E34f, 2);
comp += ceilf(var_2 * (+1.2572E-44f - var_3));
for (int i=0; i < var_1; ++i) {
  float tmp_1 = +1.4885E35f;
comp = tmp_1 / +1.4882E34f - var_4 + (var_5 / +0.0f);
comp += floorf((+1.2889E-36f * var_6 + +1.9070E-1f + var_7));
comp += +0.0f + -1.6281E-37f + var_8;
}
if (comp >= +1.6991E-37f - +1.6259E36f - asinf(-0.0f * log10f(sqrtf(var_9 + var_10)))) {
  float tmp_2 = (-0.0f - acosf(expf(asinf(logf((var_11 - +0.0f * var_12 / -0.0f))))));
float tmp_3 = +1.4344E-42f;
comp = tmp_3 * tmp_2 + asinf(-1.4016E35f / var_13 * var_14 - (-0.0f / var_15));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
