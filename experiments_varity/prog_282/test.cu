#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float* var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = (-1.2038E-44f * var_4 - +1.6361E35f);
var_2[i] = -1.8826E-2f - var_5 * var_6;
comp += var_2[i] / tmp_1 * -1.1648E35f / var_7 / coshf((var_8 - (-1.8908E-44f / powf(-1.6383E36f, -1.6132E-44f))));
for (int i=0; i < var_3; ++i) {
  comp = var_9 - atan2f(+0.0f * +1.8064E25f, (+1.0848E19f + (-0.0f + var_10)));
comp = (+1.1826E12f / (-1.9647E-44f * var_11 - +1.5201E-41f));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float* tmp_3 = initPointer( atof(argv[3]) );
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12);
  hipDeviceSynchronize();

  return 0;
}
