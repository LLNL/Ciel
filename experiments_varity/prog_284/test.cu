#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float* var_11,float var_12,float var_13,float var_14,float var_15) {
comp = +1.5895E6f * (var_2 + (+1.1424E-43f + -1.1627E25f));
comp += (var_3 / var_4 * var_5);
float tmp_1 = (-0.0f / (var_6 * var_7 - floorf((var_8 / var_9 + var_10))));
comp = tmp_1 * floorf(-1.6478E-44f);
for (int i=0; i < var_1; ++i) {
  var_11[i] = -1.4474E-5f;
comp += var_11[i] - powf(var_12 / +1.2489E36f * (var_13 * (+1.3797E-35f / -0.0f)), atanf((+1.6702E-36f - (var_14 + -1.4522E35f + var_15))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float* tmp_12 = initPointer( atof(argv[12]) );
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
