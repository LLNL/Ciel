#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18) {
comp += (var_2 * asinf(var_3 / (-1.5128E-35f / var_4 * (-1.8498E35f + -1.7030E-36f))));
float tmp_1 = -1.5426E36f;
comp = tmp_1 * (-1.0412E22f + logf((var_5 + (-1.4435E-43f - var_6))));
for (int i=0; i < var_1; ++i) {
  float tmp_2 = -1.9845E-18f;
comp = tmp_2 / (var_7 * var_8 + (+1.9783E-24f * var_9 / sinhf((var_10 * -1.6701E36f / (var_11 - (var_12 - -1.0329E-44f * +1.2002E-44f))))));
}
if (comp < atanf(var_13 * -0.0f + (-1.6679E36f - (-1.4014E-37f / var_14)))) {
  comp = (var_15 + logf(-0.0f));
comp = sinhf(var_16 - -1.7208E-36f + (var_17 * var_18));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19);
  hipDeviceSynchronize();

  return 0;
}
