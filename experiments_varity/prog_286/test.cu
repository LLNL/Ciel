#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12) {
if (comp >= (var_1 - +0.0f)) {
  if (comp >= (var_2 - var_3)) {
    comp += +0.0f + var_5;
float tmp_1 = (var_6 / logf((var_7 - (var_8 + -1.1764E-16f))));
comp = tmp_1 + (var_9 - (+0.0f + (-1.3204E34f * (-1.3519E-42f - (-1.6717E34f / var_10)))));
for (int i=0; i < var_4; ++i) {
  comp = logf((+1.9378E-36f + (var_11 * (var_12 - -1.3730E-42f))));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13);
  hipDeviceSynchronize();

  return 0;
}
