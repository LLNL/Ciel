#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3) {
for (int i=0; i < var_1; ++i) {
  comp += (-1.3351E36f + +1.7767E-36f + -1.4644E34f);
comp = (+1.8205E-35f / (var_2 - -0.0f));
comp += (var_3 - (+1.0068E-35f + -1.2231E-10f));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4);
  hipDeviceSynchronize();

  return 0;
}
