#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float* var_3,float* var_4,float var_5,float var_6,float var_7,float var_8) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    comp += (+1.2580E-8f + -0.0f - -1.2954E34f - +1.2309E-37f + (+0.0f - -1.8435E14f));
var_3[i] = sinhf((-1.8802E-43f + var_5 / (+1.7557E-42f / +1.5754E-36f)));
var_4[i] = -1.3126E-41f;
comp = var_4[i] + var_3[i] * var_6 + +1.6813E34f / floorf((var_7 + (-1.2129E-41f / var_8)));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float* tmp_4 = initPointer( atof(argv[4]) );
  float* tmp_5 = initPointer( atof(argv[5]) );
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9);
  hipDeviceSynchronize();

  return 0;
}
