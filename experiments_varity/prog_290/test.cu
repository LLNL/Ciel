#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12) {
if (comp >= var_1 + var_2) {
  comp = (+0.0f * (var_5 * sqrtf(var_6 - (var_7 - (+1.6948E-42f / var_8)))));
comp += -0.0f * -0.0f;
for (int i=0; i < var_3; ++i) {
  comp += var_9 - -1.5949E-36f + var_10 * (var_11 / +0.0f);
}
for (int i=0; i < var_4; ++i) {
  comp += (-1.7793E34f - var_12);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13);
  hipDeviceSynchronize();

  return 0;
}
