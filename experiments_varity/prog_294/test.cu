#include "hip/hip_runtime.h"


typedef long double float_80;


/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(double comp, double var_1) {
if (comp > cosf((-1.2114E34f * +1.6991E-28f * (+1.6129E-37f - -1.0721E-44f)))) {
  comp = (-0.0f + -0.0f / var_1 + (-0.0f - -1.2560E12f));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(double v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);

  compute<<<1,1>>>(tmp_1,tmp_2);
  hipDeviceSynchronize();

  return 0;
}
