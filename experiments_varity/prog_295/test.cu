#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18) {
float tmp_1 = var_1 - (var_2 / (+1.7773E-36f / +1.5831E-44f + -1.1443E-44f + -0.0f));
comp = tmp_1 * (var_3 * (+1.2327E-44f - ceilf((-0.0f / coshf(-1.0330E36f)))));
float tmp_2 = +1.8125E28f;
comp = tmp_2 + +1.9547E36f / +1.6747E-44f + (+1.2656E-37f - (var_4 - floorf(+0.0f - var_5)));
if (comp >= acosf(ceilf((var_6 - var_7 / var_8 - (var_9 + cosf(cosf(+0.0f / (+1.7116E-43f * var_10)))))))) {
  float tmp_3 = +1.2226E-4f;
comp = tmp_3 + (var_11 * sqrtf((var_12 / var_13)));
comp = asinf((var_14 / var_15 - -1.8163E-30f - (-1.5259E-36f + (var_16 + +1.0313E-9f))));
comp = -1.2212E-36f * atanf(-1.9089E11f - var_17 * (+0.0f - var_18 - -0.0f));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19);
  hipDeviceSynchronize();

  return 0;
}
