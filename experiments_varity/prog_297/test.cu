#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
float tmp_1 = +1.1614E-44f;
comp = tmp_1 - +1.4067E36f * powf(atan2f(fabsf(var_2 / -0.0f - +0.0f - var_3), var_4 / (+1.6047E-41f + var_5 - var_6)), var_7 * expf((var_8 * +1.7352E35f)));
for (int i=0; i < var_1; ++i) {
  comp = sinhf((var_9 * var_10));
}
if (comp > -1.4887E-23f * -1.0739E-4f - var_11) {
  comp += -1.8706E34f - var_12 - var_13;
float tmp_2 = +1.2238E-35f;
float tmp_3 = (+1.5282E-44f + fabsf(coshf(sqrtf(-0.0f * cosf(-1.7876E34f * ceilf(var_14 * var_15 / var_16))))));
comp = tmp_3 - tmp_2 / var_17 - logf(var_18 - var_19 + (-0.0f / (-0.0f / var_20)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
