#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11) {
for (int i=0; i < var_1; ++i) {
  if (comp > (-1.9213E35f + +1.9956E-36f - (var_2 - (var_3 * -0.0f)))) {
    comp += var_4 + +1.5921E-41f / +1.2842E-43f;
if (comp >= var_5 * var_6 / var_7 * +1.0692E36f / var_8) {
  float tmp_1 = -1.0057E-44f;
float tmp_2 = (var_9 / atanf(+1.5578E-42f - var_10 / +1.7388E3f));
comp = tmp_2 / tmp_1 - (-1.8142E-42f + coshf((+0.0f * (-0.0f + var_11 * +1.1072E-37f))));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12);
  hipDeviceSynchronize();

  return 0;
}
