#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float* var_3,float var_4) {
if (comp == floorf(-0.0f)) {
  float tmp_1 = +1.3963E-6f;
comp += tmp_1 - (+1.6619E35f * var_2 - -1.5939E-42f);
for (int i=0; i < var_1; ++i) {
  comp += log10f(floorf(sinf((+1.8762E35f - (+1.5776E-44f / var_4)))));
var_3[i] = asinf(-1.0657E-26f);
float tmp_2 = -1.9984E-7f;
comp = tmp_2 - var_3[i] * (+0.0f + (-1.9173E13f / -1.6537E25f));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float* tmp_4 = initPointer( atof(argv[4]) );
  float tmp_5 = atof(argv[5]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5);
  hipDeviceSynchronize();

  return 0;
}
