#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float* var_20,float var_21,float var_22) {
if (comp < (var_1 + (-1.0618E-14f + var_2))) {
  comp = (var_4 + tanhf((-0.0f + -0.0f)));
comp = log10f(+1.6829E16f);
comp = (var_5 * (var_6 / (var_7 - (var_8 - (+1.9521E-36f * -1.7751E35f)))));
if (comp <= (var_9 + (-1.0013E-44f + cosf((+1.5759E-43f + var_10))))) {
  float tmp_1 = -1.6744E-30f;
float tmp_2 = (var_11 + floorf(asinf(-1.1162E-2f / var_12 * ceilf((var_13 - var_14 + var_15)))));
comp = tmp_2 / tmp_1 - (var_16 / (-1.8482E-42f + ceilf(-1.3313E-44f)));
comp += (var_17 + -1.8945E20f + +1.7824E-36f - var_18 + var_19);
}
for (int i=0; i < var_3; ++i) {
  var_20[i] = +1.9524E-42f;
float tmp_3 = -1.1483E-36f;
comp += tmp_3 - var_20[i] + +1.0162E15f * var_21;
comp += var_22 / +1.2242E36f + -0.0f / -1.6344E-44f;
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float* tmp_21 = initPointer( atof(argv[21]) );
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
