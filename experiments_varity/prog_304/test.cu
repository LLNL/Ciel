#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30) {
float tmp_1 = -1.2697E-14f;
float tmp_2 = -1.3468E-36f;
comp += tmp_2 - tmp_1 - ceilf(+0.0f + var_2);
comp += (+1.2694E-44f - log10f(-1.1992E-27f / var_3 - +1.4077E-37f));
if (comp == (-1.4754E-37f + (var_4 / (var_5 * (var_6 * +1.6948E-43f / -1.8290E-41f))))) {
  comp = tanhf((+1.5556E6f * (var_7 * var_8)));
comp += fabsf(var_9 - logf(+0.0f / (var_10 / (var_11 / -1.7520E-35f))));
}
for (int i=0; i < var_1; ++i) {
  comp += fabsf(sinf(fabsf(atan2f((var_12 * (var_13 + var_14 - -0.0f)), var_15 - (var_16 * var_17)))));
comp = sinf(cosf((+0.0f / (var_18 * var_19))));
float tmp_3 = +0.0f - (var_20 * +1.8894E-42f * powf(var_21 + var_22 + (var_23 / -1.2133E36f - var_24), +0.0f + (+1.1637E14f * var_25)));
comp += tmp_3 - -1.8727E-37f - (var_26 / (+1.7552E-23f + floorf(-1.4537E-41f)));
}
if (comp >= +0.0f / cosf(var_27 * +1.1833E-36f / (var_28 * -1.4010E-44f))) {
  comp += var_29 + (-1.5582E7f + +1.8941E35f / -0.0f);
comp = logf(-1.8967E-36f);
comp = ldexpf(var_30 * -1.3929E-43f, 2);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31);
  hipDeviceSynchronize();

  return 0;
}
