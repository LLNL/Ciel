#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7) {
comp = (+1.7403E-37f / +1.0513E-3f);
if (comp <= ceilf(+1.7902E19f)) {
  float tmp_1 = (var_2 + fabsf(-1.8896E18f - var_3 + -1.1632E-36f));
comp += tmp_1 * var_4 - (+1.3793E-44f + var_5);
}
for (int i=0; i < var_1; ++i) {
  comp += +1.6315E-24f - +1.1241E34f;
float tmp_2 = -1.9265E4f;
comp += tmp_2 - fabsf(+1.2871E-36f / var_6 - var_7);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8);
  hipDeviceSynchronize();

  return 0;
}
