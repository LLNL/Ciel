#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float* var_14,float var_15,float var_16,float var_17) {
if (comp >= (-0.0f - expf(-1.7786E19f + (-1.6951E21f + var_1 + (-1.5015E24f - var_2))))) {
  comp = (-1.4463E-24f + ceilf(var_5 + (var_6 / -1.3738E-44f - +1.8329E-6f - (-0.0f + var_7))));
if (comp < (var_8 - var_9)) {
  float tmp_1 = -1.0008E35f;
float tmp_2 = -1.9518E-36f;
float tmp_3 = -1.6526E36f;
comp = tmp_3 / tmp_2 + tmp_1 + sinhf((var_10 / (var_11 - -0.0f)));
}
for (int i=0; i < var_3; ++i) {
  comp += (var_12 + (var_13 - -1.0199E-36f));
}
for (int i=0; i < var_4; ++i) {
  comp += var_15 / +1.3643E-35f;
var_14[i] = -1.7821E-35f;
comp += var_14[i] * var_16 / var_17;
comp = cosf(-1.4415E27f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float* tmp_15 = initPointer( atof(argv[15]) );
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18);
  hipDeviceSynchronize();

  return 0;
}
