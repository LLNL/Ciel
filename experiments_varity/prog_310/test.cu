#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float* var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22) {
if (comp < (var_3 - -1.4449E-43f * +0.0f * var_4 / -1.9542E-42f / +1.3316E-14f)) {
  float tmp_1 = +1.9876E-41f * +1.1972E-37f * var_5;
comp += tmp_1 * +0.0f / -0.0f;
for (int i=0; i < var_1; ++i) {
  comp = (-1.3027E11f - (var_7 * var_8));
var_6[i] = +1.1976E35f;
comp += var_6[i] + +1.7507E-44f * (var_9 / -0.0f);
}
if (comp >= -0.0f - var_10 / +1.5753E35f) {
  comp += -1.5965E-22f - expf(-1.7611E-43f - atanf(var_11 + ceilf(var_12 + sqrtf((var_13 * (-1.0828E-13f * -1.0043E-21f + +0.0f))))));
float tmp_2 = (var_14 / (+0.0f - (var_15 - (var_16 + -1.8095E-8f))));
comp = tmp_2 / (+1.5289E-42f - logf(+0.0f));
comp = ceilf(-1.9964E35f);
}
for (int i=0; i < var_2; ++i) {
  comp = acosf((var_17 / -1.4892E28f * var_18 - var_19 - powf((-1.1905E36f * fabsf(+1.0598E35f - var_20 + +0.0f)), expf((var_21 / (var_22 / (+1.2287E34f + expf(-1.3934E-7f))))))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float* tmp_7 = initPointer( atof(argv[7]) );
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
