#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float* var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    comp += (var_4 / var_5);
for (int i=0; i < var_3; ++i) {
  comp = (-1.4981E-27f * fabsf(+0.0f));
comp = var_7 * expf(-1.0115E-44f);
var_6[i] = -1.6598E-41f * (-1.3358E34f - log10f((var_8 * sinhf(+1.3993E-37f + (-1.9420E4f * +1.4445E-43f / -1.5239E34f)))));
comp = var_6[i] - var_9 * (+1.3838E-35f / (+0.0f + (-1.8751E34f / var_10)));
}
if (comp == (var_11 + +1.3706E-36f)) {
  float tmp_1 = asinf(-1.4726E23f + -0.0f * (+1.0269E-35f / var_12));
comp = tmp_1 + (-1.1191E36f * powf(-1.8951E-43f, var_13 / sqrtf((var_14 + (var_15 + logf(-1.1694E-3f - acosf((-1.0170E34f - (var_16 + (+1.0204E-12f * var_17))))))))));
comp = var_18 * var_19 + -1.8301E-10f - var_20 + var_21 * -1.8279E-4f;
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float* tmp_7 = initPointer( atof(argv[7]) );
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22);
  hipDeviceSynchronize();

  return 0;
}
