#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float* var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12) {
comp += var_4 * +1.3940E-43f;
for (int i=0; i < var_1; ++i) {
  var_5[i] = (-0.0f + var_6);
comp += var_5[i] - var_7 / asinf(-1.7177E-35f + (+0.0f * +1.2918E-30f - -1.8518E-36f));
}
for (int i=0; i < var_2; ++i) {
  comp = -0.0f - coshf((var_8 / (var_9 / var_10 * -1.2304E34f)));
}
for (int i=0; i < var_3; ++i) {
  comp = ceilf(var_11 + +1.8874E-44f * var_12);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float* tmp_6 = initPointer( atof(argv[6]) );
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13);
  hipDeviceSynchronize();

  return 0;
}
