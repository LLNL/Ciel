#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float* var_2,float* var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25) {
for (int i=0; i < var_1; ++i) {
  var_2[i] = -1.1032E-42f;
float tmp_1 = (+1.2131E-43f - (+1.7832E-35f - (var_5 / +1.7577E24f / fabsf(var_6 / var_7))));
var_3[i] = +1.6061E-21f;
comp += var_3[i] / tmp_1 + var_2[i] / var_8 * var_9;
if (comp >= (-0.0f / +1.0156E35f)) {
  float tmp_2 = var_10 / (+1.6023E-15f - (var_11 * asinf(-0.0f * var_12)));
float tmp_3 = -1.5522E34f / -1.3741E-36f - +0.0f;
comp = tmp_3 - tmp_2 - fabsf((var_13 - (var_14 - +1.5302E26f)));
comp = var_15 / var_16 + -1.9873E34f;
}
for (int i=0; i < var_4; ++i) {
  comp = logf(ldexpf(var_17 * (var_18 * sinf(-1.8472E-29f)), 2));
}
if (comp >= (+1.5056E-44f - var_19 - powf(var_20 / (+1.6543E-15f / (+1.5980E-42f / expf(-1.9057E-42f))), -1.3584E34f))) {
  comp = (var_21 * (var_22 - -1.5530E-43f / var_23));
comp = +0.0f + var_24 + ldexpf((+1.7529E-37f - var_25), 2);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float* tmp_3 = initPointer( atof(argv[3]) );
  float* tmp_4 = initPointer( atof(argv[4]) );
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26);
  hipDeviceSynchronize();

  return 0;
}
