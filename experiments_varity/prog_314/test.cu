#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,int var_3,float var_4,float var_5,float var_6,float* var_7,float* var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22) {
if (comp <= var_1 + var_2 + -0.0f) {
  comp = sinf((+1.4539E-41f * (var_4 + var_5 * (+0.0f * var_6))));
for (int i=0; i < var_3; ++i) {
  var_7[i] = cosf((var_9 * sinf(-1.0267E26f / +1.2468E23f)));
var_8[i] = -1.2872E-42f;
comp += var_8[i] * var_7[i] - -0.0f - var_10;
}
if (comp == (var_11 * logf(+1.6407E19f))) {
  float tmp_1 = (var_12 / var_13 - sqrtf((+1.0377E-37f - +1.5482E-44f + -1.5813E-35f * (-0.0f - -0.0f))));
float tmp_2 = (-1.6053E26f * +1.1100E-37f - ceilf(+1.9220E36f / ceilf(var_14 + (+1.6144E-36f / var_15 * (-1.8990E34f / (+1.5194E24f - -0.0f))))));
float tmp_3 = +0.0f;
comp = tmp_3 / tmp_2 / tmp_1 / var_16 / var_17 / var_18 + -1.0775E35f / sinhf(var_19 * -1.0626E-35f);
}
if (comp < var_20 * -1.6531E-42f * -0.0f) {
  float tmp_4 = -0.0f;
comp = tmp_4 - +0.0f / (var_21 / -1.5219E-44f - var_22);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float* tmp_8 = initPointer( atof(argv[8]) );
  float* tmp_9 = initPointer( atof(argv[9]) );
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
