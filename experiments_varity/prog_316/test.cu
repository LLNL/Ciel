#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float* var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23) {
for (int i=0; i < var_1; ++i) {
  if (comp >= cosf((-1.2614E-37f + +1.0591E35f))) {
    float tmp_1 = +1.4818E-37f;
float tmp_2 = (-1.7030E-5f + logf((-1.1908E36f + tanhf(-0.0f))));
comp = tmp_2 / tmp_1 - coshf((var_3 + floorf(var_4 / asinf((var_5 - var_6 + var_7)))));
comp += fmodf((+1.1795E-8f / (var_8 - (+1.6709E-36f - (-1.4408E1f + var_9)))), -0.0f - var_10);
for (int i=0; i < var_2; ++i) {
  comp = (var_12 * var_13 - +1.3634E34f - +1.5654E34f);
var_11[i] = +1.7711E-37f;
float tmp_3 = (var_14 + var_15);
comp = tmp_3 + var_11[i] - var_16 / -1.9434E-13f + acosf(var_17 / (var_18 / (-0.0f + -1.7130E-12f + sinhf((var_19 - var_20 * var_21 - (var_22 / var_23))))));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float* tmp_12 = initPointer( atof(argv[12]) );
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24);
  hipDeviceSynchronize();

  return 0;
}
