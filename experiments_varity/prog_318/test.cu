#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    for (int i=0; i < var_3; ++i) {
      if (comp == asinf(+1.6052E-29f * (var_4 - (+1.3418E-44f * -1.1849E-36f)))) {
        float tmp_1 = +1.4940E-23f;
float tmp_2 = -1.6240E35f + -1.2672E-41f + +1.0623E34f + atan2f(logf(var_6 * -0.0f), (-1.4919E-42f * atan2f((-1.9619E-37f * (-1.6127E-37f + acosf(var_7 * -1.0424E-41f / -1.8312E-36f * var_8))), var_9 * (-1.5333E-43f * var_10 / powf(var_11 - coshf((var_12 - coshf(var_13 / -1.9055E20f))), -1.0315E9f)))));
comp = tmp_2 * tmp_1 - +1.6798E36f * (+1.1643E34f - (var_14 * +1.4241E36f * (var_15 + var_16)));
comp += var_17 / floorf(+0.0f);
for (int i=0; i < var_5; ++i) {
  comp += +1.6689E-43f - asinf(-1.4086E35f / +0.0f * var_18 + var_19);
}
if (comp == (+1.3292E34f - var_20 * (-1.9025E-42f - var_21))) {
  comp = +1.9010E19f * var_22 * -1.4243E-5f * tanhf((-1.4812E-35f * +1.5328E36f));
}
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
