#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29) {
for (int i=0; i < var_1; ++i) {
  comp += (+1.2679E-36f / var_2 - ldexpf(var_3 - fmodf(atan2f(fabsf((+1.0507E-37f * var_4)), atan2f((var_5 - var_6 * +1.3655E36f), -1.3350E-43f + ldexpf((var_7 - var_8), 2))), var_9 * (var_10 / (-1.8408E-36f / sqrtf(+1.1004E26f)))), 2));
comp += coshf((var_11 * (var_12 + -0.0f + +1.6618E7f + +1.6203E35f)));
comp = (-1.3972E-35f - (var_13 + +1.6199E-37f + var_14));
if (comp < -0.0f - (var_15 - (var_16 / coshf((+1.3904E-44f * (-1.4277E36f / (var_17 / sinf((+1.4439E-37f - +1.9161E36f / (var_18 / -0.0f)))))))))) {
  comp = +1.5744E36f + var_19;
comp += (var_20 / fabsf((-1.4829E-37f - (-1.5397E-35f + log10f(-1.2126E-21f)))));
comp = var_21 * (var_22 / var_23);
}
if (comp <= +0.0f / (-1.3496E-35f - +1.2019E36f)) {
  float tmp_1 = +1.9743E3f;
comp = tmp_1 - (+0.0f + var_24 + (var_25 * cosf((+1.8937E13f - var_26 / ldexpf(var_27 * var_28 + var_29, 2)))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30);
  hipDeviceSynchronize();

  return 0;
}
