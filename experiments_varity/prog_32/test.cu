#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19) {
float tmp_1 = -0.0f;
float tmp_2 = -1.0091E34f;
comp = tmp_2 / tmp_1 / (var_4 / -0.0f * atanf((+1.6260E-44f - (var_5 / -1.6836E-42f - acosf((var_6 * +1.1501E35f))))));
for (int i=0; i < var_1; ++i) {
  comp = (var_7 + (+1.2508E-35f * (var_8 - (+1.2695E-30f + (var_9 * +1.8911E22f)))));
comp += cosf(ceilf(var_10 * -1.7058E21f - fabsf(+0.0f + (var_11 * coshf(-1.5107E-44f)))));
}
for (int i=0; i < var_2; ++i) {
  comp += var_12 * floorf(cosf((+0.0f * acosf((-0.0f / -1.3208E-42f)))));
}
for (int i=0; i < var_3; ++i) {
  comp = -1.7083E35f * (var_13 / +1.1087E-41f);
comp = tanhf(fmodf((-1.8072E-37f - sinhf(var_14 * +1.5227E25f - var_15 * (+1.5400E-35f * var_16))), (var_17 + -1.9998E20f)));
comp = var_18 + ceilf((+1.0309E20f / var_19));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20);
  hipDeviceSynchronize();

  return 0;
}
