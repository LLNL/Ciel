#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27) {
if (comp < var_1 + +1.1615E35f) {
  float tmp_1 = (+1.7453E34f + var_2 - var_3 + (-1.0480E29f + var_4));
float tmp_2 = var_5 + -1.6582E-43f / -1.4460E34f;
comp = tmp_2 * tmp_1 * var_6 + (+1.7474E-36f - var_7 / var_8 * +1.2533E19f + -0.0f);
if (comp > (-1.8305E34f * powf((+1.1921E27f - +1.4396E21f - (+1.1645E2f * sqrtf(var_9 / +1.7072E-36f - (+1.0696E-37f + var_10 / (var_11 + -1.8040E-35f))))), (var_12 - var_13 / (var_14 * -1.7745E-44f / -1.5398E-44f * var_15))))) {
  comp = (var_16 / +0.0f);
}
if (comp < (+1.3188E16f * var_17 * (-1.1849E-44f * (-1.8877E-43f + -0.0f)))) {
  comp += var_18 / var_19 - var_20 / (var_21 - (var_22 / var_23));
}
if (comp <= (+0.0f * +1.5448E-35f)) {
  comp = +1.7405E34f * sinhf(+1.5635E-30f + var_24);
float tmp_3 = +1.5378E-43f;
comp = tmp_3 - (var_25 - var_26 * var_27);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28);
  hipDeviceSynchronize();

  return 0;
}
