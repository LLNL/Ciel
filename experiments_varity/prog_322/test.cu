#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float* var_13,float* var_14,float var_15,float var_16,float var_17,float var_18,float* var_19,float var_20,float var_21,float var_22) {
comp = +1.9693E-12f * (-1.6217E-27f - var_3);
float tmp_1 = -1.1476E20f;
float tmp_2 = -1.4052E17f;
comp = tmp_2 - tmp_1 + atan2f(+1.0054E35f, -1.1269E16f);
if (comp >= (var_4 / +0.0f - -1.2436E25f)) {
  comp += (+1.3861E-17f / var_5 - var_6);
comp += (+1.4150E35f + +1.4644E26f - (var_7 - (var_8 - acosf(var_9 + var_10))));
float tmp_3 = -1.5285E35f;
comp += tmp_3 * var_11 / var_12;
}
for (int i=0; i < var_1; ++i) {
  var_13[i] = logf((+1.9498E-22f - (var_15 - (+0.0f - -0.0f + (+1.0670E34f - var_16)))));
var_14[i] = +0.0f;
comp += var_14[i] - var_13[i] - var_17 * coshf((var_18 - +1.3579E-36f + +1.9518E-43f));
}
for (int i=0; i < var_2; ++i) {
  var_19[i] = +1.0360E-42f;
comp += var_19[i] / var_20 + var_21 / (var_22 + -1.4828E-36f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float* tmp_14 = initPointer( atof(argv[14]) );
  float* tmp_15 = initPointer( atof(argv[15]) );
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float* tmp_20 = initPointer( atof(argv[20]) );
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
