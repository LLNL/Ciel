#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28) {
for (int i=0; i < var_1; ++i) {
  comp = -1.7997E-35f + (+0.0f * +0.0f + var_3);
comp += sqrtf((var_4 - -1.4648E22f / var_5));
comp += ldexpf(var_6 / -1.9809E12f, 2);
if (comp <= (+0.0f + +0.0f + (var_7 - (+1.4959E-43f * var_8)))) {
  comp = (var_9 - expf(atan2f(var_10 / (-1.1194E36f / (var_11 + -1.2978E3f * (-1.4336E35f - var_12))), (+1.4573E-3f - var_13 / (var_14 - (var_15 + (+0.0f - var_16)))))));
comp = var_17 / var_18;
comp = log10f((var_19 - -1.5014E-41f));
}
for (int i=0; i < var_2; ++i) {
  comp = (+1.5254E-42f - (var_20 - -1.1800E-37f * -1.3755E-36f + (var_21 - +1.4218E-30f)));
}
if (comp < +1.3531E-41f * +1.9980E-41f) {
  float tmp_1 = asinf(fabsf(var_22 + var_23 * (+0.0f / ceilf(var_24 / expf(-0.0f)))));
comp += tmp_1 - sinhf((-1.1886E18f + (-0.0f - var_25 + -0.0f + +1.8785E-43f / var_26)));
comp = (var_27 * -1.9121E3f + var_28 * +1.6747E4f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29);
  hipDeviceSynchronize();

  return 0;
}
