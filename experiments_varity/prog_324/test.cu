#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float* var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
for (int i=0; i < var_1; ++i) {
  comp += -1.9765E-42f / (-1.5601E-43f + cosf((+1.7294E36f / (var_4 - var_5))));
for (int i=0; i < var_2; ++i) {
  comp = -1.1387E-37f / var_7 + acosf((+1.3109E-42f - var_8 / (+1.7658E-36f + (-1.9130E36f / (+1.4417E-20f / var_9)))));
comp += +1.9470E35f / var_10 + (var_11 + (var_12 - -1.6893E36f));
var_6[i] = +0.0f;
comp = var_6[i] - (+1.3601E36f + var_13 - powf(+0.0f * (+1.2207E17f + +0.0f + (var_14 / +1.8077E-43f)), var_15 * +1.6241E34f));
}
for (int i=0; i < var_3; ++i) {
  comp += cosf(+1.3550E-18f / -1.8886E7f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float* tmp_7 = initPointer( atof(argv[7]) );
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
