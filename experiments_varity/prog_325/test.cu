#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float* var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = -1.8482E-36f;
var_2[i] = (var_4 / +1.3383E-35f + var_5 / +1.1280E34f * asinf((-1.8207E-18f - (var_6 / (+1.6148E-42f + -1.9208E-25f + +1.6295E35f)))));
comp = var_2[i] * tmp_1 / var_7 / var_8 - (+1.0124E-35f * +1.1107E36f / +1.9785E-37f);
comp = (var_9 * -1.2353E-14f * var_10);
for (int i=0; i < var_3; ++i) {
  comp += var_11 + var_12;
float tmp_2 = -1.8156E34f;
comp = tmp_2 / (var_13 / atanf(powf((var_14 * var_15 - (var_16 - var_17)), var_18 * +1.1786E36f)));
}
if (comp <= (+1.9894E35f * asinf((-1.7244E34f + fmodf(var_19 / (-1.9973E-37f / -1.7613E26f), +1.1054E-22f))))) {
  comp += var_20 / -0.0f;
comp += var_21 * -1.1049E34f + expf(+1.3526E-16f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float* tmp_3 = initPointer( atof(argv[3]) );
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22);
  hipDeviceSynchronize();

  return 0;
}
