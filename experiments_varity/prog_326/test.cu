#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    for (int i=0; i < var_3; ++i) {
      comp += (-1.1191E-36f / var_4);
if (comp <= (-1.6213E5f / var_5)) {
  comp += var_6 / tanhf(+1.8095E-42f);
comp += log10f(atanf(+1.4939E-41f));
comp += (var_7 / var_8);
comp += (var_9 - (-1.6014E34f - (+0.0f * sinf((var_10 / sinhf(atanf(atan2f((-1.5099E35f + -1.8193E36f), (+0.0f / (var_11 / (var_12 * var_13 * -1.0279E-44f * +1.5324E35f)))))))))));
}
if (comp == sqrtf((var_14 - var_15 - (+1.4097E-44f * -1.3049E34f)))) {
  comp += (+1.5851E-44f / var_16);
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17);
  hipDeviceSynchronize();

  return 0;
}
