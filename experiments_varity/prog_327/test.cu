#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29) {
for (int i=0; i < var_1; ++i) {
  comp += -1.9026E34f / (-0.0f * (-1.3540E10f * var_3 * +1.0384E-35f));
comp += cosf((var_4 * atan2f((+1.1936E14f + floorf(logf(-1.2351E36f + var_5))), var_6 - sinf(logf((var_7 / +1.9267E13f * var_8 * (+1.2148E-4f - +1.8449E35f / -1.4312E-41f)))))));
for (int i=0; i < var_2; ++i) {
  float tmp_1 = -1.2494E-42f;
comp = tmp_1 - var_9 * (var_10 + sinf(var_11 * var_12 + (+1.7058E34f * (var_13 + -1.8256E-37f))));
comp += tanhf(+1.9345E-35f);
}
if (comp > atan2f(fmodf((-1.2938E34f + asinf(-0.0f)), log10f(ldexpf(var_14 - -1.9462E-1f, 2))), var_15 * var_16 + (var_17 + (-1.4608E-41f + ceilf(atanf((var_18 * (+1.1557E34f / -0.0f * (var_19 - (var_20 - var_21)))))))))) {
  comp = (+1.4328E-41f - +1.8601E-22f + (+1.5481E-36f + var_22));
comp += (-1.3630E15f * sinf(-1.5582E25f));
float tmp_2 = (var_23 * coshf((+1.2650E13f * atan2f((var_24 / (-1.2761E-43f * var_25 - var_26 - +1.6921E36f)), -1.4653E34f))));
comp += tmp_2 / var_27 / (var_28 + (var_29 - acosf(tanhf((-0.0f / (+1.0854E-44f + (+1.1762E-36f * +1.3543E-37f)))))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30);
  hipDeviceSynchronize();

  return 0;
}
