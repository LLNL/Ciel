#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float* var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29) {
for (int i=0; i < var_1; ++i) {
  comp = (var_5 - var_6);
var_2[i] = (+1.2434E34f / (var_7 / (var_8 - var_9)));
comp += var_2[i] + (var_10 - -1.2385E-42f + (var_11 * +0.0f / (-1.9711E34f * +1.8458E34f)));
for (int i=0; i < var_3; ++i) {
  comp = -0.0f + (var_12 + (-1.2628E-44f / sinf(-1.0453E34f - +1.3967E34f + (var_13 * (var_14 / -1.9112E-43f)))));
comp += var_15 / var_16;
comp = (+1.4984E26f + (-1.6015E-44f - ceilf(fmodf((var_17 * +1.0433E-44f), +1.5480E35f))));
comp += -1.5524E34f / (var_18 - +1.6725E34f + var_19);
}
for (int i=0; i < var_4; ++i) {
  comp += (var_20 - var_21);
comp = (var_22 - var_23 + -0.0f);
}
if (comp >= (-1.6943E29f - var_24 / (var_25 / -1.8487E-42f))) {
  comp = expf(-1.1615E34f);
comp = +1.8922E-42f - var_26 + (+0.0f * (-1.8922E34f - (var_27 * var_28)));
float tmp_1 = -1.2620E-41f;
comp += tmp_1 - (-1.5293E-42f * -1.8979E-43f * -0.0f + +0.0f * var_29 * -0.0f);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float* tmp_3 = initPointer( atof(argv[3]) );
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30);
  hipDeviceSynchronize();

  return 0;
}
