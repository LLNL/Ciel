#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = +1.0197E-35f + (var_3 - var_4);
comp = tmp_1 + (-1.2326E36f + var_5);
if (comp > (var_6 + logf(logf(var_7 - var_8 / var_9 / (+1.0789E-36f - -1.6791E36f))))) {
  float tmp_2 = +1.7339E-36f;
comp = tmp_2 - (var_10 / +0.0f);
comp = sinf(-1.6953E-36f / atanf(+1.9214E-5f - ldexpf(var_11 * +1.9752E-8f, 2)));
}
if (comp > (-1.9102E35f / var_12 * var_13 * var_14)) {
  float tmp_3 = -1.5097E-37f - var_15;
float tmp_4 = +1.1459E-37f;
comp += tmp_4 - tmp_3 + ceilf(-1.3137E35f);
comp = +1.6520E28f / (var_16 - -1.7487E-36f - +1.4132E-35f + var_17);
}
for (int i=0; i < var_2; ++i) {
  comp += var_18 * -1.4563E28f;
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19);
  hipDeviceSynchronize();

  return 0;
}
