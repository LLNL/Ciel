#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5) {
comp += (-1.0050E-37f + var_2 + +0.0f - expf(-1.5145E-42f));
for (int i=0; i < var_1; ++i) {
  comp = -1.4568E-43f + -1.4872E34f;
float tmp_1 = +1.9446E-42f * -1.6293E36f * var_3;
comp += tmp_1 + +1.6737E36f + (var_4 - (-1.0178E8f / (-1.8077E21f * var_5)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6);
  hipDeviceSynchronize();

  return 0;
}
