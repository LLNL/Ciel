#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float* var_12,float var_13) {
if (comp >= var_3 * tanhf(+0.0f + -1.8035E-43f - powf(var_4 / (+1.8987E36f / ceilf(+0.0f)), -1.3667E-36f - +1.9443E-37f * (var_5 + var_6 / -1.4967E-35f)))) {
  for (int i=0; i < var_1; ++i) {
    comp += (var_7 * +1.7198E-41f);
if (comp >= (var_8 / (var_9 / var_10))) {
  comp += ldexpf(+0.0f + var_11, 2);
}
for (int i=0; i < var_2; ++i) {
  var_12[i] = -1.8572E14f;
comp = var_12[i] - +1.8380E-43f / -1.2608E-36f;
comp = (+1.4079E36f / (var_13 * -1.3413E-35f));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float* tmp_13 = initPointer( atof(argv[13]) );
  float tmp_14 = atof(argv[14]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14);
  hipDeviceSynchronize();

  return 0;
}
