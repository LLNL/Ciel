#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11) {
if (comp <= +0.0f + var_2 + (-1.5650E-42f - (+1.1859E-37f / var_3 / var_4))) {
  float tmp_1 = +1.2615E26f;
comp = tmp_1 - (var_5 / var_6 * cosf(var_7 - var_8));
comp += +1.8291E35f * (var_9 - -1.3872E-2f - var_10 - +1.5791E-37f);
for (int i=0; i < var_1; ++i) {
  comp += atan2f((+1.0602E-42f - -1.1852E35f - +1.5031E34f * +1.3088E-41f), (-0.0f * var_11 / -1.9901E36f));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12);
  hipDeviceSynchronize();

  return 0;
}
