#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6) {
comp += var_1 + cosf(-1.7142E-41f);
if (comp >= (+0.0f + (+1.0539E-36f * var_2))) {
  comp += ceilf(+1.2944E-43f);
}
if (comp >= (+1.2169E36f / +0.0f + +1.5210E7f - -1.4064E11f)) {
  float tmp_1 = (+1.3233E-43f / (var_3 - var_4 - var_5));
comp = tmp_1 + (+1.7426E-37f - +1.8406E-37f - cosf((-1.6288E-19f * (var_6 + (-1.9929E35f * -1.1170E36f)))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7);
  hipDeviceSynchronize();

  return 0;
}
