#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19) {
for (int i=0; i < var_1; ++i) {
  comp = -1.9786E-21f * +1.4840E-41f;
if (comp <= (-0.0f + (-1.7760E35f / var_3 + (var_4 * var_5)))) {
  float tmp_1 = -1.4259E-37f;
comp += tmp_1 + var_6 / (-1.2181E-28f - var_7 / var_8 * (var_9 * -0.0f));
float tmp_2 = powf(var_10 - -1.1517E13f / var_11, +1.0803E-41f);
comp = tmp_2 * var_12 - (+0.0f - (+1.0986E-37f * -1.5303E-35f * (var_13 + -1.8656E-5f)));
}
for (int i=0; i < var_2; ++i) {
  float tmp_3 = (-1.1301E-42f * log10f(+1.8012E-26f));
comp += tmp_3 * var_14 - +1.8300E34f - asinf(var_15 * var_16 - -1.0981E-42f);
float tmp_4 = -0.0f / (var_17 + var_18);
comp += tmp_4 * +0.0f - coshf(ceilf((+0.0f / var_19)));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20);
  hipDeviceSynchronize();

  return 0;
}
