#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22) {
for (int i=0; i < var_1; ++i) {
  float tmp_1 = -0.0f;
comp = tmp_1 + (-1.7002E-43f + -1.4704E34f - (-0.0f / floorf(var_2 + var_3)));
float tmp_2 = -0.0f;
comp += tmp_2 + (+0.0f + (-1.1545E-36f / (+1.2037E21f / var_4)));
if (comp > powf((-1.9519E-43f - +1.7762E-43f * (+0.0f * (+1.3483E34f / var_5))), -1.0702E-37f)) {
  comp += (var_6 + var_7 / var_8);
}
if (comp == -0.0f * +1.1074E-26f + (-1.2647E-43f - (var_9 + var_10))) {
  comp = var_11 + +0.0f + -1.8316E-36f - -1.1506E16f;
comp = -0.0f - powf(+1.0406E1f - (+1.9075E-35f + (var_12 * -1.3482E-35f - -1.3764E35f * -0.0f)), var_13 + sinhf(-0.0f));
}
if (comp >= var_14 / (+1.5558E-37f + var_15 * var_16)) {
  comp += (var_17 * var_18 / powf(-1.2662E-42f, -1.6396E-10f * var_19));
comp = (var_20 + +1.3093E36f);
comp = (-1.6917E-43f + fabsf(log10f((+1.2191E-37f / (var_21 / +0.0f / var_22)))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23);
  hipDeviceSynchronize();

  return 0;
}
