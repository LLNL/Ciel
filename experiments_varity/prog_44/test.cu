#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7) {
comp += (var_2 - (+1.7073E35f / (-1.7257E-42f * var_3)));
float tmp_1 = +1.8768E-42f;
float tmp_2 = -0.0f;
comp = tmp_2 + tmp_1 - sinf((var_4 * +0.0f - -1.1183E-5f - var_5));
for (int i=0; i < var_1; ++i) {
  comp += -0.0f + -1.3906E34f / floorf(cosf(sqrtf((var_6 - var_7))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8);
  hipDeviceSynchronize();

  return 0;
}
