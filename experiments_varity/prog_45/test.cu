#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21) {
comp = var_1 + -1.0156E-36f * var_2;
if (comp < (var_3 * (-0.0f / var_4))) {
  float tmp_1 = +1.3948E1f + +1.7148E35f * +1.4966E-1f;
comp += tmp_1 + (var_5 * sqrtf((var_6 - (var_7 * -1.3162E-42f))));
comp += +1.2144E-41f - (var_8 + -1.9306E-9f + var_9);
}
if (comp > -0.0f * (-1.2006E34f + var_10)) {
  float tmp_2 = +1.5972E34f;
float tmp_3 = var_11 - +1.4203E-42f / cosf(var_12 - fmodf(-1.6689E-43f - var_13, var_14 + sqrtf(+0.0f)));
comp = tmp_3 * tmp_2 + -0.0f - var_15 / (+1.4749E-44f + -1.5832E-43f);
comp = (var_16 / atan2f(var_17 / var_18 / ceilf((var_19 / (+1.8022E34f + var_20 / -1.6589E-37f + var_21))), (+0.0f / +1.7854E-37f)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22);
  hipDeviceSynchronize();

  return 0;
}
