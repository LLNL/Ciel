#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4) {
comp = asinf((+1.6287E-41f + (var_1 - -1.9177E-42f)));
if (comp > (-0.0f * ldexpf(log10f((-1.8145E-42f - sinhf((-1.5065E2f + sqrtf(var_2 * -1.6234E7f))))), 2))) {
  comp = -1.4046E-15f / log10f(cosf(var_3 + (-1.2819E35f / +0.0f * var_4)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5);
  hipDeviceSynchronize();

  return 0;
}
