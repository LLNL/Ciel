#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float* var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24) {
comp += -1.7064E-44f - var_3 - +1.0572E-42f - var_4;
for (int i=0; i < var_1; ++i) {
  float tmp_1 = (var_6 * +1.1598E-37f / (var_7 / floorf(logf((var_8 / -1.9263E-36f / var_9 * +1.7142E36f + var_10)))));
var_5[i] = log10f(tanhf(-0.0f));
float tmp_2 = -1.2157E34f;
comp = tmp_2 * var_5[i] / tmp_1 / var_11 / acosf(var_12 / +1.8103E-3f / (+0.0f * +0.0f));
}
for (int i=0; i < var_2; ++i) {
  comp = (var_13 + +1.9403E-41f);
}
if (comp >= sqrtf(var_14 + var_15 * +1.6571E34f)) {
  float tmp_3 = +1.6163E-35f * var_16;
float tmp_4 = (+1.8482E27f / var_17 * +1.0044E-43f);
comp = tmp_4 / tmp_3 * powf(coshf((+1.0795E-9f + (var_18 - +1.7313E21f / coshf(var_19 - var_20 / floorf(logf((-1.2496E-28f / -1.8555E22f * var_21))))))), coshf((+0.0f / var_22 / (var_23 * (-0.0f * (var_24 * +1.6037E-35f))))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float* tmp_6 = initPointer( atof(argv[6]) );
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25);
  hipDeviceSynchronize();

  return 0;
}
