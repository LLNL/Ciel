#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11) {
for (int i=0; i < var_1; ++i) {
  comp = (+1.5305E34f + -1.1993E13f + +1.7273E-44f);
comp += -0.0f + var_2 + var_3;
comp += (-1.8399E-36f * var_4);
if (comp >= (-1.4667E28f - +1.2950E-43f - +0.0f)) {
  comp += powf(expf((var_5 - coshf((var_6 / +0.0f - log10f(var_7 * var_8))))), +1.0475E-43f / (-1.8649E-37f / (+1.5649E-28f / +0.0f)));
comp += -1.5571E-35f * (var_9 + asinf((-1.1575E-36f - var_10 - var_11)));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12);
  hipDeviceSynchronize();

  return 0;
}
