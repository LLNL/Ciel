#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4) {
float tmp_1 = -1.8591E34f;
comp = tmp_1 * -1.3915E34f * (var_1 + (var_2 / (+1.4985E-20f - (var_3 + +1.4654E-44f))));
comp += (-1.0388E35f / var_4 * ceilf(ceilf(-1.6750E7f)));
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5);
  hipDeviceSynchronize();

  return 0;
}
