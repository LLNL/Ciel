#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26) {
if (comp < atanf(+0.0f * (var_1 / (-1.5211E-21f + +1.8764E-41f)))) {
  float tmp_1 = (+1.8938E-7f * (+1.8680E-42f - (var_2 * -1.5427E-36f * (var_3 * +1.7809E-41f))));
comp = tmp_1 + (-1.1484E-43f / -1.2457E5f - (var_4 + var_5 - var_6 - -1.2790E-37f));
comp = +1.2441E-42f * +1.8290E34f;
comp = (-1.3823E-37f / -1.7493E-4f + (+1.8868E-5f + -1.8733E-36f));
if (comp >= -1.5699E27f / var_7 + cosf(var_8 / (+0.0f * var_9))) {
  comp = fmodf(var_10 / -1.4101E-35f - var_11, var_12 / +1.0262E-16f);
comp += powf((var_13 + sinhf(+1.4771E-36f + (-1.6460E-41f * fmodf((+1.1910E34f * log10f(atan2f(expf((var_14 / (var_15 * (var_16 - -1.2970E35f)))), var_17 / var_18 + var_19 - (var_20 + var_21)))), +1.0790E-9f)))), -1.7601E-41f - (+1.2827E-44f / +0.0f - tanhf((var_22 + powf(sqrtf(-0.0f * -1.1247E-36f), -0.0f * var_23 - (var_24 / var_25 + (+1.4942E9f * var_26)))))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27);
  hipDeviceSynchronize();

  return 0;
}
