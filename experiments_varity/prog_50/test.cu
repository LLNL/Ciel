#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float* var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    comp += logf(+1.0806E-37f);
for (int i=0; i < var_3; ++i) {
  var_4[i] = (+1.1820E-37f - -0.0f - atanf(-0.0f));
comp += var_4[i] / (var_5 - (var_6 * (-0.0f - (var_7 * +1.1686E34f))));
comp = +0.0f - var_8;
}
if (comp < powf((var_9 - -1.1741E-44f), -1.2260E8f)) {
  comp = (var_10 * -1.2446E-36f + (+1.4488E11f - asinf(fabsf((-1.6245E11f + var_11 * (var_12 / -1.8190E21f / (-1.2619E-43f * +1.6124E18f)))))));
}
if (comp < (var_13 * (var_14 / (var_15 / var_16)))) {
  comp = (-1.3535E-36f / var_17);
comp = (-1.6105E-3f * var_18);
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float* tmp_5 = initPointer( atof(argv[5]) );
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19);
  hipDeviceSynchronize();

  return 0;
}
