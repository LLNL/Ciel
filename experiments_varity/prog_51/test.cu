#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    if (comp >= (+0.0f - var_3 + fmodf(-1.0742E36f, (var_4 / (+1.4778E-41f * -0.0f))))) {
      comp += (+1.9408E36f + (var_5 + (-1.8291E20f + var_6 - +1.9175E-44f)));
comp = cosf((+0.0f - (-1.3183E17f + +0.0f)));
if (comp == (-1.6748E-44f - var_7)) {
  comp = var_8 / +1.8234E-35f;
}
if (comp >= sinhf(var_9 * (var_10 - asinf(var_11 + +1.2600E36f - (var_12 + (-1.1193E-11f * +1.1195E-42f)))))) {
  comp = (var_13 * (+1.7241E34f + (var_14 * +1.5914E8f)));
comp = -1.2489E11f - (+1.6243E-42f - var_15);
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
