#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12) {
if (comp == (-0.0f * var_1)) {
  float tmp_1 = -1.0382E-41f;
float tmp_2 = (var_4 + (var_5 / +0.0f));
float tmp_3 = -1.3289E-43f;
comp = tmp_3 / tmp_2 * tmp_1 / (var_6 / acosf(var_7 - acosf((var_8 / (var_9 * var_10 / cosf((+1.9393E-43f + var_11)))))));
for (int i=0; i < var_2; ++i) {
  comp += +1.6621E7f + var_12;
}
for (int i=0; i < var_3; ++i) {
  float tmp_4 = -1.3294E-42f;
comp += tmp_4 - sinhf((-0.0f * ldexpf(+1.7197E-43f, 2)));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13);
  hipDeviceSynchronize();

  return 0;
}
