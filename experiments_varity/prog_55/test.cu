#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float* var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23) {
comp += coshf(-0.0f);
comp = (-1.5440E34f + log10f((-0.0f / powf(+1.0570E-29f, (var_4 / var_5 * var_6 - (var_7 / sinhf(+1.9926E-35f)))))));
for (int i=0; i < var_1; ++i) {
  comp += +1.1927E-44f + coshf(var_8 + +0.0f);
}
for (int i=0; i < var_2; ++i) {
  comp = (var_10 * (var_11 + (var_12 - var_13)));
comp += var_14 + (+0.0f - var_15 - -1.4749E-37f / -1.2554E2f);
var_9[i] = (-1.5388E-37f - var_16 + var_17 - var_18 + (-1.6455E-43f / +1.3705E-42f));
comp += var_9[i] * var_19 + -1.3408E-35f * -1.9019E-37f / +1.4400E-37f;
}
for (int i=0; i < var_3; ++i) {
  comp = +1.0472E6f * var_20;
comp += (+0.0f - +1.5887E-43f * (var_21 + (var_22 / var_23)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float* tmp_10 = initPointer( atof(argv[10]) );
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24);
  hipDeviceSynchronize();

  return 0;
}
