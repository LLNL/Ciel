#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    comp = +1.6770E-41f - var_5;
comp = -1.9317E-44f - (var_6 - +0.0f - sqrtf(-0.0f));
comp = (+1.1100E36f / (+1.9045E-44f / var_7));
for (int i=0; i < var_3; ++i) {
  float tmp_1 = (-1.2530E-37f / coshf(-1.4681E-36f / (+1.5111E-35f * +1.9243E-41f + var_8 / var_9)));
float tmp_2 = -1.6321E36f;
comp = tmp_2 * tmp_1 + (var_10 / var_11 - +1.4189E36f);
}
if (comp == var_12 - +1.8187E36f / -1.0852E-37f + (var_13 + +1.8867E-44f)) {
  comp = (var_14 - (var_15 / (+0.0f - var_16 / (+1.4631E-3f * var_17))));
float tmp_3 = var_18 / var_19 + var_20;
comp += tmp_3 - atanf(+0.0f - (-1.1109E36f / var_21));
comp = (+1.6903E36f / var_22 - +1.5772E-37f);
}
for (int i=0; i < var_4; ++i) {
  comp += -1.3680E-37f / +1.7708E36f * +1.1797E-44f + var_23;
comp = var_24 / +1.7194E21f * var_25 / -0.0f;
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26);
  hipDeviceSynchronize();

  return 0;
}
