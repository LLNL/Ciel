#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float* var_11,float var_12,float var_13,float var_14,float var_15) {
for (int i=0; i < var_1; ++i) {
  if (comp == var_2 * (+1.4715E-42f - -1.3782E35f)) {
    comp = (-1.4090E-41f - var_5 - (var_6 + var_7));
comp = -0.0f - var_8;
comp += var_9 * asinf(-1.4665E17f);
comp = (+0.0f + -1.0400E-35f - (var_10 * ceilf(expf(-1.9179E-35f))));
for (int i=0; i < var_3; ++i) {
  var_11[i] = (var_12 + (var_13 / +1.4589E-36f + (+1.9314E35f - (+1.0494E-43f * +1.7027E-37f))));
float tmp_1 = (var_14 + atanf(+1.3364E-8f));
float tmp_2 = -1.0532E-41f;
comp += tmp_2 - tmp_1 * var_11[i] - (+1.0629E-9f * (+1.4833E-37f - +1.7569E10f));
}
for (int i=0; i < var_4; ++i) {
  comp = -1.0301E-7f * -1.3249E23f * +1.4176E-35f;
comp = (+1.6993E-37f * (+1.0260E-35f * var_15));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float* tmp_12 = initPointer( atof(argv[12]) );
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
