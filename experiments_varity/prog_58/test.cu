#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float* var_6,float* var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float* var_14,float var_15) {
if (comp > +1.5118E34f * ceilf((var_4 + var_5))) {
  for (int i=0; i < var_1; ++i) {
    float tmp_1 = -1.3065E-36f;
float tmp_2 = +1.2489E3f;
comp += tmp_2 - tmp_1 * -1.6165E-36f + -1.1804E-44f;
for (int i=0; i < var_2; ++i) {
  var_6[i] = (var_8 * (-0.0f - sinhf(cosf(-0.0f * (var_9 * var_10 * var_11)))));
var_7[i] = -1.4603E36f;
comp = var_7[i] / var_6[i] - (+1.8357E-37f + var_12 * var_13);
comp = -1.6243E25f - -1.1032E36f / +1.0991E10f;
}
for (int i=0; i < var_3; ++i) {
  var_14[i] = +1.7771E-12f;
comp = var_14[i] - sqrtf(+1.4712E35f / +1.3325E-41f + -1.2108E-37f / (var_15 - acosf(ldexpf(sinf(+1.8169E-43f), 2))));
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float* tmp_7 = initPointer( atof(argv[7]) );
  float* tmp_8 = initPointer( atof(argv[8]) );
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float* tmp_15 = initPointer( atof(argv[15]) );
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
