#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float* var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19) {
if (comp <= (-0.0f - var_2 + var_3)) {
  float tmp_1 = (-1.2778E-37f + -0.0f);
comp = tmp_1 / (+0.0f * var_4 * (var_5 + ldexpf((-1.7950E36f - var_6), 2)));
for (int i=0; i < var_1; ++i) {
  comp = (var_8 + var_9);
var_7[i] = -0.0f + var_10;
comp = var_7[i] - +1.0158E-37f - (+1.5951E-44f + var_11 / sqrtf(-1.2438E36f));
}
if (comp >= (+0.0f - (+1.9889E36f / (+1.2658E-35f - (var_12 * (var_13 / var_14)))))) {
  float tmp_2 = +1.9292E-41f;
float tmp_3 = ldexpf(var_15 * var_16 + (var_17 + var_18), 2);
comp += tmp_3 / tmp_2 + var_19 * -1.4964E-14f;
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float* tmp_8 = initPointer( atof(argv[8]) );
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20);
  hipDeviceSynchronize();

  return 0;
}
