#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,int var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    float tmp_1 = +1.6666E35f;
comp = tmp_1 * (+1.4838E-42f - (+1.6315E-35f * (var_4 / var_5 + powf(var_6 * (-1.7386E10f + (var_7 * (var_8 + +1.2061E-37f))), -1.4054E35f - atan2f(-1.4572E8f * sinhf(+1.4628E-44f / var_9), log10f((-1.2724E19f / -0.0f / (var_10 / var_11 * -1.5114E-37f - var_12))))))));
comp += (var_13 + var_14);
for (int i=0; i < var_3; ++i) {
  comp = atan2f(-1.4266E-44f, +1.8751E-36f);
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15);
  hipDeviceSynchronize();

  return 0;
}
