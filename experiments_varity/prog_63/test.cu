#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float* var_4,float* var_5,float var_6,float var_7,float var_8) {
float tmp_1 = atanf(powf(+1.0120E-30f, -1.5396E-42f));
comp += tmp_1 / (var_2 / sinf((-1.2369E-41f - (+1.8445E35f - +1.8803E-36f / (+1.6660E-35f + (+0.0f * -1.6431E-41f))))));
if (comp >= (+1.0360E24f * var_3)) {
  comp += (-1.4489E-35f - +1.6863E-44f);
}
for (int i=0; i < var_1; ++i) {
  var_4[i] = (var_6 - var_7);
var_5[i] = -1.5390E36f;
comp = var_5[i] / var_4[i] / (var_8 - -1.6236E35f * +1.1913E-4f + -0.0f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float* tmp_5 = initPointer( atof(argv[5]) );
  float* tmp_6 = initPointer( atof(argv[6]) );
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9);
  hipDeviceSynchronize();

  return 0;
}
