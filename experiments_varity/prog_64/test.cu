#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,int var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float* var_14,float* var_15,float var_16,float var_17,float var_18,float var_19,float* var_20,float var_21,float var_22,float var_23,float var_24,float var_25) {
if (comp < +0.0f * var_1) {
  float tmp_1 = -1.8671E-35f;
comp += tmp_1 / (-1.3120E-37f / cosf((var_5 - -0.0f)));
comp = atanf(+0.0f);
comp = (+0.0f - -1.6877E-22f - (-1.7781E4f + var_6 / var_7));
for (int i=0; i < var_2; ++i) {
  float tmp_2 = +0.0f;
comp += tmp_2 * var_8 * var_9 - var_10 + var_11;
comp = +1.4191E8f - (var_12 * (-1.7829E-35f - (+1.0138E35f * var_13)));
}
for (int i=0; i < var_3; ++i) {
  float tmp_3 = (+1.5109E36f + expf(+1.8408E36f + (-1.8180E-44f - (var_16 * (+1.9724E36f / var_17)))));
var_14[i] = -1.2210E-35f;
var_15[i] = +1.4652E34f;
comp = var_15[i] / var_14[i] - tmp_3 + powf(+1.3100E-35f - -0.0f / +1.6113E-36f / (var_18 + (+1.8844E7f / +1.7618E-42f)), -1.0730E-9f / var_19);
}
for (int i=0; i < var_4; ++i) {
  comp = -1.9438E34f / (-1.9796E34f * +1.2381E-27f / var_21);
var_20[i] = -1.5665E35f;
float tmp_4 = var_22 * -0.0f + var_23;
comp += tmp_4 * var_20[i] * fabsf(-1.5118E-2f * coshf((var_24 * +1.0091E36f - (var_25 * sinf(tanhf(-0.0f))))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float* tmp_15 = initPointer( atof(argv[15]) );
  float* tmp_16 = initPointer( atof(argv[16]) );
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float* tmp_21 = initPointer( atof(argv[21]) );
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26);
  hipDeviceSynchronize();

  return 0;
}
