#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13) {
comp = (-1.6784E-35f + floorf(floorf((+1.9068E-20f + (-1.1894E26f - var_2 * var_3)))));
comp = logf((-1.2891E29f + powf((var_4 + +1.5339E36f - (-0.0f / var_5 - (+1.2398E12f * var_6))), var_7 - (var_8 + ldexpf(+1.6265E-28f, 2)))));
for (int i=0; i < var_1; ++i) {
  comp += cosf(+1.8719E-43f - var_9);
comp = var_10 * (var_11 * +1.8486E36f);
comp = -1.4844E-42f - var_12;
}
if (comp >= -1.6559E-42f + +1.6071E35f + -0.0f) {
  float tmp_1 = +1.0867E-14f;
float tmp_2 = (+1.7519E-37f / sqrtf(var_13 - -1.6463E17f));
comp += tmp_2 + tmp_1 / +0.0f - tanhf(+0.0f * +0.0f);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14);
  hipDeviceSynchronize();

  return 0;
}
