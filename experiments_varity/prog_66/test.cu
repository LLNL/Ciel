#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
comp += +1.1894E25f + var_2 * var_3;
comp = (var_4 / (var_5 + expf(logf(expf((var_6 - (+1.3359E-35f - (var_7 * +1.2098E-43f + (-0.0f - var_8)))))))));
for (int i=0; i < var_1; ++i) {
  float tmp_1 = -1.7854E-42f;
comp += tmp_1 * var_9 - -1.2149E-36f - var_10 - (+1.7379E34f + var_11);
comp += -1.1339E-44f - +1.2085E34f;
comp = (var_12 * (var_13 * sinf(-1.2139E-23f)));
}
if (comp >= +1.2367E-37f + -0.0f / (+1.7729E15f / (-1.2644E36f + var_14))) {
  comp = fabsf(-0.0f);
}
if (comp <= (+1.4110E34f + -0.0f)) {
  comp += var_15 - var_16;
float tmp_2 = +1.5469E-15f;
comp = tmp_2 - (var_17 + cosf((-0.0f + var_18 / ldexpf(+1.8918E36f / sqrtf(-1.3862E-35f + (-1.5382E29f * -1.3392E35f)), 2))));
comp = (var_19 - (+1.5999E-35f / +1.2273E-8f * (var_20 * +0.0f)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
