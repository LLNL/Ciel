#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,int var_7,int var_8,int var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21) {
for (int i=0; i < var_1; ++i) {
  for (int i=0; i < var_2; ++i) {
    if (comp > logf((var_3 - var_4 + var_5 / var_6))) {
      comp += acosf(-1.3222E-26f / -1.2847E34f / (-1.2264E-37f * (-1.6205E22f * var_10)));
comp += +1.0841E36f * asinf(sinhf(var_11 - +1.5999E-41f));
for (int i=0; i < var_7; ++i) {
  comp += var_12 - +1.3383E-36f;
comp += var_13 - atanf((var_14 / var_15 + var_16));
}
for (int i=0; i < var_8; ++i) {
  comp = var_17 + var_18 / ceilf(+1.1712E-2f);
}
for (int i=0; i < var_9; ++i) {
  comp = var_19 * acosf(var_20 * +1.7187E-37f * +1.3104E-36f);
float tmp_1 = +1.6491E-44f;
comp = tmp_1 / (+1.5345E36f - var_21);
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  int tmp_8 = atoi(argv[8]);
  int tmp_9 = atoi(argv[9]);
  int tmp_10 = atoi(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22);
  hipDeviceSynchronize();

  return 0;
}
