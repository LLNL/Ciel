#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3) {
float tmp_1 = +1.8785E-20f / atan2f((+1.7303E-35f / acosf(-1.3519E-35f / var_1)), var_2 / +1.9360E35f);
float tmp_2 = coshf(+1.8422E-44f);
float tmp_3 = +1.8333E35f;
comp += tmp_3 / tmp_2 - tmp_1 * +1.8329E-25f / (-0.0f * var_3);
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4);
  hipDeviceSynchronize();

  return 0;
}
