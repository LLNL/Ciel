#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8) {
if (comp < logf(sinhf(ceilf((+1.7086E-43f - -1.7085E23f - var_3 - var_4))))) {
  for (int i=0; i < var_1; ++i) {
    comp += (-1.9086E34f - +0.0f - var_5 - +1.4729E11f);
if (comp > (-1.1909E35f + (+1.7427E-35f * var_6 - -1.1457E35f))) {
  float tmp_1 = +1.4184E-42f;
float tmp_2 = -1.7144E-37f;
comp = tmp_2 / tmp_1 + -0.0f * (-1.7531E36f - var_7 - var_8);
comp += -1.5819E-36f / -1.0345E-12f - -1.3350E10f + +1.7103E-36f;
}
for (int i=0; i < var_2; ++i) {
  comp += (+1.4223E-42f - -1.8436E34f);
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9);
  hipDeviceSynchronize();

  return 0;
}
