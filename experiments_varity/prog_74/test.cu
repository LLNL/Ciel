#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9) {
comp = var_2 + -1.3192E-35f - -1.1696E17f;
for (int i=0; i < var_1; ++i) {
  comp = var_3 - (var_4 / var_5 / atan2f(-1.0880E-41f + var_6, var_7 * fmodf((+1.3229E-36f + +1.5420E25f), var_8 + -1.8255E-37f / log10f((-0.0f - (var_9 + (+1.2992E-44f * (+0.0f + -0.0f))))))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10);
  hipDeviceSynchronize();

  return 0;
}
