#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10) {
if (comp > (-1.3943E-36f / +0.0f - (var_2 / var_3))) {
  float tmp_1 = +1.5186E28f;
float tmp_2 = -1.1281E-1f;
comp += tmp_2 * tmp_1 / cosf(logf(-1.2048E7f));
comp += coshf((-1.0964E34f * -1.2267E-44f - var_4));
if (comp < (var_5 * var_6)) {
  float tmp_3 = (+1.3174E28f - +0.0f);
float tmp_4 = +1.8966E-7f;
comp += tmp_4 * tmp_3 * -1.5344E-44f / (+1.0506E18f * (var_7 - (+1.6832E19f - +1.2449E-36f)));
}
for (int i=0; i < var_1; ++i) {
  float tmp_5 = +1.7930E-43f + (-1.3392E35f / logf((+1.7318E0f * -1.4533E-41f)));
comp += tmp_5 - var_8 / fmodf(+1.3093E-43f, +1.7079E34f + (var_9 * -1.9509E35f - var_10));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
