#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15) {
comp = fmodf(+1.4302E35f, var_1 / tanhf((+1.6427E-37f + (-0.0f / +1.6944E36f + (var_2 / var_3)))));
if (comp < acosf(ceilf((-1.7759E36f + coshf(log10f(sinf(-1.9409E36f - sqrtf(tanhf((+1.9337E35f + -1.5819E34f * var_4 - (var_5 * var_6))))))))))) {
  comp += (var_7 - -1.5551E-24f + var_8 / -1.3514E-41f / -1.8846E-43f + var_9);
float tmp_1 = (var_10 - var_11);
comp += tmp_1 - -1.2855E-36f + var_12;
}
if (comp == (var_13 / -1.4242E35f)) {
  comp = -1.8524E-1f * var_14 * var_15;
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16);
  hipDeviceSynchronize();

  return 0;
}
