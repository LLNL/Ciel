#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float* var_3,float* var_4,float var_5,float var_6) {
if (comp == (-1.2118E35f * var_2)) {
  for (int i=0; i < var_1; ++i) {
    var_3[i] = +1.1286E-42f;
var_4[i] = +1.9543E-35f;
comp += var_4[i] / var_3[i] + (var_5 * var_6);
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float* tmp_4 = initPointer( atof(argv[4]) );
  float* tmp_5 = initPointer( atof(argv[5]) );
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7);
  hipDeviceSynchronize();

  return 0;
}
