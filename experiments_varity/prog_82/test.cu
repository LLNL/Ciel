#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float* var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27) {
if (comp <= (-0.0f * sinhf((+1.5104E-35f + var_3)))) {
  float tmp_1 = (var_4 - var_5 * var_6 - var_7);
comp = tmp_1 - (+1.0667E-36f * var_8);
if (comp > var_9 - var_10 + (-1.1912E-37f - var_11)) {
  comp = cosf((-1.5333E-37f * -0.0f));
comp += -1.2488E-41f * (-1.2186E-43f - (var_12 * (-1.2041E-42f - var_13 / -1.1994E-44f)));
comp += +1.4314E-43f / +1.2993E-28f + +1.5299E-15f - (var_14 - log10f((var_15 - +1.8415E-37f * -1.3874E-44f - +1.5091E35f)));
comp += -0.0f + -1.8204E-37f;
}
for (int i=0; i < var_1; ++i) {
  comp = var_16 - var_17 + (-1.4232E34f + (-1.3987E-44f * -0.0f * +0.0f));
}
for (int i=0; i < var_2; ++i) {
  float tmp_2 = +1.1386E-35f + var_19 - +0.0f + var_20 / -1.2414E-37f * +1.5185E-43f;
comp = tmp_2 / (var_21 * (-1.4031E13f + var_22 / (var_23 + var_24)));
var_18[i] = (var_25 / (-1.9526E-24f / +1.8340E-4f));
comp += var_18[i] / (var_26 - (var_27 - -1.7063E-37f * +1.9946E-44f));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float* tmp_19 = initPointer( atof(argv[19]) );
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28);
  hipDeviceSynchronize();

  return 0;
}
