#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18) {
if (comp == (var_3 / (+1.7996E35f + (var_4 - (+1.5343E-35f + (+1.7182E-44f * var_5)))))) {
  for (int i=0; i < var_1; ++i) {
    for (int i=0; i < var_2; ++i) {
      float tmp_1 = -0.0f;
comp += tmp_1 / (var_6 - +1.9153E-44f);
if (comp <= var_7 / var_8 - (var_9 / (var_10 / var_11))) {
  float tmp_2 = -0.0f;
float tmp_3 = -1.0340E7f;
comp = tmp_3 / tmp_2 + +1.2027E-36f - var_12;
}
if (comp <= -1.0190E-36f / (var_13 + -1.6377E-36f / (var_14 / var_15 - var_16))) {
  float tmp_4 = -1.7022E34f;
float tmp_5 = +1.0174E-44f;
comp += tmp_5 + tmp_4 * (var_17 - (+1.9672E-25f + (var_18 * -1.7206E-35f)));
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19);
  hipDeviceSynchronize();

  return 0;
}
