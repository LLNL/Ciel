#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10) {
float tmp_1 = -1.6059E-36f;
comp = tmp_1 + (var_2 - (+0.0f / -0.0f + -0.0f));
comp = var_3 / (+1.3843E-22f * var_4 + (+1.2547E2f + -1.0924E-43f));
if (comp <= sinf((var_5 - -0.0f * +1.3446E35f - -1.2055E-35f * (var_6 / var_7)))) {
  comp = ldexpf(+1.7518E36f / -1.5137E-42f / asinf(-1.5513E24f + (var_8 * +1.3731E-21f)), 2);
float tmp_2 = -1.6911E-42f;
comp += tmp_2 * (-1.8404E-3f / var_9);
}
for (int i=0; i < var_1; ++i) {
  comp = fabsf(var_10 / (+0.0f / +1.4986E11f * +1.7254E-36f));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11);
  hipDeviceSynchronize();

  return 0;
}
