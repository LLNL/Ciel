#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float* var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25) {
comp += -0.0f / (+1.2561E-36f * var_2);
float tmp_1 = -1.5926E34f;
comp = tmp_1 + var_3 - var_4 / -1.1094E-35f / (+1.6490E-42f - +1.2148E-44f);
comp += (var_5 + (var_6 * -1.7500E-36f));
for (int i=0; i < var_1; ++i) {
  comp += -1.3421E-27f + var_8;
float tmp_2 = +1.1335E34f;
var_7[i] = -1.9658E35f;
comp += var_7[i] / tmp_2 * +1.2695E36f / (-1.7763E-36f / sinhf((+1.4111E34f + var_9 + (var_10 + fmodf(sinf((var_11 * (+1.2423E-36f / -1.1896E-6f))), (var_12 * expf((+1.3143E-42f / (-1.7207E-30f / var_13)))))))));
}
if (comp <= (-1.1836E-3f / var_14 - -1.8748E-37f)) {
  comp += +1.8424E-36f - (var_15 * (var_16 - +1.4196E35f * var_17));
float tmp_3 = (+1.7800E10f / -1.7184E3f / +1.5816E-5f - (+1.3587E35f / acosf((var_18 - var_19))));
comp = tmp_3 / (+0.0f - (var_20 * coshf((+1.7935E-43f - fabsf((var_21 / (-1.8991E-37f / var_22)))))));
}
if (comp >= (var_23 + var_24)) {
  comp += log10f(+1.1894E-37f / var_25);
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float* tmp_8 = initPointer( atof(argv[8]) );
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26);
  hipDeviceSynchronize();

  return 0;
}
