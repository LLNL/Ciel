#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14) {
comp = (-1.4149E17f / -1.2899E-8f);
for (int i=0; i < var_1; ++i) {
  float tmp_1 = (var_2 - (var_3 / -1.9107E-14f + +0.0f - (var_4 + +1.8370E-25f)));
comp += tmp_1 - logf(log10f((-1.7442E36f / var_5 * (var_6 * var_7))));
comp += (-1.5960E-43f - (-1.7254E35f * -1.0943E35f + var_8));
}
if (comp <= (+1.4929E10f + var_9)) {
  float tmp_2 = +1.3666E-36f;
float tmp_3 = +1.7855E-11f;
comp += tmp_3 - tmp_2 * var_10 / floorf((var_11 / sqrtf(var_12 * +1.2344E34f / var_13 - var_14)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15);
  hipDeviceSynchronize();

  return 0;
}
