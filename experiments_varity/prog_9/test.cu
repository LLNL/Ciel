#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,int var_2,float var_3,float var_4,float* var_5,float* var_6,float* var_7,float var_8,float var_9,float var_10,float* var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20) {
comp = (+1.0268E34f * +1.6547E9f * (var_3 - var_4));
for (int i=0; i < var_1; ++i) {
  var_5[i] = -1.5800E34f;
var_6[i] = sinf((var_8 + (var_9 - cosf(-1.8832E-43f))));
var_7[i] = +1.8731E34f;
comp += var_7[i] + var_6[i] * var_5[i] * +1.5318E-35f + (-1.0675E-37f - var_10);
}
for (int i=0; i < var_2; ++i) {
  var_11[i] = -0.0f;
float tmp_1 = -1.9597E36f;
comp = tmp_1 * var_11[i] - (var_12 / (-1.4935E-36f * (var_13 * +1.5838E-37f)));
}
if (comp >= (var_14 * (-1.5700E-26f * +1.2359E-42f + cosf((-1.2237E-37f * var_15 / +1.8914E-44f / var_16))))) {
  float tmp_2 = +1.2200E35f;
comp += tmp_2 - (-1.8775E-41f * (-0.0f / +1.9521E34f / -1.7387E34f));
comp = var_17 * logf(var_18 / (-1.3151E-37f * (var_19 * var_20 * +1.4525E-29f)));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  int tmp_3 = atoi(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float* tmp_6 = initPointer( atof(argv[6]) );
  float* tmp_7 = initPointer( atof(argv[7]) );
  float* tmp_8 = initPointer( atof(argv[8]) );
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float* tmp_12 = initPointer( atof(argv[12]) );
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21);
  hipDeviceSynchronize();

  return 0;
}
