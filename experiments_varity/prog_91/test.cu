#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,int var_2,int var_3,int var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float* var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23) {
if (comp == var_1 + (+0.0f / -1.3182E-43f)) {
  comp = (+0.0f * (+1.7948E-41f - (+1.2425E-42f - var_5 / var_6)));
comp += +0.0f - var_7 + (-1.0084E-42f * var_8);
for (int i=0; i < var_2; ++i) {
  comp = var_9 / (+1.0558E-42f - -1.7427E-35f);
}
for (int i=0; i < var_3; ++i) {
  comp = -1.1553E-42f - (var_10 / -1.4254E-43f * ceilf((-1.2995E34f / var_11)));
}
for (int i=0; i < var_4; ++i) {
  comp += (-1.3662E-37f + +1.1758E-42f);
var_12[i] = (-0.0f * powf((+1.0426E-18f / -1.5964E36f), +1.8620E34f - (-1.6251E-35f / var_13)));
comp = var_12[i] + (+1.2858E-36f - sqrtf(expf((+1.6876E-44f / asinf(var_14 / cosf((+1.7135E35f - var_15 / -1.7406E-36f + var_16 * var_17)))))));
comp = var_18 / (var_19 / atan2f((var_20 - (+1.7421E-41f * (var_21 + -0.0f + (-1.6410E-36f / +1.8407E-36f)))), sqrtf((+0.0f + (var_22 * var_23)))));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  int tmp_3 = atoi(argv[3]);
  int tmp_4 = atoi(argv[4]);
  int tmp_5 = atoi(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float* tmp_13 = initPointer( atof(argv[13]) );
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24);
  hipDeviceSynchronize();

  return 0;
}
