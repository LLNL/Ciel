#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float* var_8,int var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float* var_19,float var_20,float var_21,float var_22,float var_23) {
if (comp <= (var_2 / atanf((var_3 - var_4)))) {
  if (comp < (var_5 - -0.0f * var_6 - var_7)) {
    for (int i=0; i < var_1; ++i) {
      var_8[i] = -1.3654E-22f * var_10;
comp += var_8[i] + ceilf(-1.8553E-19f + sinhf(-1.6889E14f - (var_11 + (+1.8238E-2f / var_12))));
comp += (var_13 * cosf((var_14 + -1.3239E1f)));
if (comp > (+0.0f - (var_15 - -1.5815E36f))) {
  comp += -1.8948E-42f * (var_16 - (var_17 / sinhf(sinf(sinhf(-0.0f / +1.9691E35f - -1.6180E-15f)))));
float tmp_1 = +1.8446E34f;
comp = tmp_1 - -1.7162E-28f / -1.9701E-43f / (var_18 - tanhf(log10f(-0.0f)));
comp = acosf(+0.0f);
}
for (int i=0; i < var_9; ++i) {
  var_19[i] = -1.3559E-41f;
float tmp_2 = -1.6030E34f;
comp += tmp_2 + var_19[i] / var_20 + tanhf((var_21 / var_22 / +1.9554E-35f + var_23));
}
}
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float* tmp_9 = initPointer( atof(argv[9]) );
  int tmp_10 = atoi(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float* tmp_20 = initPointer( atof(argv[20]) );
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24);
  hipDeviceSynchronize();

  return 0;
}
