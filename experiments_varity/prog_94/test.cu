#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,int var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30,float var_31,float var_32,float var_33,float var_34,float var_35) {
if (comp < var_1 + -1.0521E9f * (var_2 * var_3 - var_4)) {
  comp = ldexpf((+1.9960E-37f / (-1.4857E-35f - var_6 + (var_7 - var_8))), 2);
if (comp > cosf(+1.3397E-44f)) {
  comp = (+1.3889E-44f * (+0.0f + -1.2855E36f + var_9 - +1.7266E34f));
comp = +1.1849E-36f - var_10 - var_11;
comp += var_12 + -1.7855E-41f / (var_13 - tanhf(-1.0058E34f + var_14 - +1.0074E15f + (+1.8686E36f / var_15)));
comp = (+1.6658E-43f / -0.0f);
}
if (comp >= fmodf(fabsf((var_16 + (var_17 - -1.3960E36f))), (var_18 + (-1.0965E36f - cosf(atanf(coshf((var_19 - (var_20 / var_21 - var_22))))))))) {
  float tmp_1 = +1.8913E-41f;
comp += tmp_1 / -1.3136E-35f * var_23 - -1.8562E-42f;
}
for (int i=0; i < var_5; ++i) {
  comp += (+1.7116E-44f * var_24 / var_25 + var_26);
float tmp_2 = (var_27 + (var_28 + var_29));
comp += tmp_2 * var_30 + (var_31 - (var_32 + +0.0f));
comp += (var_33 / (var_34 + var_35 * +1.4589E19f));
}
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  int tmp_6 = atoi(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);
  float tmp_33 = atof(argv[33]);
  float tmp_34 = atof(argv[34]);
  float tmp_35 = atof(argv[35]);
  float tmp_36 = atof(argv[36]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32,tmp_33,tmp_34,tmp_35,tmp_36);
  hipDeviceSynchronize();

  return 0;
}
