#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, float var_1,float var_2,float var_3,float var_4,float var_5) {
comp = (+1.8734E34f + (-1.0251E-35f / var_1 / var_2 + -0.0f));
comp += ldexpf(var_3 - (-1.6947E-35f + +0.0f - -1.9401E1f - var_4 + var_5), 2);
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  float tmp_2 = atof(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6);
  hipDeviceSynchronize();

  return 0;
}
