#include "hip/hip_runtime.h"

/* This is a automatically generated test. Do not modify */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "common.h"

__global__
void compute(float comp, int var_1,float var_2,float var_3,float var_4,float var_5,float var_6,float var_7,float var_8,float var_9,float var_10,float var_11,float var_12,float var_13,float var_14,float var_15,float var_16,float var_17,float var_18,float var_19,float var_20,float var_21,float var_22,float var_23,float var_24,float var_25,float var_26,float var_27,float var_28,float var_29,float var_30,float var_31,float var_32) {
comp = -1.2201E4f - (-1.0292E8f - ldexpf((var_2 - (+1.8362E36f + var_3)), 2));
float tmp_1 = var_4 * +1.1051E36f / +1.6872E25f - +1.5355E35f;
comp += tmp_1 / logf(floorf(cosf(fmodf((var_5 - var_6 * -1.8305E-36f), (-1.4240E-44f - var_7)))));
comp += var_8 / acosf(var_9 + atanf(+0.0f));
for (int i=0; i < var_1; ++i) {
  comp += ldexpf(var_10 + (+1.1009E7f / var_11 / var_12 - (var_13 * var_14)), 2);
comp = +1.1117E35f - (-1.6178E-37f - var_15 / var_16 - -0.0f - -1.0436E18f);
comp = +1.1063E-42f - ldexpf((+1.3389E26f / asinf(-1.3196E-44f + var_17 / (-1.5437E17f - var_18))), 2);
}
if (comp >= var_19 / var_20 + +1.4726E35f) {
  float tmp_2 = fabsf(+1.0581E-41f);
comp += tmp_2 - (var_21 - var_22);
comp = (+1.4737E-42f + -1.9420E20f);
comp += (+1.4486E36f + +0.0f / fabsf(-1.7514E-36f / +1.0238E-44f - -0.0f / (var_23 * var_24)));
}
if (comp < (-1.3332E-36f - -1.2121E-42f - (var_25 + var_26))) {
  comp += ldexpf(-1.2136E-43f, 2);
comp = (+1.0368E-36f / -0.0f);
comp = (-1.6321E34f * (-0.0f - powf((-1.6334E-35f * (+1.8962E35f * (var_27 * (+1.1051E-41f + var_28)))), +1.5735E-43f + (var_29 * var_30 / var_31 - var_32))));
}
   dump(&comp, sizeof(comp));

}

float* initPointer(float v) {
  float *ret = (float*) malloc(sizeof(float)*10);
  for(int i=0; i < 10; ++i)
    ret[i] = v;
  float* ret2;
	hipMalloc(&ret2, sizeof(float) * 10);
	hipMemcpy(ret2, ret, sizeof(float) * 10, hipMemcpyHostToDevice);
	return ret2;
}

int main(int argc, char** argv) {
/* Program variables */

  float tmp_1 = atof(argv[1]);
  int tmp_2 = atoi(argv[2]);
  float tmp_3 = atof(argv[3]);
  float tmp_4 = atof(argv[4]);
  float tmp_5 = atof(argv[5]);
  float tmp_6 = atof(argv[6]);
  float tmp_7 = atof(argv[7]);
  float tmp_8 = atof(argv[8]);
  float tmp_9 = atof(argv[9]);
  float tmp_10 = atof(argv[10]);
  float tmp_11 = atof(argv[11]);
  float tmp_12 = atof(argv[12]);
  float tmp_13 = atof(argv[13]);
  float tmp_14 = atof(argv[14]);
  float tmp_15 = atof(argv[15]);
  float tmp_16 = atof(argv[16]);
  float tmp_17 = atof(argv[17]);
  float tmp_18 = atof(argv[18]);
  float tmp_19 = atof(argv[19]);
  float tmp_20 = atof(argv[20]);
  float tmp_21 = atof(argv[21]);
  float tmp_22 = atof(argv[22]);
  float tmp_23 = atof(argv[23]);
  float tmp_24 = atof(argv[24]);
  float tmp_25 = atof(argv[25]);
  float tmp_26 = atof(argv[26]);
  float tmp_27 = atof(argv[27]);
  float tmp_28 = atof(argv[28]);
  float tmp_29 = atof(argv[29]);
  float tmp_30 = atof(argv[30]);
  float tmp_31 = atof(argv[31]);
  float tmp_32 = atof(argv[32]);
  float tmp_33 = atof(argv[33]);

  compute<<<1,1>>>(tmp_1,tmp_2,tmp_3,tmp_4,tmp_5,tmp_6,tmp_7,tmp_8,tmp_9,tmp_10,tmp_11,tmp_12,tmp_13,tmp_14,tmp_15,tmp_16,tmp_17,tmp_18,tmp_19,tmp_20,tmp_21,tmp_22,tmp_23,tmp_24,tmp_25,tmp_26,tmp_27,tmp_28,tmp_29,tmp_30,tmp_31,tmp_32,tmp_33);
  hipDeviceSynchronize();

  return 0;
}
